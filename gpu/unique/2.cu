#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include "util.h"

int main(void)
{
  // generate 32M random numbers serially
  thrust::host_vector<int> h_vec(1024 << 20);
  // thrust::host_vector<int> h_vec(1024 << 8);
  std::generate(h_vec.begin(), h_vec.end(), rand);

  static int counter;
  static int size;
  
  size = h_vec.size();

  std::cout << "size:" << size << std::endl;

  // transfer data to the device
  thrust::device_vector<int> d_vec = h_vec;

  // sort data on the device (846M keys per second on GeForce GTX 480)
  thrust::sort(d_vec.begin(), d_vec.end());

  hipEvent_t     start, stop;
  HANDLE_ERROR( hipEventCreate( &start ) );
  HANDLE_ERROR( hipEventCreate( &stop ) );
  HANDLE_ERROR( hipEventRecord( start, 0 ) );

  thrust::unique(d_vec.begin(),d_vec.end());

  HANDLE_ERROR( hipEventRecord( stop, 0 ) );
  HANDLE_ERROR( hipEventSynchronize( stop ) );
  float   elapsedTime;
  HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );
  printf( "Time to reorder array and points to the first repeated value:  %3.1f ms\n", elapsedTime );

  size = d_vec.size();

  std::cout << "size:" << size << std::endl;

  // transfer data back to host
  thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());

  /*
  counter = 0;
  for(auto itr = h_vec.begin(); itr != h_vec.end(); ++itr)
  {
  	   if(counter % 10000==0)
	             std::cout << *itr << std::endl;

           counter = counter + 1;
  }
  */

  return 0;
}
