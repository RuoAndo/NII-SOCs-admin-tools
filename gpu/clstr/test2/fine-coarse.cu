#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <chrono>
#include <random>
#include <vector>

#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>
#include <random>

#include "timer.h"

#include "csv.hpp"
using namespace std;

// A small data structure to do RAII for a dataset of 2-dimensional points.
struct Data {
  explicit Data(int size) : size(size), bytes(size * sizeof(float)) {
    hipMalloc(&x, bytes);
    hipMalloc(&y, bytes);
  }

  Data(int size, std::vector<float>& h_x, std::vector<float>& h_y)
  : size(size), bytes(size * sizeof(float)) {
    hipMalloc(&x, bytes);
    hipMalloc(&y, bytes);
    hipMemcpy(x, h_x.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(y, h_y.data(), bytes, hipMemcpyHostToDevice);
  }

  ~Data() {
    hipFree(x);
    hipFree(y);
  }

  void clear() {
    hipMemset(x, 0, bytes);
    hipMemset(y, 0, bytes);
  }

  float* x{nullptr};
  float* y{nullptr};
  int size{0};
  int bytes{0};
};

__device__ float
squared_l2_distance(float x_1, float y_1, float x_2, float y_2) {
  return (x_1 - x_2) * (x_1 - x_2) + (y_1 - y_2) * (y_1 - y_2);
}

// In the assignment step, each point (thread) computes its distance to each
// cluster centroid and adds its x and y values to the sum of its closest
// centroid, as well as incrementing that centroid's count of assigned points.
__global__ void assign_clusters(const float* __restrict__ data_x,
                                const float* __restrict__ data_y,
                                int data_size,
                                const float* __restrict__ means_x,
                                const float* __restrict__ means_y,
                                float* __restrict__ new_sums_x,
                                float* __restrict__ new_sums_y,
                                int k,
                                int* __restrict__ counts,
				int* clusterNo) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= data_size) return;

  // printf("data_size %d \n", data_size);

  // Make global loads once.
  const float x = data_x[index];
  const float y = data_y[index];

  float best_distance = FLT_MAX;
  int best_cluster = 0;
  for (int cluster = 0; cluster < k; ++cluster) {
    const float distance =
        squared_l2_distance(x, y, means_x[cluster], means_y[cluster]);
    if (distance < best_distance) {
      best_distance = distance;
      best_cluster = cluster;
      // printf("best_cluster %d \n", cluster);
      clusterNo[index] = cluster;
    }
  }

  // Slow but simple.
  atomicAdd(&new_sums_x[best_cluster], x);
  atomicAdd(&new_sums_y[best_cluster], y);
  atomicAdd(&counts[best_cluster], 1);
}

// Each thread is one cluster, which just recomputes its coordinates as the mean
// of all points assigned to it.
__global__ void compute_new_means(float* __restrict__ means_x,
                                  float* __restrict__ means_y,
                                  const float* __restrict__ new_sum_x,
                                  const float* __restrict__ new_sum_y,
                                  const int* __restrict__ counts) {
  const int cluster = threadIdx.x;
  // Threshold count to turn 0/0 into 0/1.
  const int count = max(1, counts[cluster]);
  means_x[cluster] = new_sum_x[cluster] / count;
  means_y[cluster] = new_sum_y[cluster] / count;
}



__global__ void fine_reduce(const float* __restrict__ data_x,
                            const float* __restrict__ data_y,
                            int data_size,
                            const float* __restrict__ means_x,
                            const float* __restrict__ means_y,
                            float* __restrict__ new_sums_x,
                            float* __restrict__ new_sums_y,
                            int k,
                            int* __restrict__ counts,
			    int* clusterNo) {
  // Essentially three dimensional: n * x, n * y, n * counts.
  extern __shared__ float shared_data[];

  const int local_index = threadIdx.x;
  const int global_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (global_index >= data_size) return;

  // Load the mean values into shared memory.
  if (local_index < k) {
    shared_data[local_index] = means_x[local_index];
    shared_data[k + local_index] = means_y[local_index];
  }

  __syncthreads();

  // Assignment step.

  // Load once here.
  const float x_value = data_x[global_index];
  const float y_value = data_y[global_index];

  float best_distance = FLT_MAX;
  int best_cluster = -1;
  for (int cluster = 0; cluster < k; ++cluster) {
    const float distance = squared_l2_distance(x_value,
                                               y_value,
                                               shared_data[cluster],
                                               shared_data[k + cluster]);
    if (distance < best_distance) {
      best_distance = distance;
      best_cluster = cluster;
      clusterNo[global_index] = cluster;
    }
  }

  __syncthreads();

  // Reduction step.

  const int x = local_index;
  const int y = local_index + blockDim.x;
  const int count = local_index + blockDim.x + blockDim.x;

  for (int cluster = 0; cluster < k; ++cluster) {
    // Zeros if this point (thread) is not assigned to the cluster, else the
    // values of the point.
    shared_data[x] = (best_cluster == cluster) ? x_value : 0;
    shared_data[y] = (best_cluster == cluster) ? y_value : 0;
    shared_data[count] = (best_cluster == cluster) ? 1 : 0;
    __syncthreads();

    // Tree-reduction for this cluster.
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
      if (local_index < stride) {
        shared_data[x] += shared_data[x + stride];
        shared_data[y] += shared_data[y + stride];
        shared_data[count] += shared_data[count + stride];
      }
      __syncthreads();
    }

    // Now shared_data[0] holds the sum for x.

    if (local_index == 0) {
      const int cluster_index = blockIdx.x * k + cluster;
      new_sums_x[cluster_index] = shared_data[x];
      new_sums_y[cluster_index] = shared_data[y];
      counts[cluster_index] = shared_data[count];
    }
    __syncthreads();
  }
}

__global__ void coarse_reduce(float* __restrict__ means_x,
                              float* __restrict__ means_y,
                              float* __restrict__ new_sum_x,
                              float* __restrict__ new_sum_y,
                              int k,
                              int* __restrict__ counts) {
  extern __shared__ float shared_data[];

  const int index = threadIdx.x;
  const int y_offset = blockDim.x;

  // Load into shared memory for more efficient reduction.
  shared_data[index] = new_sum_x[index];
  shared_data[y_offset + index] = new_sum_y[index];
  __syncthreads();

  for (int stride = blockDim.x / 2; stride >= k; stride /= 2) {
    if (index < stride) {
      shared_data[index] += shared_data[index + stride];
      shared_data[y_offset + index] += shared_data[y_offset + index + stride];
    }
    __syncthreads();
  }

  // The first k threads can recompute their clusters' means now.
  if (index < k) {
    const int count = max(1, counts[index]);
    means_x[index] = new_sum_x[index] / count;
    means_y[index] = new_sum_y[index] / count;
    new_sum_y[index] = 0;
    new_sum_x[index] = 0;
    counts[index] = 0;
  }
}
int main(int argc, const char* argv[]) {
  std::vector<float> h_x;
  std::vector<float> h_y;

  unsigned int t, travdirtime;

  // Load x and y into host vectors ... (omitted)

  int N = atoi(argv[2]);
  int k = atoi(argv[4]);
  int number_of_iterations = atoi(argv[3]);

  const string csv_file = std::string(argv[1]); 
  vector<vector<string>> data2; 

  Csv objCsv(csv_file);
  if (!objCsv.getCsv(data2)) {
     cout << "read ERROR" << endl;
     return 1;
  }

  // for (int row = 0; row < data2.size(); row++) {
  for (int row = 0; row < 1024; row++) {
      vector<string> rec = data2[row]; 
      h_x.push_back(std::stof(rec[0]));
      h_y.push_back(std::stof(rec[1]));
  }

  start_timer(&t); 

  const size_t number_of_elements = h_x.size();
  Data d_data(number_of_elements, h_x, h_y);

  // Random shuffle the data and pick the first
  // k points (i.e. k random points).
  std::random_device seed;
  std::mt19937 rng(seed());
  std::shuffle(h_x.begin(), h_x.end(), rng);
  std::shuffle(h_y.begin(), h_y.end(), rng);
  Data d_means(k, h_x, h_y);
  Data d_sums(k);

  int* d_counts;
  hipMalloc(&d_counts, k * sizeof(int));
  hipMemset(d_counts, 0, k * sizeof(int));

  int* h_counts;
  h_counts = (int *)malloc(k * sizeof(int));

  int* h_clusterNo;
  h_clusterNo = (int *)malloc(N * sizeof(int)); 

  int* d_clusterNo;
  hipMalloc(&d_clusterNo, N * sizeof(int));
  hipMemset(d_clusterNo, 0, N * sizeof(int));

  const int threads = N;
  const int blocks = (number_of_elements + threads - 1) / threads;

  const int fine_shared_memory = 3 * threads * sizeof(float);
// * 2 for x and y. Will have k * blocks threads for the coarse reduction.
const int coarse_shared_memory = 2 * k * blocks * sizeof(float);
// ...
for (size_t iteration = 0; iteration < number_of_iterations; ++iteration) {
  fine_reduce<<<blocks, threads, fine_shared_memory>>>(d_data.x,
                                                       d_data.y,
                                                       d_data.size,
                                                       d_means.x,
                                                       d_means.y,
                                                       d_sums.x,
                                                       d_sums.y,
                                                       k,
                                                       d_counts,
						       d_clusterNo);
  hipDeviceSynchronize();

  coarse_reduce<<<1, k * blocks, coarse_shared_memory>>>(d_means.x,
                                                         d_means.y,
                                                         d_sums.x,
                                                         d_sums.y,
                                                         k,
                                                         d_counts);

  hipDeviceSynchronize();
}

  hipMemcpy(h_clusterNo, d_clusterNo, N * sizeof(int), hipMemcpyDeviceToHost);

  travdirtime = stop_timer(&t);
  print_timer(travdirtime); 


/*
  for(int i=0; i < N; i++)
  	  std::cout << h_x[i] << "," << h_y[i] << "," << h_clusterNo[i] << std::endl;
*/

/*

  for (size_t iteration = 0; iteration < number_of_iterations; ++iteration) {
    hipMemset(d_counts, 0, k * sizeof(int));
    d_sums.clear();

    assign_clusters<<<blocks, threads>>>(d_data.x,
                                         d_data.y,
                                         d_data.size,
                                         d_means.x,
                                         d_means.y,
                                         d_sums.x,
                                         d_sums.y,
                                         k,
                                         d_counts,
					 d_clusterNo);
    hipDeviceSynchronize();

    compute_new_means<<<1, k>>>(d_means.x,
                                d_means.y,
                                d_sums.x,
                                d_sums.y,
                                d_counts);
    hipDeviceSynchronize();
  }

  hipMemcpy(h_clusterNo, d_clusterNo, N * sizeof(int), hipMemcpyDeviceToHost);

  for(int i=0; i < N; i++)
  	  std::cout << h_x[i] << "," << h_y[i] << "," << h_clusterNo[i] << std::endl;
*/

}
