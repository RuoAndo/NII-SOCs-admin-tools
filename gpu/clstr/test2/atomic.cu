#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <chrono>
#include <random>
#include <vector>

#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>
#include <random>

#include "timer.h"

#include "csv.hpp"
using namespace std;

// A small data structure to do RAII for a dataset of 2-dimensional points.
struct Data {
  explicit Data(int size) : size(size), bytes(size * sizeof(float)) {
    hipMalloc(&x, bytes);
    hipMalloc(&y, bytes);
  }

  Data(int size, std::vector<float>& h_x, std::vector<float>& h_y)
  : size(size), bytes(size * sizeof(float)) {
    hipMalloc(&x, bytes);
    hipMalloc(&y, bytes);
    hipMemcpy(x, h_x.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(y, h_y.data(), bytes, hipMemcpyHostToDevice);
  }

  ~Data() {
    hipFree(x);
    hipFree(y);
  }

  void clear() {
    hipMemset(x, 0, bytes);
    hipMemset(y, 0, bytes);
  }

  float* x{nullptr};
  float* y{nullptr};
  int size{0};
  int bytes{0};
};

__device__ float
squared_l2_distance(float x_1, float y_1, float x_2, float y_2) {
  return (x_1 - x_2) * (x_1 - x_2) + (y_1 - y_2) * (y_1 - y_2);
}

// In the assignment step, each point (thread) computes its distance to each
// cluster centroid and adds its x and y values to the sum of its closest
// centroid, as well as incrementing that centroid's count of assigned points.
__global__ void assign_clusters(const float* __restrict__ data_x,
                                const float* __restrict__ data_y,
                                int data_size,
                                const float* __restrict__ means_x,
                                const float* __restrict__ means_y,
                                float* __restrict__ new_sums_x,
                                float* __restrict__ new_sums_y,
                                int k,
                                int* __restrict__ counts,
				int* clusterNo) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= data_size) return;

  // printf("data_size %d \n", data_size);

  // Make global loads once.
  const float x = data_x[index];
  const float y = data_y[index];

  float best_distance = FLT_MAX;
  int best_cluster = 0;
  for (int cluster = 0; cluster < k; ++cluster) {
    const float distance =
        squared_l2_distance(x, y, means_x[cluster], means_y[cluster]);
    if (distance < best_distance) {
      best_distance = distance;
      best_cluster = cluster;
      // printf("best_cluster %d \n", cluster);
      clusterNo[index] = cluster;
    }
  }

  // Slow but simple.
  atomicAdd(&new_sums_x[best_cluster], x);
  atomicAdd(&new_sums_y[best_cluster], y);
  atomicAdd(&counts[best_cluster], 1);
}

// Each thread is one cluster, which just recomputes its coordinates as the mean
// of all points assigned to it.
__global__ void compute_new_means(float* __restrict__ means_x,
                                  float* __restrict__ means_y,
                                  const float* __restrict__ new_sum_x,
                                  const float* __restrict__ new_sum_y,
                                  const int* __restrict__ counts) {
  const int cluster = threadIdx.x;
  // Threshold count to turn 0/0 into 0/1.
  const int count = max(1, counts[cluster]);
  means_x[cluster] = new_sum_x[cluster] / count;
  means_y[cluster] = new_sum_y[cluster] / count;
}

int main(int argc, const char* argv[]) {

  unsigned int t, travdirtime;

  std::vector<float> h_x;
  std::vector<float> h_y;

  // Load x and y into host vectors ... (omitted)

  if (argc != 5)
  {
	printf("./a.out file data N(lines) I(number_of_iterations) K(clusters) \n");
	printf("./a.out file data 1000 1000 3 \n");
	exit(1);
  }

  int N = atoi(argv[2]);

  /*
  int k = 3;
  int number_of_iterations = 1000;
  */

  int k = atoi(argv[4]);
  int number_of_iterations = atoi(argv[3]);

  const string csv_file = std::string(argv[1]); 
  vector<vector<string>> data2; 

  Csv objCsv(csv_file);
  if (!objCsv.getCsv(data2)) {
     cout << "read ERROR" << endl;
     return 1;
  }

  // for (int row = 0; row < data2.size(); row++) {
  for (int row = 0; row < N; row++) {
      vector<string> rec = data2[row]; 
      h_x.push_back(std::stof(rec[0]));
      h_y.push_back(std::stof(rec[1]));
  }

  start_timer(&t); 

  const size_t number_of_elements = h_x.size();
  Data d_data(number_of_elements, h_x, h_y);

  // Random shuffle the data and pick the first
  // k points (i.e. k random points).
  std::random_device seed;
  std::mt19937 rng(seed());
  std::shuffle(h_x.begin(), h_x.end(), rng);
  std::shuffle(h_y.begin(), h_y.end(), rng);
  Data d_means(k, h_x, h_y);
  Data d_sums(k);

  int* d_counts;
  hipMalloc(&d_counts, k * sizeof(int));
  hipMemset(d_counts, 0, k * sizeof(int));

  int* h_counts;
  h_counts = (int *)malloc(k * sizeof(int));

  int* h_clusterNo;
  h_clusterNo = (int *)malloc(N * sizeof(int)); 

  int* d_clusterNo;
  hipMalloc(&d_clusterNo, N * sizeof(int));
  hipMemset(d_clusterNo, 0, N * sizeof(int));

  const int threads = N;
  const int blocks = (number_of_elements + threads - 1) / threads;

  for (size_t iteration = 0; iteration < number_of_iterations; ++iteration) {
    hipMemset(d_counts, 0, k * sizeof(int));
    d_sums.clear();

    assign_clusters<<<blocks, threads>>>(d_data.x,
                                         d_data.y,
                                         d_data.size,
                                         d_means.x,
                                         d_means.y,
                                         d_sums.x,
                                         d_sums.y,
                                         k,
                                         d_counts,
					 d_clusterNo);
    hipDeviceSynchronize();

    compute_new_means<<<1, k>>>(d_means.x,
                                d_means.y,
                                d_sums.x,
                                d_sums.y,
                                d_counts);
    hipDeviceSynchronize();
  }

  hipMemcpy(h_clusterNo, d_clusterNo, N * sizeof(int), hipMemcpyDeviceToHost);

  travdirtime = stop_timer(&t);
  print_timer(travdirtime); 

  /*
  for(int i=0; i < N; i++)
  	  std::cout << h_x[i] << "," << h_y[i] << "," << h_clusterNo[i] << std::endl;
  */
}
