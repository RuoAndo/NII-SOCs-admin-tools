#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <chrono>
#include <random>
#include <vector>

#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>
#include <random>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "csv.hpp"
using namespace std;

// A small data structure to do RAII for a dataset of 2-dimensional points.
struct Data {
  explicit Data(int size) : size(size), bytes(size * sizeof(float)) {
    hipMalloc(&x, bytes);
    hipMalloc(&y, bytes);
  }

  Data(int size, std::vector<float>& h_x, std::vector<float>& h_y)
  : size(size), bytes(size * sizeof(float)) {
    hipMalloc(&x, bytes);
    hipMalloc(&y, bytes);
    hipMemcpy(x, h_x.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(y, h_y.data(), bytes, hipMemcpyHostToDevice);
  }

  ~Data() {
    hipFree(x);
    hipFree(y);
  }

  void clear() {
    hipMemset(x, 0, bytes);
    hipMemset(y, 0, bytes);
  }

  float* x{nullptr};
  float* y{nullptr};
  int size{0};
  int bytes{0};
};

__device__ float
squared_l2_distance(float x_1, float y_1, float x_2, float y_2) {
  return (x_1 - x_2) * (x_1 - x_2) + (y_1 - y_2) * (y_1 - y_2);
}

// In the assignment step, each point (thread) computes its distance to each
// cluster centroid and adds its x and y values to the sum of its closest
// centroid, as well as incrementing that centroid's count of assigned points.
__global__ void assign_clusters(const thrust::device_ptr<float> data_x,
                                const thrust::device_ptr<float> data_y,
                                int data_size,
                                const thrust::device_ptr<float> means_x,
                                const thrust::device_ptr<float> means_y,
                                thrust::device_ptr<float> new_sums_x,
                                thrust::device_ptr<float> new_sums_y,
                                int k,
                                thrust::device_ptr<int> counts,
				int* clusterNo) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= data_size) return;

  // int* clusterNo;

  // Make global loads once.
  const float x = data_x[index];
  const float y = data_y[index];

  float best_distance = FLT_MAX;
  int best_cluster = 0;
  for (int cluster = 0; cluster < k; ++cluster) {
    const float distance =
        squared_l2_distance(x, y, means_x[cluster], means_y[cluster]);
    if (distance < best_distance) {
      best_distance = distance;
      best_cluster = cluster;
      clusterNo[index] = best_cluster;
    }
  }

  atomicAdd(thrust::raw_pointer_cast(new_sums_x + best_cluster), x);
  atomicAdd(thrust::raw_pointer_cast(new_sums_y + best_cluster), y);
  atomicAdd(thrust::raw_pointer_cast(counts + best_cluster), 1);
}

// Each thread is one cluster, which just recomputes its coordinates as the mean
// of all points assigned to it.
__global__ void compute_new_means(thrust::device_ptr<float> means_x,
                                  thrust::device_ptr<float> means_y,
                                  const thrust::device_ptr<float> new_sum_x,
                                  const thrust::device_ptr<float> new_sum_y,
                                  const thrust::device_ptr<int> counts) {
  const int cluster = threadIdx.x;
  const int count = max(1, counts[cluster]);
  means_x[cluster] = new_sum_x[cluster] / count;
  means_y[cluster] = new_sum_y[cluster] / count;
}

int main(int argc, const char* argv[]) {
  // std::vector<float> h_x;
  // std::vector<float> h_y;

  // Load x and y into host vectors ... (omitted)

  int SHIFT = atoi(argv[1]);
  int k = 4;
  int number_of_iterations = 1000;

  int N = 1024 << SHIFT;
  cout << N << endl;
  
  std::vector<float> h_x(N);
  std::vector<float> h_y(N);
 
  std::random_device rnd;
  std::mt19937 mt(rnd());
  std::uniform_int_distribution<> rand100(0, 99);
  std::uniform_int_distribution<> rand200(100, 199);
  std::uniform_int_distribution<> rand300(200, 299);
  std::uniform_int_distribution<> rand400(300, 399);  

  for(int i = 0; i < N; i++)
  {
	if (i % 4 == 0)
		h_x[i] = rand100(mt);
	if (i % 4 == 1)
		h_x[i] = rand200(mt);
	if (i % 4 == 2)
		h_x[i] = rand300(mt);
	if (i % 4 == 3)
		h_x[i] = rand400(mt);
		
	if (i % 4 == 0)
		h_y[i] = rand100(mt);
	if (i % 4 == 1)
		h_y[i] = rand200(mt);
	if (i % 4 == 2)
		h_y[i] = rand300(mt);
	if (i % 4 == 3)
		h_y[i] = rand400(mt);
  }

  /*
  int N = atoi(argv[2]);
  int k = 3;
  int number_of_iterations = 1000;

  const string csv_file = std::string(argv[1]); 
  vector<vector<string>> data2; 

  Csv objCsv(csv_file);
  if (!objCsv.getCsv(data2)) {
     cout << "read ERROR" << endl;
     return 1;
  }

  for (int row = 0; row < 1024; row++) {
      vector<string> rec = data2[row]; 
      h_x.push_back(std::stof(rec[0]));
      h_y.push_back(std::stof(rec[1]));
  }
  */

  const size_t number_of_elements = h_x.size();

  int* h_clusterNo;
  h_clusterNo = (int *)malloc(N * sizeof(int)); 

  int* d_clusterNo;
  hipMalloc(&d_clusterNo, N * sizeof(int));
  hipMemset(d_clusterNo, 0, N * sizeof(int));

  thrust::device_vector<float> d_x = h_x;
  thrust::device_vector<float> d_y = h_y;

  std::mt19937 rng(std::random_device{}());
  std::shuffle(h_x.begin(), h_x.end(), rng);
  std::shuffle(h_y.begin(), h_y.end(), rng);
  thrust::device_vector<float> d_mean_x(h_x.begin(), h_x.begin() + k);
  thrust::device_vector<float> d_mean_y(h_y.begin(), h_y.begin() + k);

  thrust::device_vector<float> d_sums_x(k);
  thrust::device_vector<float> d_sums_y(k);
  thrust::device_vector<int> d_counts(k, 0);

  const int threads = 1024;
  const int blocks = (number_of_elements + threads - 1) / threads;

  for (size_t iteration = 0; iteration < number_of_iterations; ++iteration) {
    thrust::fill(d_sums_x.begin(), d_sums_x.end(), 0);
    thrust::fill(d_sums_y.begin(), d_sums_y.end(), 0);
    thrust::fill(d_counts.begin(), d_counts.end(), 0);

    assign_clusters<<<blocks, threads>>>(d_x.data(),
                                         d_y.data(),
                                         number_of_elements,
                                         d_mean_x.data(),
                                         d_mean_y.data(),
                                         d_sums_x.data(),
                                         d_sums_y.data(),
                                         k,
                                         d_counts.data(),
					 d_clusterNo);
    hipDeviceSynchronize();

    compute_new_means<<<1, k>>>(d_mean_x.data(),
                                d_mean_y.data(),
                                d_sums_x.data(),
                                d_sums_y.data(),
                                d_counts.data());
    hipDeviceSynchronize();
  }

  hipMemcpy(h_clusterNo, d_clusterNo, N * sizeof(int), hipMemcpyDeviceToHost);

  std::remove("tmp");
  ofstream outputfile("tmp"); 

  for(int i=0; i < N; i++)
  {
	outputfile << h_x[i] << "," << h_y[i] << "," << h_clusterNo[i] << std::endl;
  	// std::cout << h_x[i] << "," << h_y[i] << "," << h_clusterNo[i] << std::endl;
  }

/*
  hipMemcpy(h_clusterNo, d_clusterNo, N * sizeof(int), hipMemcpyDeviceToHost);

  for(int i=0; i < N; i++)
  	  std::cout << h_x[i] << "," << h_y[i] << "," << h_clusterNo[i] << std::endl;
*/

}
