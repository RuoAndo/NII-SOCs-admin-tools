#include <hipblas.h>

#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/sequence.h>

#include <stdio.h>
#include <iostream>

/*
#include "Utilities.cuh"
#include "TimingGPU.cuh"
*/

#include "csv.hpp"
using namespace std;

int main( int argc, char* argv[] )
{
  int N = atoi(argv[2]);
  
  thrust::host_vector<float> h_vec_1(N);
  thrust::host_vector<float> h_vec_2(N);
  thrust::host_vector<float> h_vec_3(N);
  thrust::host_vector<float> h_vec_4(N);
  thrust::host_vector<float> h_vec_5(N);
  thrust::host_vector<float> h_vec_6(N);   

  const string csv_file = std::string(argv[1]); 
  vector<vector<string>> data; 

  Csv objCsv(csv_file);
  if (!objCsv.getCsv(data)) {
     cout << "read ERROR" << endl;
     return 1;
  }

  for (int row = 0; row < data.size(); row++) {
      vector<string> rec = data[row]; 

      h_vec_1[row] = std::stof(rec[0]);
      h_vec_2[row] = std::stof(rec[1]);
      h_vec_3[row] = std::stof(rec[2]);
      h_vec_4[row] = std::stof(rec[3]);
      h_vec_5[row] = std::stof(rec[4]);
      h_vec_6[row] = std::stof(rec[5]);
  }

  thrust::host_vector<float> d_vec_1(N);
  thrust::host_vector<float> d_vec_2(N);
  thrust::host_vector<float> d_vec_3(N);
  thrust::host_vector<float> d_vec_4(N);
  thrust::host_vector<float> d_vec_5(N);
  thrust::host_vector<float> d_vec_6(N);
      
  thrust::copy(h_vec_1.begin(), h_vec_1.end(), d_vec_1.begin());
  thrust::copy(h_vec_2.begin(), h_vec_2.end(), d_vec_2.begin());
  thrust::copy(h_vec_3.begin(), h_vec_3.end(), d_vec_3.begin());
  thrust::copy(h_vec_4.begin(), h_vec_4.end(), d_vec_4.begin());
  thrust::copy(h_vec_5.begin(), h_vec_5.end(), d_vec_5.begin());
  thrust::copy(h_vec_6.begin(), h_vec_6.end(), d_vec_6.begin());   

  float sum1 = thrust::reduce(d_vec_1.begin(), d_vec_1.end());
  float sum2 = thrust::reduce(d_vec_2.begin(), d_vec_2.end());
  float sum3 = thrust::reduce(d_vec_3.begin(), d_vec_3.end());
  float sum4 = thrust::reduce(d_vec_4.begin(), d_vec_4.end());
  float sum5 = thrust::reduce(d_vec_5.begin(), d_vec_5.end());
  float sum6 = thrust::reduce(d_vec_6.begin(), d_vec_6.end());

  std::cout << sum1 << std::endl;
  std::cout << sum2 << std::endl;
  std::cout << sum3 << std::endl;
  std::cout << sum4 << std::endl;
  std::cout << sum5 << std::endl;
  std::cout << sum6 << std::endl;

  return 0;
}
