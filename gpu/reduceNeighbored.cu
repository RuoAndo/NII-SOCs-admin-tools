#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

// Recursive Implementation of Interleaved Pair Approach
int recursiveReduce(int *data, int const size)
{
    // terminate check
    if (size == 1) return data[0];

    // renew the stride
    int const stride = size / 2;

    // in-place reduction
    for (int i = 0; i < stride; i++)
    {
        data[i] += data[i + stride];
    }

    // call recursively
    return recursiveReduce(data, stride);
}

__global__ void reduceNeighbored (int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    if (idx >= n) return;

    // in-place reduction in global memory
    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        if ((tid % (2 * stride)) == 0)
        {
            idata[tid] += idata[tid + stride];
        }

        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s starting reduction at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    bool bResult = false;

    // initialization
    int size = 1 << 24; // total number of elements to reduce
    printf("    with array size %d  ", size);

    // execution configuration
    int blocksize = 1024;   // initial block size

    if(argc > 1)
    {
        blocksize = atoi(argv[1]);   // block size from command line argument
    }

    dim3 block (blocksize, 1);
    dim3 grid  ((size + block.x - 1) / block.x, 1);
    printf("grid %d block %d\n", grid.x, block.x);

    // allocate host memory
    size_t bytes = size * sizeof(int);
    int *h_idata = (int *) malloc(bytes);
    int *h_odata = (int *) malloc(grid.x * sizeof(int));
    int *tmp     = (int *) malloc(bytes);

    // initialize the array
    for (int i = 0; i < size; i++)
    {
        // mask off high 2 bytes to force max number to 255
        h_idata[i] = (int)( rand() & 0xFF );
    }

    memcpy (tmp, h_idata, bytes);

    double iStart, iElaps;
    int gpu_sum = 0;

    // allocate device memory
    int *d_idata = NULL;
    int *d_odata = NULL;
    CHECK(hipMalloc((void **) &d_idata, bytes));
    CHECK(hipMalloc((void **) &d_odata, grid.x * sizeof(int)));

    // cpu reduction
    iStart = seconds();
    int cpu_sum = recursiveReduce (tmp, size);
    iElaps = seconds() - iStart;
    printf("cpu reduce      elapsed %f sec cpu_sum: %d\n", iElaps, cpu_sum);

    // kernel 1: reduceNeighbored
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = seconds();
    reduceNeighbored<<<grid, block>>>(d_idata, d_odata, size);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];

    printf("gpu Neighbored  elapsed %f sec gpu_sum: %d <<<grid %d block "
           "%d>>>\n", iElaps, gpu_sum, grid.x, block.x);

    // free host memory
    free(h_idata);
    free(h_odata);

    // free device memory
    CHECK(hipFree(d_idata));
    CHECK(hipFree(d_odata));

    // reset device
    CHECK(hipDeviceReset());

    // check the results
    bResult = (gpu_sum == cpu_sum);

    if(!bResult) printf("Test failed!\n");

    return EXIT_SUCCESS;
}
