#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

double cpuSecond()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

#define CHECK(call)                                                  \
{                                                                    \
    const hipError_t error = call;                                  \
    if (error != hipSuccess)                                        \
    {                                                                \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                \
        printf("code:%d, reason: %s\n", error,                       \
                hipGetErrorString(error));                          \
        exit(1);                                                     \
    }                                                                \
}

void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }

    if (match) printf("Arrays match.\n\n");

    return;
}

void initialData(float *ip, int size)
{
    time_t t;
    srand((unsigned) time(&t));

    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)( rand() & 0xFF ) / 10.0f;
    }

    return;
}

void sumOnHost(float *A, float *B, float *C, const int N)
{
    for (int idx = 0; idx < N; idx++)
    {
        C[idx] = A[idx] + B[idx];
    }
}
__global__ void sumOnGPU(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) C[i] = A[i] + B[i];
}

int main(int argc, char **argv)
{

    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    // printf("Device name %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    int nElement = 1 << atoi(argv[1]);
    printf("Vector size %d\n", nElement);

    // malloc host memory
    size_t nBytes = nElement * sizeof(float);

    float *hostA, *hostB, *hostRef, *gpuRef;
    hostA     = (float *)malloc(nBytes);
    hostB     = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef  = (float *)malloc(nBytes);

    double iStart, iElaps_device, iElaps_host, iRatio;

    iStart = cpuSecond();
    initialData(hostA, nElement);
    initialData(hostB, nElement);
    iElaps_host = cpuSecond() - iStart;
    
    memset(hostRef, 0, nBytes);
    memset(gpuRef,  0, nBytes);

    iStart = cpuSecond();
    sumOnHost(hostA, hostB, hostRef, nElement);
    iElaps_host = cpuSecond() - iStart;
    printf("sumOnHost : Time elapsed %f sec\n", iElaps_host);
    
    float *device_A, *device_B, *device_C;
    CHECK(hipMalloc((float**)&device_A, nBytes));
    CHECK(hipMalloc((float**)&device_B, nBytes));
    CHECK(hipMalloc((float**)&device_C, nBytes));

    CHECK(hipMemcpy(device_A, hostA, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(device_B, hostB, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(device_C, gpuRef, nBytes, hipMemcpyHostToDevice));

    // invoke kernel at host side
    //int iLen = 512;
    int iLen = atoi(argv[2]);
    dim3 block (iLen);
    dim3 grid  ((nElement + block.x - 1) / block.x);

    iStart = cpuSecond();
    sumOnGPU<<<grid, block>>>(device_A, device_B, device_C, nElement);
    CHECK(hipDeviceSynchronize());
    iElaps_device = cpuSecond() - iStart;
    printf("sumOnGPU %d, %d : Time elapsed %f sec\n", grid.x, block.x, iElaps_device);

    iRatio = iElaps_host / iElaps_device;
    printf("ratio %f \n", iRatio); 

    CHECK(hipGetLastError()) ;

    CHECK(hipMemcpy(gpuRef, device_C, nBytes, hipMemcpyDeviceToHost));

    // checkResult(hostRef, gpuRef, nElement);

    CHECK(hipFree(device_A));
    CHECK(hipFree(device_B));
    CHECK(hipFree(device_C));

    free(hostA);
    free(hostB);
    free(hostRef);
    free(gpuRef);

    return(0);
}
