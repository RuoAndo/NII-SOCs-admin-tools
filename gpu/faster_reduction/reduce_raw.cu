#include "hip/hip_runtime.h"
#if __linux__ && defined(__INTEL_COMPILER)
#define __sync_fetch_and_add(ptr,addend) _InterlockedExchangeAdd(const_cast<void*>(reinterpret_cast<volatile void*>(ptr)), addend)
#endif
#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>
#include <map>

#include <sys/socket.h>
#include <netinet/in.h>
#include <arpa/inet.h>

#include "csv.hpp"
// typedef std::basic_string<char,std::char_traits<char>,tbb::tbb_allocator<char> > MyString;

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include "util.h"

using namespace std;
// static MyString* Data;

void sumArraysOnHost(float *A, float *B, float *C, const int N)
{
    for (int idx = 0; idx < N; idx++)
    {
        C[idx] = A[idx] + B[idx];
    }
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) C[i] = A[i] + B[i];
}

int main( int argc, char* argv[] ) {

  int N = atoi(argv[2]);  
  std::string timestamp;

  int counter = 0;
  std::map<unsigned long long, int> mp;  

  thrust::host_vector<unsigned long long> h_vec_1(N);
  thrust::host_vector<unsigned long long> h_vec_2(N);   

  thrust::device_vector<unsigned long long> d_vec_1(N);
  thrust::device_vector<unsigned long long> d_vec_2(N);

  size_t nBytes = N * sizeof(unsigned long long);

  /*
  unsigned long long *h_A, *h_B, *hostRef, *gpuRef;
  h_A     = (unsigned long long *)malloc(nBytes);
  h_B     = (unsigned long long *)malloc(nBytes);
  hostRef = (unsigned long long *)malloc(nBytes);
  gpuRef  = (unsigned long long *)malloc(nBytes);
  */

  // Data = new MyString[N];
  const string csv_file = std::string(argv[1]); 
  vector<vector<string>> data; 

  try {
      	  Csv objCsv(csv_file);
	  if (!objCsv.getCsv(data)) {
	    cout << "read ERROR" << endl;
	    return 1;
	    }

	  for (int row = 0; row < data.size(); row++) {
	    vector<string> rec = data[row]; 

	    timestamp = rec[0];
	    std::string pair = rec[1];

	    h_vec_1[row] = stoull(rec[0].c_str());

	    /*
	    h_vec_1[row] = stoull(string(pair.c_str()));
	    h_vec_2[row] = 1; // atol(pair.c_str());
	    */

	    // h_A[row] = rec[0];

	  }

      	  thrust::copy(h_vec_1.begin(), h_vec_1.end(), d_vec_1.begin());
	  thrust::copy(h_vec_1.begin(), h_vec_1.end(), d_vec_2.begin());

	  for (int i = 0; i < N; i++) {
	      std::cout << h_vec_1[i] << std::endl;

	      unsigned long long f = h_vec_1[i];

	      auto ff = [=]  __device__ (unsigned long long x) {return x - f ;};
              thrust::transform(d_vec_1.begin(),d_vec_1.end(),d_vec_2.begin(),ff);

	      counter = 0;
	      for(int j = 0; j < N; j++) {
	      	      if (d_vec_2[j]==0) {
		      	     // std::cout << d_vec_1[i] << "," << f << "," << d_vec_2[j] << std::endl;
			     counter = counter + 1;
			     }
	      }
	      
	      // std::cout << d_vec_1[i] << "," << counter << std::endl;
	      mp[d_vec_1[i]] = counter; 
	  }

	  for(auto x : mp) {
          	   std::cout << x.first << "," << x.second << "\n";
	  }
	  
   }
   catch (...) {
	  cout << "EXCEPTION!" << endl;
	  return 1;
   }
	
   // delete[] Data;       
   return 0;
}
