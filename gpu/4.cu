#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include "util.h"

int main(void)
{
  // generate 32M random numbers serially
  // thrust::host_vector<int> h_vec(1024 << 20);
  thrust::host_vector<int> h_vec(1024 << 4);
  std::generate(h_vec.begin(), h_vec.end(), rand);

  static int i;
  static int size;
  
  size = h_vec.size();

  std::cout << "size:" << size << std::endl;

  // transfer data to the device
  thrust::device_vector<int> d_vec = h_vec;

  thrust::host_vector<int> h_vec_2(1024 << 4);

  for (i = 0; i < h_vec_2.size(); i++)
      h_vec_2[i] = i;

  thrust::device_vector<int> d_vec_2 = h_vec_2;

      /*
      const int N = 6;
      int    keys[N] = {  1,   4,   2,   8,   5,   7};
      char values[N] = {'a', 'b', 'c', 'd', 'e', 'f'};

      thrust::sort_by_key(keys, keys + N, values);
      */

      // thrust::device_vector<int> vec(13);
      
  thrust::device_vector<int>::iterator iter = d_vec_2.begin();

  thrust::sort_by_key(d_vec_2.begin(), d_vec_2.end(), d_vec.begin());
  				       		      
  // sort data on the device (846M keys per second on GeForce GTX 480)
  // thrust::sort(d_vec.begin(), d_vec.end());

  // for(auto itr=d_vec.begin(); itr!=d_vec.end(); itr++)
  //	   std::cout << *itr << std::endl;

  for(int i = 0; i < d_vec.size(); i++)
  	   std::cout << d_vec[i] << "," << d_vec_2[i] << std::endl;

  hipEvent_t     start, stop;
  HANDLE_ERROR( hipEventCreate( &start ) );
  HANDLE_ERROR( hipEventCreate( &stop ) );
  HANDLE_ERROR( hipEventRecord( start, 0 ) );

  return 0;
}
