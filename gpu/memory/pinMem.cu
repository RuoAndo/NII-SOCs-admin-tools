#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char **argv)
{
    int dev = 0;
    CHECK(hipSetDevice(dev));

    unsigned int isize = 1 << 22;
    unsigned int nbytes = isize * sizeof(float);

    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));

    if (!deviceProp.canMapHostMemory)
    {
        printf("Device %d does not support mapping CPU host memory!\n", dev);
        CHECK(hipDeviceReset());
        exit(EXIT_SUCCESS);
    }

    printf("%s starting at ", argv[0]);
    printf("device %d: %s memory size %d nbyte %5.2fMB canMap %d\n", dev,
           deviceProp.name, isize, nbytes / (1024.0f * 1024.0f),
           deviceProp.canMapHostMemory);

    float *h_a;
    CHECK(hipHostMalloc ((float **)&h_a, nbytes));

    float *d_a;
    CHECK(hipMalloc((float **)&d_a, nbytes));

    memset(h_a, 0, nbytes);

    for (int i = 0; i < isize; i++) h_a[i] = 100.10f;

    CHECK(hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(h_a, d_a, nbytes, hipMemcpyDeviceToHost));

    CHECK(hipFree(d_a));
    CHECK(hipHostFree(h_a));

    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
