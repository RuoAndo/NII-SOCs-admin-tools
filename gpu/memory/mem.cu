#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char **argv)
{
    int dev = 0;
    CHECK(hipSetDevice(dev));

    unsigned int isize = 1 << 22;
    unsigned int nbytes = isize * sizeof(float);

    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s starting at ", argv[0]);
    printf("device %d: %s memory size %d nbyte %5.2fMB\n", dev,
           deviceProp.name, isize, nbytes / (1024.0f * 1024.0f));

    float *h_a = (float *)malloc(nbytes);
    float *d_a;
    CHECK(hipMalloc((float **)&d_a, nbytes));

    for(unsigned int i = 0; i < isize; i++) h_a[i] = 0.5f;

    CHECK(hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(h_a, d_a, nbytes, hipMemcpyDeviceToHost));

    CHECK(hipFree(d_a));
    free(h_a);
    
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
