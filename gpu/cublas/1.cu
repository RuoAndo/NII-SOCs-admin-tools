#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#include <time.h>

__global__ void initIdentityGPU(int **devMatrix, int numR, int numC) {
    int x = blockDim.x*blockIdx.x + threadIdx.x;
    int y = blockDim.y*blockIdx.y + threadIdx.y;
    if(y < numR && x < numC) {
          if(x == y)
              devMatrix[y][x] = 1;
          else
              devMatrix[y][x] = 0;
    }
}

int M = 10000;
int N = 10000;

void generate_random_vector(int N, float **outX)
{
    int i;
    double rMax = (double)RAND_MAX;
    float *X = (float *)malloc(sizeof(float) * N);

    for (i = 0; i < N; i++)
    {
        int r = rand();
        double dr = (double)r;
        X[i] = (dr / rMax) * 100.0;
    }

    *outX = X;
}

void generate_random_dense_matrix(int M, int N, float **outA)
{
    int i, j;
    double rMax = (double)RAND_MAX;
    float *A = (float *)malloc(sizeof(float) * M * N);

    // For each column
    for (j = 0; j < N; j++)
    {
        // For each row
        for (i = 0; i < M; i++)
        {
            double dr = (double)rand();
            A[j * M + i] = (dr / rMax) * 100.0;
        }
    }

    *outA = A;
}

void generate_identity_matrix(int M, int N, float **outA)
{
    int i, j;
    // double rMax = (double)RAND_MAX;
    float *A = (float *)malloc(sizeof(float) * M * N);

    // For each column
    for (j = 0; j < N; j++)
    {
        // For each row
        for (i = 0; i < M; i++)
        {
	    if(i == j)
	    	 A[N,M] = 1;
        }
    }

    *outA = A;
}

int main(int argc, char **argv)
{
    int i;
    float *A, *dA;
    float *I, *dI;
    float *X, *dX;
    float *Y, *dY;
    float beta;
    float alpha;
    hipblasHandle_t handle = 0;

    struct timespec startTime, endTime, sleepTime;

    alpha = 3.0f;
    beta = 16.0f;

    srand(9384);
    generate_random_dense_matrix(M, N, &A);
    generate_random_vector(N, &X);
    generate_random_vector(M, &Y);
    generate_identity_matrix(M, N, &I);
    
    CHECK_CUBLAS(hipblasCreate(&handle));

    CHECK(hipMalloc((void **)&dA, sizeof(float) * M * N));
    CHECK(hipMalloc((void **)&dX, sizeof(float) * N));
    CHECK(hipMalloc((void **)&dY, sizeof(float) * M));

    CHECK_CUBLAS(hipblasSetVector(N, sizeof(float), X, 1, dX, 1));
    CHECK_CUBLAS(hipblasSetVector(M, sizeof(float), Y, 1, dY, 1));
    CHECK_CUBLAS(hipblasSetMatrix(M, N, sizeof(float), A, M, dA, M));

    clock_gettime(CLOCK_REALTIME, &startTime);
    sleepTime.tv_sec = 0;
    sleepTime.tv_nsec = 123;

    CHECK_CUBLAS(hipblasSgemv(handle, HIPBLAS_OP_N, M, N, &alpha, dA, M, dX, 1,
                             &beta, dY, 1));

    clock_gettime(CLOCK_REALTIME, &endTime);

    printf("elapsed time:");

    if (endTime.tv_nsec < startTime.tv_nsec) {
    	      printf("%ld.%09ld", endTime.tv_sec - startTime.tv_sec - 1 ,endTime.tv_nsec + 1000000000 - startTime.tv_nsec);
    } else {
	      printf("%ld.%09ld", endTime.tv_sec - startTime.tv_sec, endTime.tv_nsec - startTime.tv_nsec);
    }
    printf(" sec\n");

    // Retrieve the output vector from the device
    CHECK_CUBLAS(hipblasGetVector(M, sizeof(float), dY, 1, Y, 1));

    /*
    for (i = 0; i < 10; i++)
    {
        printf("%2.2f\n", Y[i]);
    }
    */

    printf("...\n");

    free(A);
    free(X);
    free(Y);

    CHECK(hipFree(dA));
    CHECK(hipFree(dY));
    CHECK_CUBLAS(hipblasDestroy(handle));

    return 0;
}
