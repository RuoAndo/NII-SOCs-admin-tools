#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <time.h>

// #define RAND_MAX 100

void generate_random_vector(int N, float **outX)
{
    int i;
    double rMax = (double)RAND_MAX;
    float *X = (float *)malloc(sizeof(float) * N);
    
    for (i = 0; i < N; i++)
    {
        // int r = rand();
        // double dr = (double)r;
	X[i] = (float)(rand() & 0xFF) / 100.0f; 
        // X[i] = (dr / rMax) * 100.0;
    }

    *outX = X;
}

int main(int argc, char *argv[])
{
   // timeval st,et;

   float *hstA,*hstB,*hstC;
   float *devA,*devB,*devC;

   float alpha = 1.0f;
   float beta = 0.0f;

   if (argc <= 1)
   {
	printf("usage: ./a.out MATRIX_SIZE \n");
   }
   
   int num = atoi(argv[1]);

   int n2 = num*num;
   size_t memSz = n2 * sizeof(float);

   hstA=(float*)malloc(memSz);
   hstB=(float*)malloc(memSz);
   hstC=(float*)malloc(memSz);

   /* insertion */

   srand((unsigned int)time(NULL));

   struct timespec startTime, endTime, sleepTime;

   printf("matrix size %d : %d \n", num, num);

   clock_gettime(CLOCK_REALTIME, &startTime);
   sleepTime.tv_sec = 0;
   sleepTime.tv_nsec = 123;

   generate_random_vector(memSz, &hstA);
   generate_random_vector(memSz, &hstB);
   generate_random_vector(memSz, &hstC);

   clock_gettime(CLOCK_REALTIME, &endTime);

   // printf("開始時刻　 = %10ld.%09ld\n", startTime.tv_sec, startTime.tv_nsec);
   // printf("終了時刻　 = %10ld.%09ld\n", endTime.tv_sec, endTime.tv_nsec);
   printf("generate_random_vector: ");
   if (endTime.tv_nsec < startTime.tv_nsec) {
     printf("%ld.%09ld", endTime.tv_sec - startTime.tv_sec - 1,endTime.tv_nsec + 1000000000 - startTime.tv_nsec);
   } else {
     printf("%ld.%09ld", endTime.tv_sec - startTime.tv_sec ,endTime.tv_nsec - startTime.tv_nsec);
   }
   printf(" sec \n");
					    
   for(int i = 0; i < 10; i++)
   	     printf("%lf ", hstA[i]);
   printf("\n");

   for(int i = 0; i < 10; i++)
   	     printf("%lf ", hstB[i]);    
   printf("\n");

   for(int i = 0; i < 10; i++)
   	     printf("%lf ", hstC[i]);    
   printf("\n");
 
   // gettimeofday(&st,NULL);

   clock_gettime(CLOCK_REALTIME, &startTime);
   sleepTime.tv_sec = 0;
   sleepTime.tv_nsec = 123;

   hipMalloc((void **)&devA,memSz);
   hipMalloc((void **)&devB,memSz);
   hipMalloc((void **)&devC,memSz);

   hipblasSetVector(n2, sizeof(float), hstA, 1, devA, 1);
   hipblasSetVector(n2, sizeof(float), hstB, 1, devB, 1);

   clock_gettime(CLOCK_REALTIME, &endTime);

   printf("Malloc and SetVector: ");
   if (endTime.tv_nsec < startTime.tv_nsec) {
     printf("%ld.%09ld", endTime.tv_sec - startTime.tv_sec - 1,endTime.tv_nsec + 1000000000 - startTime.tv_nsec);
   } else {
     printf("%ld.%09ld", endTime.tv_sec - startTime.tv_sec ,endTime.tv_nsec - startTime.tv_nsec);
   }
   printf(" sec \n");

   hipblasHandle_t handle; 
   hipblasCreate(&handle);

   clock_gettime(CLOCK_REALTIME, &startTime);
   sleepTime.tv_sec = 0;
   sleepTime.tv_nsec = 123;

   hipblasSgemm(    
        handle,
        HIPBLAS_OP_N, 
        HIPBLAS_OP_N, 
        num,    
        num,    
        num,    
        &alpha, 
        devA,   
        num,    
        devB,   
        num,    
        &beta,  
        devC,   
        num
   );

   clock_gettime(CLOCK_REALTIME, &endTime);

   // printf("開始時刻　 = %10ld.%09ld\n", startTime.tv_sec, startTime.tv_nsec);
   // printf("終了時刻　 = %10ld.%09ld\n", endTime.tv_sec, endTime.tv_nsec);
   printf("cublasSgemm: ");
   if (endTime.tv_nsec < startTime.tv_nsec) {
     printf("%ld.%09ld", endTime.tv_sec - startTime.tv_sec - 1,endTime.tv_nsec + 1000000000 - startTime.tv_nsec);
   } else {
     printf("%ld.%09ld", endTime.tv_sec - startTime.tv_sec ,endTime.tv_nsec - startTime.tv_nsec);
   }
   printf(" sec \n");

   int status;
   status = hipblasDestroy(handle);
   hipblasGetVector(n2, sizeof(float), devC, 1, hstC, 1);

   for(int i = 0; i < 10; i++)
   	   printf("%lf ", hstC[i]);
   printf("\n");

   hipFree(devA);
   hipFree(devB);
   hipFree(devC);

   free(hstA);
   free(hstB);
   free(hstC);

}
