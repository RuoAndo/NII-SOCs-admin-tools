#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define NSTREAM 6
#define BDIM 128

void initialData(float *ip, int size)
{
    int i;

    for(i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

void sumArraysOnHost(float *A, float *B, float *C, const int N)
{
    for (int idx = 0; idx < N; idx++)
        C[idx] = A[idx] + B[idx];
}

__global__ void sumArrays(float *A, float *B, float *C, const int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N)
        for (int i = 0; i < N; ++i)
        {
            C[idx] = A[idx] + B[idx];
        }
}

int main(int argc, char **argv)
{
    printf("> %s Starting...\n", argv[0]);

    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("> Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    if (deviceProp.major < 3 || (deviceProp.major == 3 && deviceProp.minor < 5))
    {
        if (deviceProp.concurrentKernels == 0)
        {
            printf("> GPU does not support concurrent kernel execution (SM 3.5 or higher required)\n");
            printf("> CUDA kernel runs will be serialized\n");
        }
        else
        {
            printf("> GPU does not support HyperQ\n");
            printf("> CUDA kernel runs will have limited concurrency\n");
        }
    }

    printf("> Compute Capability %d.%d hardware with %d multi-processors\n",
           deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

    int nElem = 1 << 20;
    printf("> vector size = %d\n", nElem);
    size_t nBytes = nElem * sizeof(float);

    float *h_A, *h_B, *hostRef, *gpuRef;
    CHECK(hipHostAlloc((void**)&h_A, nBytes, hipHostMallocDefault));
    CHECK(hipHostAlloc((void**)&h_B, nBytes, hipHostMallocDefault));
    CHECK(hipHostAlloc((void**)&gpuRef, nBytes, hipHostMallocDefault));
    CHECK(hipHostAlloc((void**)&hostRef, nBytes, hipHostMallocDefault));

    initialData(h_A, nElem);
    initialData(h_B, nElem);
    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    sumArraysOnHost(h_A, h_B, hostRef, nElem);

    float *d_A, *d_B, *d_C;
    CHECK(hipMalloc((float**)&d_A, nBytes));
    CHECK(hipMalloc((float**)&d_B, nBytes));
    CHECK(hipMalloc((float**)&d_C, nBytes));

    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    dim3 block (BDIM);
    dim3 grid  ((nElem + block.x - 1) / block.x);
    printf("> grid (%d, %d) block (%d, %d)\n", grid.x, grid.y, block.x, block.y);

    CHECK(hipEventRecord(start, 0));
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    float memcpy_h2d_time;
    CHECK(hipEventElapsedTime(&memcpy_h2d_time, start, stop));

    CHECK(hipEventRecord(start, 0));
    sumArrays<<<grid, block>>>(d_A, d_B, d_C, nElem);
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    float kernel_time;
    CHECK(hipEventElapsedTime(&kernel_time, start, stop));

    CHECK(hipEventRecord(start, 0));
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    float memcpy_d2h_time;
    CHECK(hipEventElapsedTime(&memcpy_d2h_time, start, stop));
    float itotal = kernel_time + memcpy_h2d_time + memcpy_d2h_time;

    printf("\n");
    printf("Measured timings (throughput):\n");
    printf(" Memcpy host to device\t: %f ms (%f GB/s)\n",
           memcpy_h2d_time, (nBytes * 1e-6) / memcpy_h2d_time);
    printf(" Memcpy device to host\t: %f ms (%f GB/s)\n",
           memcpy_d2h_time, (nBytes * 1e-6) / memcpy_d2h_time);
    printf(" Kernel\t\t\t: %f ms (%f GB/s)\n",
           kernel_time, (nBytes * 2e-6) / kernel_time);
    printf(" Total\t\t\t: %f ms (%f GB/s)\n",
           itotal, (nBytes * 2e-6) / itotal);

    int iElem = nElem / NSTREAM;
    size_t iBytes = iElem * sizeof(float);
    grid.x = (iElem + block.x - 1) / block.x;

    hipStream_t stream[NSTREAM];

    for (int i = 0; i < NSTREAM; ++i)
    {
        CHECK(hipStreamCreate(&stream[i]));
    }

    CHECK(hipEventRecord(start, 0));

    for (int i = 0; i < NSTREAM; ++i)
    {
        int ioffset = i * iElem;
        CHECK(hipMemcpyAsync(&d_A[ioffset], &h_A[ioffset], iBytes,
                              hipMemcpyHostToDevice, stream[i]));
        CHECK(hipMemcpyAsync(&d_B[ioffset], &h_B[ioffset], iBytes,
                              hipMemcpyHostToDevice, stream[i]));
        sumArrays<<<grid, block, 0, stream[i]>>>(&d_A[ioffset], &d_B[ioffset], &d_C[ioffset], iElem);
        CHECK(hipMemcpyAsync(&gpuRef[ioffset], &d_C[ioffset], iBytes,
                              hipMemcpyDeviceToHost, stream[i]));
    }

    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    float execution_time;
    CHECK(hipEventElapsedTime(&execution_time, start, stop));

    printf("\n");
    printf("Actual results from overlapped data transfers:\n");
    printf(" overlap with %d streams : %f ms (%f GB/s)\n", NSTREAM,
           execution_time, (nBytes * 2e-6) / execution_time );
    printf(" speedup                : %f \n",
           ((itotal - execution_time) * 100.0f) / itotal);

    CHECK(hipGetLastError());

    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));

    CHECK(hipHostFree(h_A));
    CHECK(hipHostFree(h_B));
    CHECK(hipHostFree(hostRef));
    CHECK(hipHostFree(gpuRef));

    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));

    for (int i = 0; i < NSTREAM; ++i)
    {
        CHECK(hipStreamDestroy(stream[i]));
    }

    CHECK(hipDeviceReset());
    return(0);
}
