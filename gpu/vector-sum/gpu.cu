#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call)                                                  \
{                                                                    \
    const hipError_t error = call;                                  \
    if (error != hipSuccess)                                        \
    {                                                                \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                \
        printf("code:%d, reason: %s\n", error,                       \
                hipGetErrorString(error));                          \
        exit(1);                                                     \
    }                                                                \
}

void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i],
                   gpuRef[i], i);
            break;
        }
    }

    if (match) printf("Arrays match.\n\n");

    return;
}


void initialData(float *ip, int size)
{
    time_t t;
    srand((unsigned) time(&t));

    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }

    return;
}


void sumArraysOnHost(float *A, float *B, float *C, const int N)
{
    for (int idx = 0; idx < N; idx++)
        C[idx] = A[idx] + B[idx];
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    C[i] = A[i] + B[i];
    C[i] = A[i] - B[i];
}


int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);

    int dev = 0;
    hipSetDevice(dev);

    int nElem = 1000000000;
    printf("Vector size %d\n", nElem);

    size_t nBytes = nElem * sizeof(float);

    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A     = (float *)malloc(nBytes);
    h_B     = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef  = (float *)malloc(nBytes);

    initialData(h_A, nElem);
    initialData(h_B, nElem);

    memset(hostRef, 0, nBytes);
    memset(gpuRef,  0, nBytes);

    float *d_A, *d_B, *d_C;
    hipMalloc((float**)&d_A, nBytes);
    hipMalloc((float**)&d_B, nBytes);
    hipMalloc((float**)&d_C, nBytes);

    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice);

    dim3 block (nElem);
    dim3 grid  (10);

    sumArraysOnGPU<<<grid, block>>>(d_A, d_B, d_C, nElem);
    printf("Execution configure <<<%d, %d>>>\n", grid.x, block.x);

    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));

    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    hipDeviceReset();
    return(0);
}
