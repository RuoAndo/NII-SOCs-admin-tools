#include "../common/common.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

int main(int argc, char **argv)
{
    int ngpus;

    printf("> starting %s", argv[0]);

    CHECK(hipGetDeviceCount(&ngpus));
    printf(" CUDA-capable devices: %i\n", ngpus);

    return EXIT_SUCCESS;
}
