#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>

#include <sys/socket.h>
#include <netinet/in.h>
#include <arpa/inet.h>

#include "tbb/concurrent_hash_map.h"
#include "tbb/blocked_range.h"
#include "tbb/parallel_for.h"
#include "tbb/tick_count.h"
#include "tbb/task_scheduler_init.h"
//  #include "tbb/tbb_allocator.hz"
#include "utility.h"

#include "csv.hpp"
typedef std::basic_string<char,std::char_traits<char>,tbb::tbb_allocator<char> > MyString;

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include "util.h"
#include "timer.h"

using namespace tbb;
using namespace std;

static bool verbose = false;
static bool silent = false;

// const int size_factor = 2;
// typedef concurrent_hash_map<MyString,int> StringTable;
typedef concurrent_hash_map<MyString,std::vector<string>> StringTable;
std::vector<string> v_pair;
std::vector<string> v_count;
static MyString* Data;

std::vector<std::string> split_string_2(std::string str, char del) {
  int first = 0;
  int last = str.find_first_of(del);

  std::vector<std::string> result;

  while (first < str.size()) {
    std::string subStr(str, first, last - first);

    result.push_back(subStr);

    first = last + 1;
    last = str.find_first_of(del, first);

    if (last == std::string::npos) {
      last = str.size();
    }
  }

  return result;
}

int main( int argc, char* argv[] ) {

  // int counter = 0;
  int N = atoi(argv[2]);  
  unsigned int t, travdirtime; 

  int ngpus;
  // unsigned int t, travdirtime;

  printf("> starting %s", argv[0]);
  hipGetDeviceCount(&ngpus);
  printf(" CUDA-capable devices: %i\n", ngpus);     

  thrust::host_vector<long> h_vec_1_0(N);
  thrust::host_vector<long> h_vec_2_0(N);

  thrust::host_vector<long> h_vec_1_1(N);
  thrust::host_vector<long> h_vec_2_1(N);
  thrust::host_vector<long> h_vec_1_2(N);
  thrust::host_vector<long> h_vec_2_2(N);   

  tbb::tick_count mainStartTime = tbb::tick_count::now();
        srand(2);
        utility::thread_number_range threads(tbb::task_scheduler_init::default_num_threads,0);
        if ( silent ) verbose = false;

        Data = new MyString[N];

	const string csv_file = std::string(argv[1]); 
	vector<vector<string>> data; 

	  cout << "reading file..." << endl;
	  mainStartTime = tbb::tick_count::now();
	  start_timer(&t); 

	  Csv objCsv(csv_file);
	  if (!objCsv.getCsv(data)) {
	    cout << "read ERROR" << endl;
	    return 1;
	  }
	 
      	  for (int row = 0; row < data.size(); row++) {
	    vector<string> rec = data[row]; 
	    std::string timestamp = rec[0];

	    for(size_t c = timestamp.find_first_of("\""); c != string::npos; c = c = timestamp.find_first_of("\"")){
	      timestamp.erase(c,1);
	    }
	    for(size_t c = timestamp.find_first_of("."); c != string::npos; c = c = timestamp.find_first_of(".")){
	      timestamp.erase(c,1);
	    }
	    for(size_t c = timestamp.find_first_of(" "); c != string::npos; c = c = timestamp.find_first_of(" ")){
	      timestamp.erase(c,1);
	    }
	    for(size_t c = timestamp.find_first_of(":"); c != string::npos; c = c = timestamp.find_first_of(":")){
	      timestamp.erase(c,1);
	    }
	    for(size_t c = timestamp.find_first_of("/"); c != string::npos; c = c = timestamp.find_first_of("/")){
	      timestamp.erase(c,1);
	    }

	    /* sequential 0 */
	    h_vec_1_0.push_back(std::atol(timestamp.c_str()));
	    h_vec_2_0.push_back(1);
	    
	    if(row<(N/2))
	    {
		h_vec_1_1.push_back(std::atol(timestamp.c_str()));
	    	h_vec_2_1.push_back(1);
	    }
	    if(row>(N/2))
	    {
		h_vec_1_2.push_back(std::atol(timestamp.c_str()));
	   	h_vec_2_2.push_back(1);
	    }
	    
	  }

	  travdirtime = stop_timer(&t);
          print_timer(travdirtime); 
	  utility::report_elapsed_time((tbb::tick_count::now() - mainStartTime).seconds());

	  cout << "sequential" << endl;
	  // tbb::tick_count mainStartTime = tbb::tick_count::now();
	  start_timer(&t); 
	  thrust::device_vector<long> key_in_0 = h_vec_1_0;
	  thrust::device_vector<long> value_in_0 = h_vec_2_0;

	  thrust::sort(key_in_0.begin(), key_in_0.end());
	  thrust::device_vector<long> dkey_out_0(N,0);
	  thrust::device_vector<long> dvalue_out_0(N,0);

	  auto new_end_0 = thrust::reduce_by_key(key_in_0.begin(),
					       key_in_0.end(),
					       value_in_0.begin(),
					       dkey_out_0.begin(),
	  				       dvalue_out_0.begin());

	  long new_size_0 = new_end_0.first - dkey_out_0.begin();

	  travdirtime = stop_timer(&t);
          print_timer(travdirtime); 
	  utility::report_elapsed_time((tbb::tick_count::now() - mainStartTime).seconds());

	  std::remove("tmp-sequential");
	  ofstream outputfile("tmp-sequential");

	  cout << "writing file.." << endl;
	  mainStartTime = tbb::tick_count::now();
	  start_timer(&t); 
	  for(long i=0; i <new_size_0; i++)
	    {
		outputfile << dkey_out_0[i] << "," << dvalue_out_0[i] << endl;
	    }

	  outputfile.close();

	  travdirtime = stop_timer(&t);
          print_timer(travdirtime); 
	  utility::report_elapsed_time((tbb::tick_count::now() - mainStartTime).seconds());
	  
	  /* streams 1,2 */

	  cout << "streams" << endl;
	  start_timer(&t); 
	  mainStartTime = tbb::tick_count::now();
	  hipStream_t *stream = (hipStream_t *)malloc(sizeof(hipStream_t) * 2);

	  for (int i = 0; i < 2; i++)
	  {
	      hipSetDevice(i);
	      hipStreamCreate(&stream[i]);
	  }
	
	  hipSetDevice(0);
	  thrust::device_vector<long> key_in_1 = h_vec_1_1;
	  thrust::device_vector<long> value_in_1 = h_vec_2_1;
	  thrust::sort(thrust::cuda::par.on(stream[0]), key_in_1.begin(), key_in_1.end());

	  hipSetDevice(1);
	  thrust::device_vector<long> key_in_2 = h_vec_1_2;
	  thrust::device_vector<long> value_in_2 = h_vec_2_2; 
	  thrust::sort(thrust::cuda::par.on(stream[1]), key_in_2.begin(), key_in_2.end());
	  
	  for (int i = 0; i < 2; i++)
	  {
	  	  hipSetDevice(i);
	          hipStreamSynchronize(stream[i]);
          }

	  for (int i = 0; i < 2; i++)
	  {
	  	  hipSetDevice(i);
              	  hipStreamDestroy(stream[i]);
          }
	  
	  travdirtime = stop_timer(&t);
          print_timer(travdirtime); 
	  utility::report_elapsed_time((tbb::tick_count::now() - mainStartTime).seconds());
	  
	  for (int i = 0; i < 2; i++)
	  {
	  	 hipSetDevice(i);
              	 hipStreamCreate(&stream[i]);
          }

	  hipSetDevice(0);
	  thrust::device_vector<long> dkey_out_1(N,0);
	  thrust::device_vector<long> dvalue_out_1(N,0);
	  auto new_end_1 = thrust::reduce_by_key(thrust::cuda::par.on(stream[0]), key_in_1.begin(),
					       key_in_1.end(),
					       value_in_1.begin(),
					       dkey_out_1.begin(),
	  				       dvalue_out_1.begin());
					       
	  hipSetDevice(1);
	  thrust::device_vector<long> dkey_out_2(N,0);
	  thrust::device_vector<long> dvalue_out_2(N,0);
	  auto new_end_2 = thrust::reduce_by_key(thrust::cuda::par.on(stream[1]), key_in_2.begin(),
					       key_in_2.end(),
					       value_in_2.begin(),
					       dkey_out_2.begin(),
	  				       dvalue_out_2.begin());

	  for (int i = 0; i < 2; i++)
	  {
	  	   hipSetDevice(i);
	           hipStreamSynchronize(stream[i]);
          }

	  for (int i = 0; i < 2; i++)
	  {   
		   hipSetDevice(i);
              	   hipStreamDestroy(stream[i]);
          }

          /*
	  travdirtime = stop_timer(&t);
          print_timer(travdirtime); 
	  utility::report_elapsed_time((tbb::tick_count::now() - mainStartTime).seconds());
	  */

	  hipSetDevice(0);
	  long new_size_1 = new_end_1.first - dkey_out_1.begin();

	  hipSetDevice(1);
	  long new_size_2 = new_end_2.first - dkey_out_2.begin();

	  hipSetDevice(0);
          thrust::host_vector<long> hkey_out_3(N,0);
	  thrust::host_vector<long> hvalue_out_3(N,0);

	  hipSetDevice(0);
	  for(long i=0; i <new_size_1; i++)
	    {
		hkey_out_3.push_back(dkey_out_1[i]);
		hvalue_out_3.push_back(dvalue_out_1[i]);
	    }

	  hipSetDevice(1);
	  for(long i=0; i <new_size_2; i++)
	    {
		hkey_out_3.push_back(dkey_out_2[i]);
		hvalue_out_3.push_back(dvalue_out_2[i]);
	    }

	  hipSetDevice(0);
	  		
          thrust::device_vector<long> dkey_out_3 = hkey_out_3;
	  thrust::device_vector<long> dvalue_out_3 = hvalue_out_3;

	  thrust::device_vector<long> dkey_out_4(N,0);
	  thrust::device_vector<long> dvalue_out_4(N,0);

	  auto new_end_4= thrust::reduce_by_key(dkey_out_3.begin(),
					       dkey_out_3.end(),
					       dvalue_out_3.begin(),
					       dkey_out_4.begin(),
	  				       dvalue_out_4.begin());

	  long new_size_4 = new_end_4.first - dkey_out_4.begin();
	 
	  cout << "writing file..." << endl;

	  std::remove("tmp-streams");
	  ofstream outputfile2("tmp-streams");
	  
	  for(long i=0; i <new_size_1; i++)
	    {
		outputfile2 << dkey_out_1[i] << "," << dvalue_out_1[i] << endl;
	    }

	  for(long i=0; i <new_size_2; i++)
	    {
		outputfile2 << dkey_out_2[i] << "," << dvalue_out_2[i] << endl;
	    }

	  outputfile2.close();

}
