#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>

#include <sys/socket.h>
#include <netinet/in.h>
#include <arpa/inet.h>

#include "tbb/concurrent_hash_map.h"
#include "tbb/blocked_range.h"
#include "tbb/parallel_for.h"
#include "tbb/tick_count.h"
#include "tbb/task_scheduler_init.h"
#include "utility.h"

#include "csv.hpp"
// typedef std::basic_string<char,std::char_traits<char>,tbb::tbb_allocator<char> > MyString;

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include "util.h"
#include "timer.h"

using namespace tbb;
using namespace std;

static bool verbose = false;
static bool silent = false;

std::vector<std::string> split_string_2(std::string str, char del) {
  int first = 0;
  int last = str.find_first_of(del);

  std::vector<std::string> result;

  while (first < str.size()) {
    std::string subStr(str, first, last - first);

    result.push_back(subStr);

    first = last + 1;
    last = str.find_first_of(del, first);

    if (last == std::string::npos) {
      last = str.size();
    }
  }

  return result;
}

int main( int argc, char* argv[] ) {

  int N = atoi(argv[2]);  
  unsigned int t, travdirtime; 
  // int counter;

  int ngpus;

  printf("> starting %s", argv[0]);
  hipGetDeviceCount(&ngpus);
  printf(" CUDA-capable devices: %i\n", ngpus);     

  thrust::host_vector<long> h_vec_1_0(N);
  thrust::host_vector<long> h_vec_2_0(N);
  thrust::device_vector<long> d_vec_1_unique(N);

  tbb::tick_count mainStartTime = tbb::tick_count::now();

  const string csv_file = std::string(argv[1]); 
  vector<vector<string>> data; 

  cout << "reading file...";
  mainStartTime = tbb::tick_count::now();

  Csv objCsv(csv_file);
  if (!objCsv.getCsv(data)) {
     cout << "read ERROR" << endl;
     return 1;
     }

  start_timer(&t); 
  mainStartTime = tbb::tick_count::now();
  for (int row = 0; row < data.size(); row++) {
      vector<string> rec = data[row];
      std::string timestamp = rec[0];

      for(size_t c = timestamp.find_first_of("\""); c != string::npos; c = c = timestamp.find_first_of("\"")){
      		 timestamp.erase(c,1);
      }
      for(size_t c = timestamp.find_first_of("."); c != string::npos; c = c = timestamp.find_first_of(".")){
      		 timestamp.erase(c,1);
      }
      for(size_t c = timestamp.find_first_of(" "); c != string::npos; c = c = timestamp.find_first_of(" ")){
 	         timestamp.erase(c,1);
      }
      for(size_t c = timestamp.find_first_of(":"); c != string::npos; c = c = timestamp.find_first_of(":")){
	         timestamp.erase(c,1);
      }
      for(size_t c = timestamp.find_first_of("/"); c != string::npos; c = c = timestamp.find_first_of("/")){
	         timestamp.erase(c,1);
      }
      
      h_vec_1_0.push_back(std::atol(timestamp.c_str()));
      h_vec_2_0.push_back(1);

      // h_A[row] = std::atol(timestamp.c_str());

      if(row % 1000000 == 0)
      {
	utility::report_elapsed_time((tbb::tick_count::now() - mainStartTime).seconds());
      }
    }

    cout << "done. # ";
    utility::report_elapsed_time((tbb::tick_count::now() - mainStartTime).seconds());
    travdirtime = stop_timer(&t);
    print_timer(travdirtime);
    
    mainStartTime = tbb::tick_count::now();
    thrust::device_vector<long> d_vec_1_0 = h_vec_1_0;
    // std::cout << h_vec_1_0.size() << endl;
    auto new_end_0 = thrust::unique(h_vec_1_0.begin(), h_vec_1_0.end());
    int new_size_0 = new_end_0 - h_vec_1_0.begin();
    std::cout << h_vec_1_0.size() << "->" << new_size_0 << endl;
    // std::cout << h_vec_1_0.size() << endl;
    std::cout << "unique # "; // << std::endl;
    utility::report_elapsed_time((tbb::tick_count::now() - mainStartTime).seconds());

    mainStartTime = tbb::tick_count::now();
    size_t nBytes = new_size_0 * sizeof(long);
    long *d_A;
    hipMalloc((long**)&d_A, nBytes);
    long *h_A;
    h_A = (long *)malloc(nBytes);

    for(int i = 0; i < new_size_0; i++)
    	    h_A[i] = h_vec_1_0[i];
    // hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);

    // int ngpus = 2;
    long **d_src = (long **)malloc(sizeof(long) * ngpus);
    long **d_rcv = (long **)malloc(sizeof(long) * ngpus);  

    for (int i = 0; i < ngpus; i++)
    {
	hipSetDevice(i);
    	hipMalloc(&d_src[i], nBytes);
    	hipMalloc(&d_rcv[i], nBytes);
	hipMemcpy(d_src[i], h_A, nBytes, hipMemcpyHostToDevice);
    }
    std::cout << "GPU0: host_vector->device memory # "; // << std::endl;
    utility::report_elapsed_time((tbb::tick_count::now() - mainStartTime).seconds());

    start_timer(&t); 
    mainStartTime = tbb::tick_count::now();
    hipMemcpy(d_src[0], d_src[1], nBytes, hipMemcpyDeviceToDevice);
    std::cout << "GPU0->GPU1: device memory -> device memory # "; // << std::endl;
    utility::report_elapsed_time((tbb::tick_count::now() - mainStartTime).seconds());
    travdirtime = stop_timer(&t);
    print_timer(travdirtime);

    mainStartTime = tbb::tick_count::now();
    hipSetDevice(1);
    thrust::device_vector<long> d_vec_1(new_size_0);    

    for(int i = 0; i < new_size_0; i++)
    	    d_vec_1[i] = d_src[1][i];
    std::cout << "GPU1: device memory-> device vector # "; // << std::endl;
    utility::report_elapsed_time((tbb::tick_count::now() - mainStartTime).seconds());

    // long *new_end = thrust::unique(h_A, h_A + N);
    // for( auto itr = new_end.begin(); itr < new_end.end(); itr++)

    mainStartTime = tbb::tick_count::now();
    hipSetDevice(1);
    thrust::device_vector<long> d_vec_2(new_size_0);    
    for(int i = 0; i < new_size_0; i++)
    	    d_vec_2[i] = h_vec_1_0[i];
    std::cout << "GPU0->GPU1: host_vector -> device_vector # ";
    utility::report_elapsed_time((tbb::tick_count::now() - mainStartTime).seconds());

    mainStartTime = tbb::tick_count::now();
    hipSetDevice(1);
    // thrust::device_vector<long> d_vec_2(new_size_0);    
    for(int i = 0; i < new_size_0; i++)
    	    h_vec_1_0[i] = d_vec_2[i];
    std::cout << "GPU1: device_vector -> host_vector # ";
    utility::report_elapsed_time((tbb::tick_count::now() - mainStartTime).seconds());

    // utility::report_elapsed_time((tbb::tick_count::now() - mainStartTime).seconds());

}
