#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>

#include <sys/socket.h>
#include <netinet/in.h>
#include <arpa/inet.h>

#include "tbb/concurrent_hash_map.h"
#include "tbb/blocked_range.h"
#include "tbb/parallel_for.h"
#include "tbb/tick_count.h"
#include "tbb/task_scheduler_init.h"
//  #include "tbb/tbb_allocator.hz"
#include "utility.h"

#include "csv.hpp"
typedef std::basic_string<char,std::char_traits<char>,tbb::tbb_allocator<char> > MyString;

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include "util.h"
#include "timer.h"

using namespace tbb;
using namespace std;

static bool verbose = false;
static bool silent = false;

// const int size_factor = 2;
// typedef concurrent_hash_map<MyString,int> StringTable;
typedef concurrent_hash_map<MyString,std::vector<string>> StringTable;
std::vector<string> v_pair;
std::vector<string> v_count;
static MyString* Data;

std::vector<std::string> split_string_2(std::string str, char del) {
  int first = 0;
  int last = str.find_first_of(del);

  std::vector<std::string> result;

  while (first < str.size()) {
    std::string subStr(str, first, last - first);

    result.push_back(subStr);

    first = last + 1;
    last = str.find_first_of(del, first);

    if (last == std::string::npos) {
      last = str.size();
    }
  }

  return result;
}

int main( int argc, char* argv[] ) {

  // int counter = 0;
  int N = atoi(argv[2]);  
  unsigned int t, travdirtime; 

  int ngpus;
  // unsigned int t, travdirtime;

  printf("> starting %s", argv[0]);
  hipGetDeviceCount(&ngpus);
  printf(" CUDA-capable devices: %i\n", ngpus);     

  thrust::host_vector<long> h_vec_1_0(N);
  thrust::host_vector<long> h_vec_2_0(N);

  thrust::host_vector<long> h_vec_1_1(N);
  thrust::host_vector<long> h_vec_2_1(N);
  thrust::host_vector<long> h_vec_1_2(N);
  thrust::host_vector<long> h_vec_2_2(N);   

  tbb::tick_count mainStartTime = tbb::tick_count::now();
        srand(2);
        utility::thread_number_range threads(tbb::task_scheduler_init::default_num_threads,0);
        if ( silent ) verbose = false;

        Data = new MyString[N];

	const string csv_file = std::string(argv[1]); 
	vector<vector<string>> data; 

	  cout << "reading file..." << endl;
	  mainStartTime = tbb::tick_count::now();
	  start_timer(&t); 

	  Csv objCsv(csv_file);
	  if (!objCsv.getCsv(data)) {
	    cout << "read ERROR" << endl;
	    return 1;
	  }

	  unsigned long long *timestamp_h;
          long *counted_h;    

    	  unsigned long long tBytes = data.size() * sizeof(unsigned long long);
	  long cBytes = data.size() * sizeof(long);   

	  timestamp_h = (unsigned long long *)malloc(tBytes);
	  counted_h = (long *)malloc(cBytes);  

	  unsigned long *timestamp_d;
	  long *counted_d;
	  
	  hipMalloc((unsigned long long**)&timestamp_d, tBytes);
	  hipMalloc((long**)&counted_d, cBytes);
	   
      	  for (int row = 0; row < data.size(); row++) {
	    vector<string> rec = data[row]; 
	    std::string timestamp = rec[0];

	    for(size_t c = timestamp.find_first_of("\""); c != string::npos; c = c = timestamp.find_first_of("\"")){
	      timestamp.erase(c,1);
	    }
	    for(size_t c = timestamp.find_first_of("."); c != string::npos; c = c = timestamp.find_first_of(".")){
	      timestamp.erase(c,1);
	    }
	    for(size_t c = timestamp.find_first_of(" "); c != string::npos; c = c = timestamp.find_first_of(" ")){
	      timestamp.erase(c,1);
	    }
	    for(size_t c = timestamp.find_first_of(":"); c != string::npos; c = c = timestamp.find_first_of(":")){
	      timestamp.erase(c,1);
	    }
	    for(size_t c = timestamp.find_first_of("/"); c != string::npos; c = c = timestamp.find_first_of("/")){
	      timestamp.erase(c,1);
	    }

	    timestamp_h[row] = std::stoull(timestamp);
	    counted_h[row] = 1;
	    
	  }

	  travdirtime = stop_timer(&t);
          print_timer(travdirtime); 
	  utility::report_elapsed_time((tbb::tick_count::now() - mainStartTime).seconds());

	  cout << "reading file..." << endl;
	  mainStartTime = tbb::tick_count::now();
	  start_timer(&t); 

	  hipStream_t stream[2]; 

	  for (int i = 0; i < 2; ++i)
              {
	        hipStreamCreate(&stream[i]);
          	}

          for (int i = 0; i < 2; ++i)
	      {
	    	hipMemcpyAsync(&timestamp_d[data.size()], &timestamp_h[data.size()], tBytes,
						   hipMemcpyHostToDevice, stream[i]);
              }	

	  travdirtime = stop_timer(&t);
          print_timer(travdirtime); 
	  utility::report_elapsed_time((tbb::tick_count::now() - mainStartTime).seconds());

}
