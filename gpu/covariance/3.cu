#include <hipblas.h>

#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/sequence.h>

#include <stdio.h>
#include <iostream>

#include "Utilities.cuh"
#include "TimingGPU.cuh"

#include "csv.hpp"
using namespace std;

#define SAMPLE_RATIO 100

/********/
/* MAIN */
/********/
int main( int argc, char* argv[] )
{
  int N = atoi(argv[2]);
  
  thrust::host_vector<long> h_vec_1(N);
  thrust::host_vector<long> h_vec_2(N);   

  const string csv_file = std::string(argv[1]); 
  vector<vector<string>> data; 

  Csv objCsv(csv_file);
  if (!objCsv.getCsv(data)) {
     cout << "read ERROR" << endl;
     return 1;
  }

  for (int row = 0; row < data.size(); row++) {
      vector<string> rec = data[row]; 
      std::string timestamp = rec[0];

      h_vec_1.push_back(std::stof(rec[0].c_str()));
      h_vec_2[row] = std::stof(rec[1]);
  }

  thrust::host_vector<long> dout_2(N);
  thrust::host_vector<long> dout_3(N);
  thrust::host_vector<long> d_vec_2(N);
  
  thrust::copy(h_vec_2.begin(), h_vec_2.end(), d_vec_2.begin());   
  thrust::inclusive_scan(d_vec_2.begin(), d_vec_2.end(), dout_2.begin());
  thrust::exclusive_scan(d_vec_2.begin(), d_vec_2.end(), dout_3.begin());

  for(int i=0;i<N;i++)
  {
	// cout << d_vec_2[i] << "," << dout_2[i] << "," << dout_3[i] << endl;
	cout << d_vec_2[i] << "," << dout_3[i] << endl;
  }	  
  return 0;
}
