#include <hipblas.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/sequence.h>

#include <stdio.h>
#include <iostream>

#include "Utilities.cuh"
#include "TimingGPU.cuh"

#include "csv.hpp"
using namespace std;

/*************************************/
/* CONVERT LINEAR INDEX TO ROW INDEX */
/*************************************/
template <typename T>
struct linear_index_to_row_index : public thrust::unary_function<T,T> {

    T Ncols; // --- Number of columns

    __host__ __device__ linear_index_to_row_index(T Ncols) : Ncols(Ncols) {}

    __host__ __device__ T operator()(T i) { return i / Ncols; }
};

/********/
/* MAIN */
/********/
int main( int argc, char* argv[] )
{
  int N = atoi(argv[2]);
  
  thrust::host_vector<long> h_vec_1(N);
  thrust::host_vector<long> h_vec_2(N);   

  const string csv_file = std::string(argv[1]); 
  vector<vector<string>> data; 

  Csv objCsv(csv_file);
  if (!objCsv.getCsv(data)) {
     cout << "read ERROR" << endl;
     return 1;
  }

  for (int row = 0; row < data.size(); row++) {
      vector<string> rec = data[row]; 
      std::string timestamp = rec[0];

      h_vec_1.push_back(std::stof(rec[0].c_str()));
      h_vec_2[row] = std::stof(rec[1]);
  }

  thrust::host_vector<long> dout_2(N);
  thrust::host_vector<long> dout_3(N);
  thrust::host_vector<long> d_vec_2(N);
  
  thrust::copy(h_vec_2.begin(), h_vec_2.end(), d_vec_2.begin());   
  thrust::inclusive_scan(d_vec_2.begin(), d_vec_2.end(), dout_2.begin());
  thrust::exclusive_scan(d_vec_2.begin(), d_vec_2.end(), dout_3.begin());

  const int Nsamples = 2;
    // --- Number of realizations for each random variable (number of rows of the X matrix)
  const int NX = N;
    // --- Number of random variables (number of columns of the X matrix)

    // --- Matrix allocation and initialization
  thrust::device_vector<long> d_X(Nsamples * NX);

    for(int i=0;i<N;i++)
    {	    
    	    d_X[i] = dout_2[i];
	    d_X[i+NX] = dout_3[i];
    }

    // --- cuBLAS handle creation
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    /*************************************************/
    /* CALCULATING THE MEANS OF THE RANDOM VARIABLES */
    /*************************************************/
    // --- Array containing the means multiplied by Nsamples
    thrust::device_vector<long> d_means(NX);
    thrust::device_vector<long> d_ones(Nsamples, 1.f);

    long alpha = 1.f / (long)Nsamples;
    long beta  = 0.f;
    hipblasSgemv(handle, HIPBLAS_OP_T, Nsamples, NX, &alpha, thrust::raw_pointer_cast(d_X.data()), Nsamples, 
                thrust::raw_pointer_cast(d_ones.data()), 1, &beta, thrust::raw_pointer_cast(d_means.data()), 1);

    /**********************************************/
    /* SUBTRACTING THE MEANS FROM THE MATRIX ROWS */
    /**********************************************/
    thrust::transform(
                d_X.begin(), d_X.end(),
                thrust::make_permutation_iterator(
                        d_means.begin(),
                        thrust::make_transform_iterator(thrust::make_counting_iterator(0), linear_index_to_row_index<int>(Nsamples))),
                d_X.begin(),
                thrust::minus<float>());    

    /*************************************/
    /* CALCULATING THE COVARIANCE MATRIX */
    /*************************************/
    thrust::device_vector<float> d_cov(NX * NX);

    alpha = 1.f;
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, NX, NX, Nsamples, &alpha,
                thrust::raw_pointer_cast(d_X.data()), Nsamples, thrust::raw_pointer_cast(d_X.data()), Nsamples, &beta,
                thrust::raw_pointer_cast(d_cov.data()), NX);

    // --- Final normalization by Nsamples - 1
    thrust::transform(
                d_cov.begin(), d_cov.end(),
                thrust::make_constant_iterator((float)(Nsamples-1)),
                d_cov.begin(),
                thrust::divides<float>());  

    // for(int i = 0; i < NX * NX; i++) std::cout << d_cov[i] << "\n";

    int counter = 0;
    for(int i = 0; i < NX; i++)
    {
        for(int j = 0; j < NX; j++)
	{
		if(i == j)
		{
		std::cout << d_X[counter%NX] << "," << d_cov[i+j*NX] << std::endl;
		}
	counter = counter + 1;
	}
	
    }

    return 0;
}
