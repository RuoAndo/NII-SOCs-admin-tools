#include <hipblas.h>

#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/sequence.h>

#include <stdio.h>
#include <iostream>

#include "Utilities.cuh"
#include "TimingGPU.cuh"

#include "csv.hpp"

/*************************************/
/* CONVERT LINEAR INDEX TO ROW INDEX */
/*************************************/
template <typename T>
struct linear_index_to_row_index : public thrust::unary_function<T,T> {

    T Ncols; // --- Number of columns

    __host__ __device__ linear_index_to_row_index(T Ncols) : Ncols(Ncols) {}

    __host__ __device__ T operator()(T i) { return i / Ncols; }
};

using namespace std;

#define SAMPLE_RATIO 100

/********/
/* MAIN */
/********/
int main( int argc, char* argv[] )
{
    int N = atoi(argv[2]);  

    const int Nsamples = 2;
    // --- Number of realizations for each random variable (number of rows of the X matrix)

    const int NX  = N;
    // --- Number of random variables (number of columns of the X matrix)

    int counter = 0;

  thrust::host_vector<float> h_vec_1(N);
  thrust::host_vector<float> h_vec_2(N);   

  const string csv_file = std::string(argv[1]); 
  vector<vector<string>> data; 

  Csv objCsv(csv_file);
  if (!objCsv.getCsv(data)) {
     cout << "read ERROR" << endl;
     return 1;
  }

  /*
  std::remove("tmp");
  ofstream outputfile("tmp");
  */

  for (int row = 0; row < data.size(); row++) {
      vector<string> rec = data[row]; 
      std::string timestamp = rec[0];

      // std::cout << rec[1] << std::endl; 

      h_vec_1.push_back(std::stof(rec[0].c_str()));
      // h_vec_2.push_back(std::stof(rec[1]));
      h_vec_2[row] = std::stof(rec[1]);
      // h_vec_1[row] = std::stof(rec[0]);

      // std::cout << std::stof(rec[1]) << std::endl;
      // std::cout << h_vec_2[row] << std::endl; 
  }

  // --- Matrix allocation and initialization
  thrust::device_vector<float> d_X(Nsamples * NX);
  thrust::device_vector<float> d_X_bak(Nsamples * NX);

    for (size_t i = 0; i < NX; i++)
    	{
	d_X[i] = h_vec_2[i];
   	d_X_bak[i] = d_X[i];
	}

	// d_X[NX]=0;
	// d_X_bak[NX]=0;
	
    for (size_t i = 0; i < NX-1; i++)
    	{
	d_X[i+NX] = d_X[i];
	d_X_bak[i+NX] = d_X[i];
	}

    // d_X[NX*Nsamples-1] = 0;
    //d_X_bak[NX*Nsamples-1] = 0;
    // d_X[NX-1] = 0;
    // d_X_bak[NX-1] = 0;
    		  

    // for (size_t i = 0; i < NX * Nsamples; i++)
    	// std::cout << d_X[i] << std::endl;

    // --- cuBLAS handle creation
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    /*************************************************/
    /* CALCULATING THE MEANS OF THE RANDOM VARIABLES */
    /*************************************************/
    // --- Array containing the means multiplied by Nsamples
    thrust::device_vector<float> d_means(NX);
    thrust::device_vector<float> d_ones(Nsamples, 1.f);

    float alpha = 1.f / (float)Nsamples;
    float beta  = 0.f;
    hipblasSgemv(handle, HIPBLAS_OP_T, Nsamples, NX, &alpha, thrust::raw_pointer_cast(d_X.data()), Nsamples, 
                               thrust::raw_pointer_cast(d_ones.data()), 1, &beta, thrust::raw_pointer_cast(d_means.data()), 1);

    /**********************************************/
    /* SUBTRACTING THE MEANS FROM THE MATRIX ROWS */
    /**********************************************/
    thrust::transform(
                d_X.begin(), d_X.end(),
                thrust::make_permutation_iterator(
                        d_means.begin(),
                        thrust::make_transform_iterator(thrust::make_counting_iterator(0), linear_index_to_row_index<int>(Nsamples))),
                d_X.begin(),
                thrust::minus<float>());    

    /*************************************/
    /* CALCULATING THE COVARIANCE MATRIX */
    /*************************************/
    thrust::device_vector<float> d_cov(NX * NX);

    alpha = 1.f;
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, NX, NX, Nsamples, &alpha,
                               thrust::raw_pointer_cast(d_X.data()), Nsamples, thrust::raw_pointer_cast(d_X.data()), Nsamples, &beta,
                               thrust::raw_pointer_cast(d_cov.data()), NX);

    // --- Final normalization by Nsamples - 1
    thrust::transform(
                d_cov.begin(), d_cov.end(),
                thrust::make_constant_iterator((float)(Nsamples-1)),
                d_cov.begin(),
                thrust::divides<float>());  

    // for(int i = 0; i < N * Nsamples; i++) std::cout << d_X[i] << "\n";
    // std::cout << "size:"  << d_cov.size() << std::endl;

    // for (size_t i = 0; i < d_X.size(); i++) std::cout << "d_X_bak:" << d_X_bak[i] << std::endl;

    counter = 0;
    for(int i = 0; i < NX; i++)
    {
        for(int j = 0; j < NX; j++)
	{
		if(i == j)
		{
			std::cout << d_X_bak[counter%NX] << "," << d_cov[i+j*NX] << "," << d_X_bak[counter%NX] / d_cov[i+j*NX] << std::endl;
		}
	counter = counter + 1;
	}
	
    }
	
    return 0;
}
