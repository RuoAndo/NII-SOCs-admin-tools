#include "hip/hip_runtime.h"
#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>

#include <hip/hip_runtime.h>
#include <stdio.h>
#define DIM 128

#include "csv.hpp"
#include "timer.h"
using namespace std;

extern __shared__ int dsmem[];

int recursiveReduce(int *data, int const size)
{
    if (size == 1) return data[0];

    int const stride = size / 2;

    for (int i = 0; i < stride; i++)
        data[i] += data[i + stride];

    return recursiveReduce(data, stride);
}

// unroll4 + complete unroll for loop + gmem
__global__ void reduceGmem(int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= n) return;

    // in-place reduction in global memory
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];

    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];

    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];

    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];

    __syncthreads();

    // unrolling warp
    if (tid < 32)
    {
        volatile int *vsmem = idata;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceSmem(int *g_idata, int *g_odata, unsigned int n)
{
    __shared__ int smem[DIM];

    // set thread ID
    unsigned int tid = threadIdx.x;

    // boundary check
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= n) return;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // set to smem by each threads
    smem[tid] = idata[tid];
    __syncthreads();

    // in-place reduction in shared memory
    if (blockDim.x >= 1024 && tid < 512) smem[tid] += smem[tid + 512];

    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) smem[tid] += smem[tid + 256];

    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) smem[tid] += smem[tid + 128];

    __syncthreads();

    if (blockDim.x >= 128 && tid < 64)  smem[tid] += smem[tid + 64];

    __syncthreads();

    // unrolling warp
    if (tid < 32)
    {
        volatile int *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}

__global__ void sumArraysOnGPU(int *A, int B, int *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) C[i] = A[i] - B;
}

int main(int argc, char **argv)
{
    int N = atoi(argv[2]);

    unsigned int t, travdirtime; 

    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("%s starting reduction at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    hipSetDevice(dev);

    // initialization
    // int size = 1 << 24; // total number of elements to reduce
    // printf("    with array size %d  ", size);

    int size = N;

    // execution configuration
    int blocksize = DIM;   // initial block size

    dim3 block (blocksize, 1);
    dim3 grid  ((size + block.x - 1) / block.x, 1);
    printf("grid %d block %d\n", grid.x, block.x);

    // allocate host memory
    size_t bytes = size * sizeof(int);
    int *h_idata = (int *) malloc(bytes);
    int *h_odata = (int *) malloc(grid.x * sizeof(int));
    int *tmp     = (int *) malloc(bytes);
    int *h_stddev = (int *) malloc(bytes);
    
    // initialize the array
    /*
    for (int i = 0; i < size; i++)
        h_idata[i] = (int)( rand() & 0xFF );
    */

    const string csv_file = std::string(argv[1]); 
    vector<vector<string>> data; 

    Csv objCsv(csv_file);
    if (!objCsv.getCsv(data)) {
     std::cout << "read ERROR" << std::endl;
     return 1;
    }

    for (int row = 0; row < data.size(); row++) {
      vector<string> rec = data[row]; 
      h_idata[row] = atoi( rec[1].c_str());
    }

    memcpy (tmp, h_idata, bytes);

    int gpu_sum = 0;

    // allocate device memory
    int *d_idata = NULL;
    int *d_odata = NULL;
    int *d_stddev = NULL;

    hipMalloc((void **) &d_idata, bytes);
    hipMalloc((void **) &d_odata, grid.x * sizeof(int));
    hipMalloc((void **) &d_stddev, bytes);

    // reduce gmem
    start_timer(&t); 
    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
    reduceGmem<<<grid.x, block>>>(d_idata, d_odata, size);
    hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    travdirtime = stop_timer(&t);
    
    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];

    printf("reduceGmem: %d <<<grid %d block %d>>>\n", gpu_sum, grid.x,
           block.x);
    print_timer(travdirtime);

    // reduce smem
    start_timer(&t); 
    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
    reduceSmem<<<grid.x, block>>>(d_idata, d_odata, size);
    hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    travdirtime = stop_timer(&t);

    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];

    printf("reduceSmem: %d <<<grid %d block %d>>>\n", gpu_sum, grid.x, block.x);
    print_timer(travdirtime);

    float avg = gpu_sum / N;

    start_timer(&t); 
    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
    sumArraysOnGPU<<<grid.x, block>>>(d_idata, avg, d_stddev, N);
    hipMemcpy(h_stddev, d_stddev, bytes, hipMemcpyDeviceToHost);
    printf("reduceArray: %f <<<grid %d block %d>>>\n", avg, grid.x, block.x);
    travdirtime = stop_timer(&t);
    print_timer(travdirtime);

    cout << "writing file..." << endl;
    std::remove("tmp");
    ofstream outputfile("tmp"); 

    start_timer(&t); 
    for(int i = 0; i < N; i++)
        outputfile << h_idata[i] << "," << h_stddev[i] << std::endl;

    outputfile.close();

    travdirtime = stop_timer(&t);
    print_timer(travdirtime);



    // free host memory
    free(h_idata);
    free(h_odata);

    // free device memory
    hipFree(d_idata);
    hipFree(d_odata);

    // reset device
    hipDeviceReset();

    return EXIT_SUCCESS;
}
