#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/for_each.h>

#include <hiprand.h>

#include <iterator>
#include <iostream>
#include <ctime>

int main() {
  using namespace std;

  const int N = 100000;
  thrust::device_vector<float> dscore(N);
  thrust::device_vector<int>   dhist(101, 0);

  { 
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, static_cast<unsigned long long>(time(nullptr)));
    hiprandGenerateNormal(gen, dscore.data().get(), N, 50.30f, 15.0f);
    hiprandDestroyGenerator(gen);
  }

  int* histPtr = dhist.data().get(); 
  thrust::for_each(begin(dscore), end(dscore),
                   [=] __device__ (float val) -> void {
                     int i = static_cast<int>(val+0.5f); 
                     if ( i >= 0 && i <= 100 ) { 
                       atomicAdd(histPtr+i, 1); 
                     }
                   });
		   
  for ( int item : dhist ) {
    cout << item << endl;
  }
}
