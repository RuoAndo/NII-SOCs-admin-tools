#include "hip/hip_runtime.h"
#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>

#include <hip/hip_runtime.h>
#include <stdio.h>
#define DIM 128

#include "csv.hpp"
#include "timer.h"
using namespace std;

extern __shared__ int dsmem[];

int recursiveReduce(int *data, int const size)
{
    if (size == 1) return data[0];

    int const stride = size / 2;

    for (int i = 0; i < stride; i++)
        data[i] += data[i + stride];

    return recursiveReduce(data, stride);
}

// unroll4 + complete unroll for loop + gmem
__global__ void reduceGmem(int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= n) return;

    // in-place reduction in global memory
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];

    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];

    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];

    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];

    __syncthreads();

    // unrolling warp
    if (tid < 32)
    {
        volatile int *vsmem = idata;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceSmem(int *g_idata, int *g_odata, unsigned int n)
{
    __shared__ int smem[DIM];

    // set thread ID
    unsigned int tid = threadIdx.x;

    // boundary check
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= n) return;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // set to smem by each threads
    smem[tid] = idata[tid];
    __syncthreads();

    // in-place reduction in shared memory
    if (blockDim.x >= 1024 && tid < 512) smem[tid] += smem[tid + 512];

    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) smem[tid] += smem[tid + 256];

    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) smem[tid] += smem[tid + 128];

    __syncthreads();

    if (blockDim.x >= 128 && tid < 64)  smem[tid] += smem[tid + 64];

    __syncthreads();

    // unrolling warp
    if (tid < 32)
    {
        volatile int *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}

__global__ void sumArraysOnGPU(unsigned long long *A, unsigned long long start_time, unsigned long long *C, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) C[i] = (A[i] - start_time) / 3;
}

__global__ void sumArraysOnGPU_2(unsigned long long *A, unsigned long long start_time, unsigned long long *C, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) C[i] = A[i] * 3 + start_time;
}

int main(int argc, char **argv)
{
    int N = atoi(argv[2]);

    unsigned int t, travdirtime; 

    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("%s starting reduction at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    hipSetDevice(dev);

    int size = N;

    // execution configuration
    // int blocksize = DIM;   // initial block size

    // dim3 block (blocksize, 1);
    // dim3 grid  ((size + block.x - 1) / block.x, 1);
    // printf("grid %d block %d\n", grid.x, block.x);

    // allocate host memory
    size_t bytes = size * sizeof(unsigned long long);
    unsigned long long *h_idata = (unsigned long long *) malloc(bytes);
    // unsigned long long *h_odata = (unsigned long long *) malloc(grid.x * sizeof(long));
    unsigned long long *h_odata = (unsigned long long *) malloc(bytes);
    unsigned long long *h_odata_2 = (unsigned long long *) malloc(bytes);
    
    const string csv_file = std::string(argv[1]); 
    vector<vector<string>> data; 

    Csv objCsv(csv_file);
    if (!objCsv.getCsv(data)) {
     std::cout << "read ERROR" << std::endl;
     return 1;
    }

    for (int row = 0; row < data.size(); row++) {
      vector<string> rec = data[row]; 
      h_idata[row] = stoull(rec[0]);
    }

    // allocate device memory
    unsigned long long *d_idata = NULL;
    unsigned long long *d_odata = NULL;
    unsigned long long *d_odata_2 = NULL;

    hipMalloc((unsigned long long **)&d_idata, bytes);
    // hipMalloc((void **) &d_odata, grid.x * sizeof(int));
    hipMalloc((unsigned long long **) &d_odata, bytes);
    hipMalloc((unsigned long long **) &d_odata_2, bytes);

    // unsigned long long start_time = 20190109000000000;
    unsigned long long start_time = 20190109000000000;

    start_timer(&t); 
    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
    sumArraysOnGPU<<<1, N>>>(d_idata, start_time, d_odata, N);    
    hipMemcpy(h_odata, d_odata, bytes, hipMemcpyDeviceToHost);
    travdirtime = stop_timer(&t);
    print_timer(travdirtime);

    start_timer(&t); 
    hipMemcpy(d_odata, h_odata, bytes, hipMemcpyHostToDevice);
    sumArraysOnGPU_2<<<1, N>>>(d_odata, start_time, d_odata_2, N);    
    hipMemcpy(h_odata_2, d_odata_2, bytes, hipMemcpyDeviceToHost);
    travdirtime = stop_timer(&t);
    print_timer(travdirtime);

    cout << "writing file..." << endl;
    std::remove("tmp");
    ofstream outputfile("tmp"); 

    start_timer(&t); 
    for(int i = 0; i < 10; i++)
        std::cout << h_idata[i] << "," << h_odata[i] << "," << h_odata_2[i] << std::endl;

    outputfile.close();

    travdirtime = stop_timer(&t);
    print_timer(travdirtime);

    // free host memory
    free(h_idata);
    free(h_odata);
    free(h_odata_2);

    // free device memory
    hipFree(d_idata);
    hipFree(d_odata);
    hipFree(d_odata_2);

    // reset device
    hipDeviceReset();

    return EXIT_SUCCESS;
}
