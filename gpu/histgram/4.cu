#include <hipblas.h>

#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/sequence.h>

#include <stdio.h>
#include <iostream>

/*
#include "Utilities.cuh"
#include "TimingGPU.cuh"
*/

#include "csv.hpp"
using namespace std;

int main( int argc, char* argv[] )
{
  int N = atoi(argv[2]);
  
  thrust::host_vector<unsigned long long> h_vec_1(N);
  thrust::host_vector<long> h_vec_2(N);   

  const string csv_file = std::string(argv[1]); 
  vector<vector<string>> data; 

  Csv objCsv(csv_file);
  if (!objCsv.getCsv(data)) {
     cout << "read ERROR" << endl;
     return 1;
  }

  for (int row = 0; row < data.size(); row++) {
      vector<string> rec = data[row]; 
      std::string timestamp = rec[0];
      std::string bytes = rec[3];

      for(size_t c = timestamp.find_first_of("\""); c != string::npos; c = c = timestamp.find_first_of("\"")){
	  timestamp.erase(c,1);
	}

      for(size_t c = timestamp.find_first_of("\/"); c != string::npos; c = c = timestamp.find_first_of("\/")){
	  timestamp.erase(c,1);
	}

      for(size_t c = timestamp.find_first_of("\:"); c != string::npos; c = c = timestamp.find_first_of("\:")){
	  timestamp.erase(c,1);
	}

      for(size_t c = timestamp.find_first_of(" "); c != string::npos; c = c = timestamp.find_first_of(" ")){
	  timestamp.erase(c,1);
	}

      for(size_t c = timestamp.find_first_of("."); c != string::npos; c = c = timestamp.find_first_of(".")){
	  timestamp.erase(c,1);
	}

      for(size_t c = bytes.find_first_of("\""); c != string::npos; c = c = bytes.find_first_of("\"")){
	  bytes.erase(c,1);
	}

/*
      std::cout << timestamp << std::endl;
      std::cout << bytes << std::endl;
*/

      // h_vec_1.push_back(std::stoull(timestamp.c_str()));
      h_vec_1[row] = std::stoull(timestamp.c_str());
      h_vec_2[row] = std::stol(bytes);
  }

  int in_size = N;

  thrust::device_vector<unsigned long long> key_in(N);
  thrust::device_vector<long> value_in(N);

  /*
  thrust::device_vector<unsigned long long> key_in(in_size) = h_vec_1;
  thrust::device_vector<long> value_in(in_size) = h_vec_2;
  */

  thrust::copy(h_vec_1.begin(), h_vec_1.end(), key_in.begin());
  thrust::copy(h_vec_2.begin(), h_vec_2.end(), value_in.begin());

  thrust::device_vector<unsigned long long> key_out(in_size, 0);
  thrust::device_vector<long> value_out(in_size, 0);

  thrust::sort(key_in.begin(), key_in.end());

  auto new_end = thrust::reduce_by_key(key_in.begin(),
                                     key_in.end(),
                                     value_in.begin(),
                                     key_out.begin(),
                                     value_out.begin());

  long new_size = new_end.first - key_out.begin();
  
  for(long i=0; i < new_size;i++)
  {
   std::cout << key_out[i] << "," << value_out[i] << "," << std::endl;
  }
   std::cout << std::endl;

  return 0;
}
