#include <hipblas.h>

#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/sequence.h>

#include <stdio.h>
#include <iostream>

/*
#include "Utilities.cuh"
#include "TimingGPU.cuh"
*/

#include "csv.hpp"
using namespace std;

int main( int argc, char* argv[] )
{
  int N = atoi(argv[2]);
  
  thrust::host_vector<long> h_vec_1(N);
  thrust::host_vector<long> h_vec_2(N);   

  const string csv_file = std::string(argv[1]); 
  vector<vector<string>> data; 

  Csv objCsv(csv_file);
  if (!objCsv.getCsv(data)) {
     cout << "read ERROR" << endl;
     return 1;
  }

  for (int row = 0; row < data.size(); row++) {
      vector<string> rec = data[row]; 
      std::string timestamp = rec[0];
      std::string bytes = rec[3];

      for(size_t c = timestamp.find_first_of("\""); c != string::npos; c = c = timestamp.find_first_of("\"")){
	  timestamp.erase(c,1);
	}

      for(size_t c = timestamp.find_first_of("\/"); c != string::npos; c = c = timestamp.find_first_of("\/")){
	  timestamp.erase(c,1);
	}

      for(size_t c = timestamp.find_first_of("\:"); c != string::npos; c = c = timestamp.find_first_of("\:")){
	  timestamp.erase(c,1);
	}

      for(size_t c = timestamp.find_first_of(" "); c != string::npos; c = c = timestamp.find_first_of(" ")){
	  timestamp.erase(c,1);
	}

      for(size_t c = timestamp.find_first_of("."); c != string::npos; c = c = timestamp.find_first_of(".")){
	  timestamp.erase(c,1);
	}

      for(size_t c = bytes.find_first_of("\""); c != string::npos; c = c = bytes.find_first_of("\"")){
	  bytes.erase(c,1);
	}

      std::cout << timestamp << std::endl;
      std::cout << bytes << std::endl;

      h_vec_1.push_back(std::stol(timestamp.c_str()));
      h_vec_2[row] = std::stol(bytes);
  }

  thrust::host_vector<long> dout_2(N);
  thrust::host_vector<long> dout_3(N);
  thrust::host_vector<long> d_vec_2(N);
  
  thrust::copy(h_vec_2.begin(), h_vec_2.end(), d_vec_2.begin());   
  thrust::inclusive_scan(d_vec_2.begin(), d_vec_2.end(), dout_2.begin());
  thrust::exclusive_scan(d_vec_2.begin(), d_vec_2.end(), dout_3.begin());

  for(int i=0;i<N;i++)
  {
	// cout << d_vec_2[i] << "," << dout_2[i] << "," << dout_3[i] << endl;
	cout << d_vec_2[i] << "," << dout_3[i] << endl;
  }	  
  return 0;
}
