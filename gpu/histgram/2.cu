#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>

#include <hip/hip_runtime.h>
#include <stdio.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/sequence.h>

#include "csv.hpp"
#include "timer.h"
using namespace std;

struct is_eq_zero {
    __host__ __device__ bool operator()(long a) const {
                return (a==0);
		}
};

std::vector<std::string> split_string_2(std::string str, char del) {
  int first = 0;
  int last = str.find_first_of(del);

  std::vector<std::string> result;

  while (first < str.size()) {
    std::string subStr(str, first, last - first);

    result.push_back(subStr);

    first = last + 1;
    last = str.find_first_of(del, first);

    if (last == std::string::npos) {
      last = str.size();
    }
  }

  return result;
}

int main(int argc, char **argv)
{
    int N = atoi(argv[2]);
    int INTVL = atoi(argv[3]);

    unsigned int t, travdirtime; 

    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("%s starting reduction at ", argv[0]);
    printf("device %d: %s \n", dev, deviceProp.name);
    hipSetDevice(dev);

    unsigned long long start_time = 20190114000000000;

    if (argc < 4) {
            printf("Usage: ./2 file_name nLines span IP_address \n"); return 0;
    }      

    char del = '.';
    // std::string stringIP;
    std::string IPstring;

    std::string stringIP = argv[4];

    for (const auto subStr : split_string_2(stringIP, del)) {
    	unsigned long ipaddr_src;
	ipaddr_src = atoi(subStr.c_str());
	std::bitset<8> trans =  std::bitset<8>(ipaddr_src);
	std::string trans_string = trans.to_string();
	IPstring = IPstring + trans_string;
	}

    unsigned long long s = bitset<32>(IPstring).to_ullong();
    std::cout << "match:" << stringIP << "," << IPstring << "," << s << std::endl;

    thrust::host_vector<unsigned long long> h_timestamp(N);
    thrust::host_vector<long> h_sourceIP(N);

    thrust::host_vector<long> h_IP_to_match(N);
  
    const string csv_file = std::string(argv[1]); 
    vector<vector<string>> data; 

    Csv objCsv(csv_file);
    if (!objCsv.getCsv(data)) {
     std::cout << "read ERROR" << std::endl;
     return 1;
    }

    /* hard coded */
    for (int row = 0; row < data.size(); row++) {
      vector<string> rec = data[row]; 
      h_timestamp[row] = stoll(rec[0]);
      h_sourceIP[row] = stol(rec[1]);
      // h_IP_to_match[row] = 2639437048;
      h_IP_to_match[row] = s;
    }

    // thrust::host_vector<long long> h_timestamp(N);
    thrust::host_vector<unsigned long long> h_out(N);
    thrust::host_vector<unsigned long long> h_out_2(N);
    
    for(int i=0; i < N; i++)
    {
	h_out[i] = (h_timestamp[i] - start_time) / INTVL;
    }

    for(int i=0; i < N; i++)
    {
	h_out_2[i] = (h_out[i] * INTVL) + start_time;
    }

    /* check */
    /*
    for(int i=0; i < 5; i++)
    	    std::cout << h_timestamp[i] << "," << h_out_2[i] << std::endl;
    */

    thrust::device_vector<long long> d_timestamp(N);
    thrust::device_vector<long long> d_out(N);
    thrust::device_vector<long long> d_out_2(N);

    thrust::copy(h_timestamp.begin(), h_timestamp.end(), d_timestamp.begin());
    thrust::copy(h_out.begin(), h_out.end(), d_out.begin());
    thrust::copy(h_out_2.begin(), h_out_2.end(), d_out_2.begin());

    thrust::device_vector<long> d_IP_to_match(N);
    thrust::device_vector<long> d_sourceIP(N);

    thrust::copy(h_IP_to_match.begin(), h_IP_to_match.end(), d_IP_to_match.begin());
    thrust::copy(h_sourceIP.begin(), h_sourceIP.end(), d_sourceIP.begin());
    
    thrust::device_vector<unsigned long> dev_c(N);
    thrust::transform(begin(d_sourceIP), end(d_sourceIP), // dev_a for input
                      begin(d_IP_to_match),               // dev_b for input
		      begin(dev_c),                       // dev_c for output
		      [] __device__ (long x, long y) -> unsigned long { return x - y; });

    thrust::device_vector<long long> dev_c_2(N);
    thrust::copy(dev_c.begin(), dev_c.end(), dev_c_2.begin());

    thrust::sort_by_key(dev_c.begin(), dev_c.end(), d_sourceIP.begin());
    thrust::sort_by_key(dev_c_2.begin(), dev_c_2.end(), d_out_2.begin());

    /* check */
    for(int i=0; i < 10; i++)
    	    std::cout << dev_c[i] << "," << d_sourceIP[i] << "," << d_out_2[i] << std::endl;

    int N_count = thrust::count_if(dev_c.begin(), dev_c.end(), is_eq_zero());
    thrust::device_vector<long> d_b(N_count);
    thrust::copy_if(dev_c.begin(), dev_c.end(), d_b.begin(), is_eq_zero());
    std::cout << d_b.size() << std::endl;

    std::cout << endl;

    thrust::device_vector<long long> d_timestamp_2(d_b.size());
    thrust::copy_n(thrust::device, d_out_2.begin(), d_b.size(), d_timestamp_2.begin());

    /*
    for(int i=0; i < d_b.size(); i++)
    	    std::cout << dev_c[i] << "," << d_timestamp_2[i] << std::endl;
    */

    thrust::device_vector<int> uni_vect(d_b.size(), 1); 

    thrust::device_vector<long long> dkey_out(d_b.size(),0);
    thrust::device_vector<int> dvalue_out(d_b.size(),0);

    thrust::sort(d_timestamp_2.begin(), d_timestamp_2.end());
    auto new_end = thrust::reduce_by_key(d_timestamp_2.begin(),d_timestamp_2.end(),uni_vect.begin(),
					 dkey_out.begin(),dvalue_out.begin());

    int new_size = new_end.first - dkey_out.begin();

    for(long i=0; i <10; i++)
    	     std::cout << dkey_out[i] << "," << dvalue_out[i] << endl;
	     
    /*
    cout << "writing file..." << endl;
    std::remove("tmp");
    ofstream outputfile("tmp"); 

    bitset<32> bs(d_sourceIP[0]);
    
    string bs1 = bs.to_string().substr(0,8);
    int bi1 =  bitset<8>(bs1).to_ulong();
    
    string bs2 = bs.to_string().substr(8,8);
    int bi2 =  bitset<8>(bs2).to_ulong();

    string bs3 = bs.to_string().substr(16,8);
    int bi3 =  bitset<8>(bs3).to_ulong();

    string bs4 = bs.to_string().substr(24,8);
    int bi4 =  bitset<8>(bs4).to_ulong();

    string sourceIP = to_string(bi1) + "." + to_string(bi2) + "." + to_string(bi3) + "." + to_string(bi4);

    for(int i=0; i < d_b.size(); i++)
       	    outputfile << sourceIP << "," << d_out_2[i] << std::endl;

    outputfile.close();
    */

    return EXIT_SUCCESS;
}
