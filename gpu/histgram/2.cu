#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>

#include <hip/hip_runtime.h>
#include <stdio.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/sequence.h>

#include "csv.hpp"
#include "timer.h"
using namespace std;

struct is_eq_zero {
    __host__ __device__ bool operator()(long a) const {
                return (a==0);
		}
};

int main(int argc, char **argv)
{
    int N = atoi(argv[2]);
    int INTVL = atoi(argv[3]);

    unsigned int t, travdirtime; 

    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("%s starting reduction at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    hipSetDevice(dev);

    unsigned long long start_time = 20190114000000000;

    thrust::host_vector<unsigned long long> h_timestamp(N);
    thrust::host_vector<long> h_sourceIP(N);

    thrust::host_vector<long> h_IP_to_match(N);
  
    const string csv_file = std::string(argv[1]); 
    vector<vector<string>> data; 

    Csv objCsv(csv_file);
    if (!objCsv.getCsv(data)) {
     std::cout << "read ERROR" << std::endl;
     return 1;
    }

    for (int row = 0; row < data.size(); row++) {
      vector<string> rec = data[row]; 
      h_timestamp[row] = stoll(rec[0]);
      h_sourceIP[row] = stol(rec[1]);
      h_IP_to_match[row] = 2639437048;
    }

    // thrust::host_vector<long long> h_timestamp(N);
    thrust::host_vector<unsigned long long> h_out(N);
    thrust::host_vector<unsigned long long> h_out_2(N);
    
    for(int i=0; i < N; i++)
    {
	h_out[i] = (h_timestamp[i] - start_time) / INTVL;
    }

    for(int i=0; i < N; i++)
    {
	h_out_2[i] = (h_out[i] * INTVL) + start_time;
    }

    for(int i=0; i < 5; i++)
    	    std::cout << h_timestamp[i] << "," << h_out_2[i] << std::endl;

    thrust::device_vector<long long> d_timestamp(N);
    thrust::device_vector<long long> d_out(N);
    thrust::device_vector<long long> d_out_2(N);

    thrust::copy(h_timestamp.begin(), h_timestamp.end(), d_timestamp.begin());
    thrust::copy(h_out.begin(), h_out.end(), d_out.begin());
    thrust::copy(h_out_2.begin(), h_out_2.end(), d_out_2.begin());

    thrust::device_vector<long> d_IP_to_match(N);
    thrust::device_vector<long> d_sourceIP(N);

    thrust::copy(h_IP_to_match.begin(), h_IP_to_match.end(), d_IP_to_match.begin());
    thrust::copy(h_sourceIP.begin(), h_sourceIP.end(), d_sourceIP.begin());
    
    thrust::device_vector<unsigned long> dev_c(N);
    thrust::transform(begin(d_sourceIP), end(d_sourceIP), // dev_a for input
                      begin(d_IP_to_match),               // dev_b for input
		      begin(dev_c),                       // dev_c for output
		      [] __device__ (long x, long y) -> unsigned long { return x - y; });

    thrust::device_vector<long long> dev_c_2(N);
    thrust::copy(dev_c.begin(), dev_c.end(), dev_c_2.begin());

    thrust::sort_by_key(dev_c.begin(), dev_c.end(), d_sourceIP.begin());
    thrust::sort_by_key(dev_c_2.begin(), dev_c_2.end(), d_out_2.begin());

    for(int i=0; i < 10; i++)
    	    std::cout << dev_c[i] << "," << d_sourceIP[i] << "," << d_out_2[i] << std::endl;

    int N_count = thrust::count_if(dev_c.begin(), dev_c.end(), is_eq_zero());
    thrust::device_vector<long> d_b(N_count);
    thrust::copy_if(dev_c.begin(), dev_c.end(), d_b.begin(), is_eq_zero());
    std::cout << d_b.size() << std::endl;

    std::cout << endl;

    cout << "writing file..." << endl;
    std::remove("tmp");
    ofstream outputfile("tmp"); 
    for(int i=0; i < d_b.size(); i++)
    	    outputfile << dev_c[i] << "," << d_sourceIP[i] << "," << d_out_2[i] << std::endl;
    outputfile.close();

    /*
    cout << "writing file..." << endl;
    std::remove("tmp");
    ofstream outputfile("tmp"); 

    for(int i=0; i < N; i++)
    {
	if(dev_c[i] == 0)
		outputfile << d_out_2[i] << "," << d_sourceIP[i] << "," << d_IP_to_match[i] << "," << dev_c[i] << endl;
    }		      

    outputfile.close();
    */
    
    /*
    int N_count = thrust::count_if(dev_c.begin(), dev_c.end(), is_smaller_than());
    thrust::device_vector<long> d_b(N_count);
    thrust::copy_if(dev_c.begin(), dev_c.end(), d_b.begin(), is_smaller_than());
    std::cout << d_b.size() << std::endl;
    */

    /*
    size_t bytes = size * sizeof(unsigned long long);
    unsigned long long *d_tmp = NULL;

    hipMalloc((unsigned long long **)&d_tmp, bytes);
    hipMemcpy(d_tmp, h_tmp, bytes, hipMemcpyHostToDevice);
    */

    /*
    cout << "writing file..." << endl;
    std::remove("tmp");
    ofstream outputfile("tmp"); 

    for(int i=0; i < N; i++)
    {
	d_out_2[i] = (d_out[i] * 5) + start_time;
	outputfile << d_out_2[i] <<"," << h_sourceIP[i] << std::endl;	
    }
    outputfile.close();
    */

    /*
    cout << "writing file..." << endl;
    std::remove("tmp");
    ofstream outputfile("tmp"); 

    start_timer(&t); 
    for(int i = 0; i < 10; i++)
        std::cout << h_idata[i] << "," << h_odata[i] << "," << h_odata_2[i] << std::endl;

    outputfile.close();

    travdirtime = stop_timer(&t);
    print_timer(travdirtime);
    */

    // reset device
    //　hipDeviceReset();

    return EXIT_SUCCESS;
}
