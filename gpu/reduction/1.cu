#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

int recursiveReduceWithStrideOnCPU(int *data, int const size)
{
    if (size == 1) return data[0];

    int const stride = size / 2;
    
    for (int i = 0; i < stride; i++)
    {
        data[i] += data[i + stride];
    }
    return recursiveReduceWithStrideOnCPU(data, stride);
}

__global__ void reduceNeighbored (int *g_idata, int *g_odata, unsigned int n)
{
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int *idata = g_idata + blockIdx.x * blockDim.x;

    if (idx >= n) return;
    
    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        if ((tid % (2 * stride)) == 0)
        {
            idata[tid] += idata[tid + stride];
        }

        __syncthreads();
    }

    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceneighboredwithoutIF (int *g_idata, int *g_odata,
                                      unsigned int n)
{
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int *idata = g_idata + blockIdx.x * blockDim.x;

    if(idx >= n) return;

    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        int index = 2 * stride * tid;

        if (index < blockDim.x)
        {
            idata[index] += idata[index + stride];
        }
        __syncthreads();
    }
    
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

int main(int argc, char **argv)
{
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s starting reduction at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    bool bResult = false;

    int size = 1 << 24; 
    printf("    with array size %d  ", size);

    // execution configuration
    int blocksize = 512;

    if(argc > 1)
    {
        blocksize = atoi(argv[1]);  
    }

    dim3 block (blocksize, 1);
    dim3 grid  ((size + block.x - 1) / block.x, 1);
    printf("grid %d block %d\n", grid.x, block.x);

    size_t bytes = size * sizeof(int);
    int *h_idata = (int *) malloc(bytes);
    int *h_odata = (int *) malloc(grid.x * sizeof(int));
    int *tmp     = (int *) malloc(bytes);

    for (int i = 0; i < size; i++)
    {
        h_idata[i] = (int)( rand() & 0xFF );
    }

    memcpy (tmp, h_idata, bytes);

    double iStart, iElaps;
    int gpu_sum = 0;

    int *d_idata = NULL;
    int *d_odata = NULL;
    CHECK(hipMalloc((void **) &d_idata, bytes));
    CHECK(hipMalloc((void **) &d_odata, grid.x * sizeof(int)));

    iStart = seconds();
    int cpu_sum = recursiveReduceWithStrideOnCPU (tmp, size);
    iElaps = seconds() - iStart;
    printf("cpu reduce      elapsed %f sec cpu_sum: %d\n", iElaps, cpu_sum);

    // kernel 1: reduceNeighbored
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = seconds();
    reduceNeighbored<<<grid, block>>>(d_idata, d_odata, size);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];

    printf("gpu Neighbored  elapsed %f sec gpu_sum: %d <<<grid %d block "
           "%d>>>\n", iElaps, gpu_sum, grid.x, block.x);

    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = seconds();
    reduceneighboredwithoutIF<<<grid, block>>>(d_idata, d_odata, size);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];

    printf("gpu Neighbored2 elapsed %f sec gpu_sum: %d <<<grid %d block "
           "%d>>>\n", iElaps, gpu_sum, grid.x, block.x);

    CHECK(hipFree(d_idata));
    CHECK(hipFree(d_odata));
    CHECK(hipDeviceReset());

    bResult = (gpu_sum == cpu_sum);
    if(!bResult) printf("Test failed!\n");

    return EXIT_SUCCESS;
}
