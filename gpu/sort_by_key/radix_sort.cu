#include "hip/hip_runtime.h"
// -*- compile-command: "nvcc -D THRUST_SORT_TYPE=uint32_t -arch sm_50 -o sort sort_32.cu"; -*-

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>

//
//
//

#include <stdbool.h>

static
void
cuda_assert(const hipError_t code, const char* const file, const int line, const bool abort)
{
  if (code != hipSuccess)
    {
      fprintf(stderr,"cuda_assert: %s %s %d\n",hipGetErrorString(code),file,line);

      if (abort)
        {
          hipDeviceReset();
          exit(code);
        }
    }
}

#define cuda(...) { cuda_assert((cuda##__VA_ARGS__), __FILE__, __LINE__, true); }

//
//
//

#ifndef THRUST_SORT_TYPE
#define THRUST_SORT_TYPE   uint64_t
#endif

#define THRUST_SORT_WARMUP 100
#define THRUST_SORT_BENCH  100

//
//
//

static
void
sort(thrust::device_vector<THRUST_SORT_TYPE>& d_vec,
     hipEvent_t                              start,
     hipEvent_t                              end,
     float                                  * min_ms,
     float                                  * max_ms,
     float                                  * elapsed_ms)
{
  cuda(EventRecord(start,0));

  thrust::sort(d_vec.begin(), d_vec.end());

  cuda(EventRecord(end,0));
  cuda(EventSynchronize(end));

  float t_ms;
  cuda(EventElapsedTime(&t_ms,start,end));

  *min_ms      = min(*min_ms,t_ms);
  *max_ms      = max(*max_ms,t_ms);
  *elapsed_ms += t_ms;
}

//
//
//

static
void
bench(const struct hipDeviceProp_t* const props, const uint32_t count)
{
  thrust::host_vector<THRUST_SORT_TYPE> h_vec(count);

  // random fill
  std::generate(h_vec.begin(), h_vec.end(), rand);

  // transfer data to the device
  thrust::device_vector<THRUST_SORT_TYPE> d_vec = h_vec;

  hipEvent_t start, end;
  cuda(EventCreate(&start));
  cuda(EventCreate(&end));

  float min_ms     = FLT_MAX;
  float max_ms     = 0.0f;
  float elapsed_ms = 0.0f;

  for (int ii=0; ii<THRUST_SORT_WARMUP; ii++)
    sort(d_vec,start,end,&min_ms,&max_ms,&elapsed_ms);

  min_ms     = FLT_MAX;
  max_ms     = 0.0f;
  elapsed_ms = 0.0f;

  for (int ii=0; ii<THRUST_SORT_BENCH; ii++)
    sort(d_vec,start,end,&min_ms,&max_ms,&elapsed_ms);

  cuda(EventDestroy(start));
  cuda(EventDestroy(end));

  //
  //
  //
#define STRINGIFY2(s) #s
#define STRINGIFY(s)  STRINGIFY2(s)

  fprintf(stdout,"%s, %u, %s, %u, %u, %.3f, %.3f, %.3f, %.3f, %.3f, %.3f\n",
          props->name,
          props->multiProcessorCount,
          STRINGIFY(THRUST_SORT_TYPE),
          count,
          THRUST_SORT_BENCH,
          elapsed_ms,
          (double)elapsed_ms / THRUST_SORT_BENCH,
          (double)min_ms,
          (double)max_ms,
          (double)(THRUST_SORT_BENCH * count) / (1000.0 * elapsed_ms),
          (double)count                       / (1000.0 * min_ms));
}

//
//
//

int
main(int argc, char** argv)
{
  const int32_t device = (argc == 1) ? 0 : atoi(argv[1]);

  struct hipDeviceProp_t props;
  cuda(GetDeviceProperties(&props,device));

  printf("%s (%2d)\n",props.name,props.multiProcessorCount);

  cuda(SetDevice(device));

  //
  //
  //
  const uint32_t count_lo   = argc <= 2 ? 2048   : strtoul(argv[2],NULL,0);
  // const uint32_t count_hi   = argc <= 3 ? 262144 : strtoul(argv[3],NULL,0);
  // const uint32_t count_hi   = argc <= 3 ? 1024 << 21 : strtoul(argv[3],NULL,0);
  const uint32_t count_hi   = argc <= 3 ? 100000000 : strtoul(argv[3],NULL,0);
  const uint32_t count_step = argc <= 4 ? 2048   : strtoul(argv[4],NULL,0);

  //
  // LABELS
  //
  fprintf(stdout,
          "Device, "
          "Multiprocessors, "
          "Type, "
          "Keys, "
          "Trials, "
          "Total Msecs, "
          "Avg. Msecs, "
          "Min Msecs, "
          "Max Msecs, "
          "Avg. Mkeys/s, "
          "Max. Mkeys/s\n");

  //
  // SORT
  //
  for (uint32_t count=count_lo; count<=count_hi; count+=count_step)
    bench(&props,count);

  //
  // RESET
  //
  cuda(DeviceReset());

  return 0;
}
