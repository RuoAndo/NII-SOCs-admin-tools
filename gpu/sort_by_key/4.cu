#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include "util.h"

int main(void)
{
  // generate random numbers serially
  thrust::host_vector<int> h_vec(1024 << 14);
  thrust::host_vector<int> h_vec_2(1024 << 14);
  std::generate(h_vec.begin(), h_vec.end(), rand);

  static int i;
  static int size;
  
  size = h_vec.size();

  std::cout << "size:" << size << std::endl;

  // transfer data to the device
  thrust::device_vector<int> d_vec = h_vec;

  for (i = 0; i < h_vec_2.size(); i++)
      h_vec_2[i] = i;

  thrust::device_vector<int> d_vec_2 = h_vec_2;
      
  // thrust::device_vector<int>::iterator iter = d_vec_2.begin();

  hipEvent_t     start, stop;
  HANDLE_ERROR( hipEventCreate( &start ) );
  HANDLE_ERROR( hipEventCreate( &stop ) );
  
  HANDLE_ERROR( hipEventRecord( start, 0 ) );

  thrust::sort_by_key(d_vec_2.begin(), d_vec_2.end(), d_vec.begin());
  
  HANDLE_ERROR( hipEventRecord( stop, 0 ) );
  HANDLE_ERROR( hipEventSynchronize( stop ) );
  float   elapsedTime;
  HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );
  printf( "Time to sort_by_key :  %3.1f ms\n", elapsedTime );

  /*
  for(int i = 0; i < d_vec.size(); i++)
  	   std::cout << d_vec[i] << "," << d_vec_2[i] << std::endl;
  */
 
  return 0;
}
