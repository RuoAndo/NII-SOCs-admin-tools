#if __linux__ && defined(__INTEL_COMPILER)
#define __sync_fetch_and_add(ptr,addend) _InterlockedExchangeAdd(const_cast<void*>(reinterpret_cast<volatile void*>(ptr)), addend)
#endif
#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>

#include <sys/socket.h>
#include <netinet/in.h>
#include <arpa/inet.h>

#include "csv.hpp"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include "util.h"

using namespace std;

std::vector<std::string> split_string_2(std::string str, char del) {
  int first = 0;
  int last = str.find_first_of(del);

  std::vector<std::string> result;

  while (first < str.size()) {
    std::string subStr(str, first, last - first);

    result.push_back(subStr);

    first = last + 1;
    last = str.find_first_of(del, first);

    if (last == std::string::npos) {
      last = str.size();
    }
  }

  return result;
}

int main( int argc, char* argv[] ) {

    int N = atoi(argv[3]);  

    thrust::host_vector<long> h_vec_1_1(N);
    thrust::host_vector<long> h_vec_2_1(N);
    thrust::host_vector<long> h_vec_1_2(N);
    thrust::host_vector<long> h_vec_2_2(N);

    int ngpus;
    printf("> starting %s", argv[0]);

    hipGetDeviceCount(&ngpus);
    printf(" CUDA-capable devices: %i\n", 2);
    hipStream_t *stream = (hipStream_t *)malloc(sizeof(hipStream_t) * 2);
      
    vector<vector<string>> data;
    const string csv_file_1 = std::string(argv[1]);

    cout << "reading file 1..." << endl;
    Csv objCsv_1(csv_file_1);
    if (!objCsv_1.getCsv(data)) {
    cout << "read ERROR" << endl;
    return 1;
    }
	 
    for (int row = 0; row < data.size(); row++) {
        vector<string> rec = data[row];
	std::string timestamp = rec[0];

	for(size_t c = timestamp.find_first_of("\""); c != string::npos; c = c = timestamp.find_first_of("\"")){
	      timestamp.erase(c,1);
	     }
        for(size_t c = timestamp.find_first_of("."); c != string::npos; c = c = timestamp.find_first_of(".")){
	      timestamp.erase(c,1);
	     }
	for(size_t c = timestamp.find_first_of(" "); c != string::npos; c = c = timestamp.find_first_of(" ")){
	      timestamp.erase(c,1);
	     }
	for(size_t c = timestamp.find_first_of(":"); c != string::npos; c = c = timestamp.find_first_of(":")){
	      timestamp.erase(c,1);
	     }
	for(size_t c = timestamp.find_first_of("/"); c != string::npos; c = c = timestamp.find_first_of("/")){
	      timestamp.erase(c,1);
	     }

	h_vec_1_1[row] = std::atol(timestamp.c_str());
	h_vec_2_1[row] = 1;
	    
   }

   cout << "reading file 2..." << endl;
   const string csv_file_2 = std::string(argv[2]);
   Csv objCsv_2(csv_file_2);
   if (!objCsv_2.getCsv(data)) {
   cout << "read ERROR" << endl;
   return 1;
   }

   for (int row = 0; row < data.size(); row++) {
	    vector<string> rec = data[row]; 
	    std::string timestamp = rec[0];

	    for(size_t c = timestamp.find_first_of("\""); c != string::npos; c = c = timestamp.find_first_of("\"")){
	      timestamp.erase(c,1);
	    }
	    for(size_t c = timestamp.find_first_of("."); c != string::npos; c = c = timestamp.find_first_of(".")){
	      timestamp.erase(c,1);
	    }
	    for(size_t c = timestamp.find_first_of(" "); c != string::npos; c = c = timestamp.find_first_of(" ")){
	      timestamp.erase(c,1);
	    }
	    for(size_t c = timestamp.find_first_of(":"); c != string::npos; c = c = timestamp.find_first_of(":")){
	      timestamp.erase(c,1);
	    }
	    for(size_t c = timestamp.find_first_of("/"); c != string::npos; c = c = timestamp.find_first_of("/")){
	      timestamp.erase(c,1);
	    }

	    h_vec_1_2[row] = std::atol(timestamp.c_str());
	    h_vec_2_2[row] = 1;	    
   } 

   for(int i=0; i< 5; i++)
   {
	 cout << "h_vec_1_1" << "," << h_vec_1_1[i] << endl;
	 cout << "h_vec_1_2" << "," << h_vec_1_2[i] << endl;
   }

   cout << "creating stream..." << endl;
   for (int i = 0; i < 1; i++)
    	hipStreamCreate(&stream[i]);

	/*
   thrust::device_vector<long> key_in_1 = h_vec_1_1;
   thrust::device_vector<long> value_in_1 = h_vec_2_1;

   thrust::device_vector<long> key_in_2 = h_vec_1_2;
   thrust::device_vector<long> value_in_2 = h_vec_2_2; 

   thrust::sort(thrust::cuda::par.on(stream[0]), key_in_1.begin(), key_in_1.end());
   thrust::sort(thrust::cuda::par.on(stream[1]), key_in_2.begin(), key_in_2.end());

   for(int i=0; i< 5; i++)
   {
	 cout << "key_in_1" << "," << key_in_1[i] << endl;
	 cout << "value_in_1" << "," << value_in_1[i] << endl;
   }

   cout << "sync 1" << endl;
	
   hipStreamSynchronize(stream[0]);
   hipStreamSynchronize(stream[1]);
   hipStreamDestroy(stream[0]);
   hipStreamDestroy(stream[1]);

   cout << "creating stream 2..." << endl;
   for (int i = 0; i < 2; i++)
    	hipStreamCreate(&stream[i]);

   thrust::device_vector<long> dkey_out_1(N,0);
   thrust::device_vector<long> dvalue_out_1(N,0);
   thrust::device_vector<long> dkey_out_2(N,0);
   thrust::device_vector<long> dvalue_out_2(N,0);
	
   auto new_end_1 = thrust::reduce_by_key(thrust::cuda::par.on(stream[0]),
						key_in_1.begin(),
		    		             	key_in_1.end(),
					     	value_in_1.begin(),
					     	dkey_out_1.begin(),
	  				     	dvalue_out_1.begin());

   auto new_end_2 = thrust::reduce_by_key(thrust::cuda::par.on(stream[1]),
						key_in_2.begin(),
		    		             	key_in_2.end(),
					     	value_in_2.begin(),
					     	dkey_out_2.begin(),
	  				     	dvalue_out_2.begin());
   hipStreamSynchronize(stream[0]);
   hipStreamSynchronize(stream[1]);
   hipStreamDestroy(stream[0]);
   hipStreamDestroy(stream[1]);

   long new_size_1 = new_end_1.first - dkey_out_1.begin();
   long new_size_2 = new_end_2.first - dkey_out_2.begin();

   for(int i=0; i <5; i++)
   {
   	cout << dkey_out_1[i] << "," << dvalue_out_1[i] << endl;
   }

   for(int i=0; i <5; i++)
   {
	cout << dkey_out_2[i] << "," << dvalue_out_2[i] << endl;
   }
	*/

        return 0;	
}
