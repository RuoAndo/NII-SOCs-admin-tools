#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void kernel_for_sum(float *g_data, float value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    g_data[idx] = g_data[idx] + value;
}

int inspectResult(float *data, const int n, const float x)
{
    for (int i = 0; i < n; i++)
    {
        if (data[i] != x)
        {
            printf("Error! data[%d] = %f, ref = %f\n", i, data[i], x);
            return 0;
        }
    }

    return 1;
}

int main(int argc, char *argv[])
{
    int devID = 0;
    hipDeviceProp_t deviceProps;
    CHECK(hipGetDeviceProperties(&deviceProps, devID));
    printf("> %s running on", argv[0]);
    printf(" CUDA device [%s]\n", deviceProps.name);

    int num = 1 << 24;
    int nbytes = num * sizeof(int);
    float value = 10.0f;

    // allocate host memory
    float *h_a = 0;
    CHECK(hipHostMalloc((void **)&h_a, nbytes));
    memset(h_a, 0, nbytes);

    float *d_a = 0;
    CHECK(hipMalloc((void **)&d_a, nbytes));
    CHECK(hipMemset(d_a, 255, nbytes));

    dim3 block = dim3(512);
    dim3 grid  = dim3((num + block.x - 1) / block.x);

    hipEvent_t stop;
    CHECK(hipEventCreate(&stop));

    CHECK(hipMemcpyAsync(d_a, h_a, nbytes, hipMemcpyHostToDevice));
    kernel_for_sum<<<grid, block>>>(d_a, value);
    CHECK(hipMemcpyAsync(h_a, d_a, nbytes, hipMemcpyDeviceToHost));
    CHECK(hipEventRecord(stop, 0));

    unsigned long int counter = 0;

    // polling
    while (hipEventQuery(stop) == hipErrorNotReady) {
        counter++;
    }

    printf("CPU executed %lu iterations while waiting for GPU to finish\n", counter);

    bool bFinalResults = (bool) inspectResult(h_a, num, value);

    CHECK(hipEventDestroy(stop));
    CHECK(hipHostFree(h_a));
    CHECK(hipFree(d_a));

    CHECK(hipDeviceReset());

    exit(bFinalResults ? EXIT_SUCCESS : EXIT_FAILURE);
}
