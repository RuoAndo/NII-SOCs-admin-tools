#if __linux__ && defined(__INTEL_COMPILER)
#define __sync_fetch_and_add(ptr,addend) _InterlockedExchangeAdd(const_cast<void*>(reinterpret_cast<volatile void*>(ptr)), addend)
#endif
#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>

#include <sys/socket.h>
#include <netinet/in.h>
#include <arpa/inet.h>

#include "tbb/concurrent_hash_map.h"
#include "tbb/blocked_range.h"
#include "tbb/parallel_for.h"
#include "tbb/tick_count.h"
#include "tbb/task_scheduler_init.h"
//  #include "tbb/tbb_allocator.hz"
#include "utility.h"

#include "csv.hpp"
typedef std::basic_string<char,std::char_traits<char>,tbb::tbb_allocator<char> > MyString;

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include "util.h"
#include "timer.h"

using namespace tbb;
using namespace std;

static bool verbose = false;
static bool silent = false;

// const int size_factor = 2;
// typedef concurrent_hash_map<MyString,int> StringTable;
typedef concurrent_hash_map<MyString,std::vector<string>> StringTable;
std::vector<string> v_pair;
std::vector<string> v_count;
static MyString* Data;

std::vector<std::string> split_string_2(std::string str, char del) {
  int first = 0;
  int last = str.find_first_of(del);

  std::vector<std::string> result;

  while (first < str.size()) {
    std::string subStr(str, first, last - first);

    result.push_back(subStr);

    first = last + 1;
    last = str.find_first_of(del, first);

    if (last == std::string::npos) {
      last = str.size();
    }
  }

  return result;
}

int main( int argc, char* argv[] ) {

  // int counter = 0;
  int N = atoi(argv[2]);  
  unsigned int t, travdirtime; 

  thrust::host_vector<long> h_vec_1_0(N);
  thrust::host_vector<long> h_vec_2_0(N);

  thrust::host_vector<long> h_vec_1_1(N);
  thrust::host_vector<long> h_vec_2_1(N);
  thrust::host_vector<long> h_vec_1_2(N);
  thrust::host_vector<long> h_vec_2_2(N);   

    try {
        tbb::tick_count mainStartTime = tbb::tick_count::now();
        srand(2);
        utility::thread_number_range threads(tbb::task_scheduler_init::default_num_threads,0);
        if ( silent ) verbose = false;

        Data = new MyString[N];

	const string csv_file = std::string(argv[1]); 
	vector<vector<string>> data; 

	try {
	  Csv objCsv(csv_file);
	  if (!objCsv.getCsv(data)) {
	    cout << "read ERROR" << endl;
	    return 1;
	  }

	 std::remove("tmp0");
	 ofstream outputfile("tmp0");
	 
      	  for (int row = 0; row < data.size(); row++) {
	    vector<string> rec = data[row]; 
	    std::string timestamp = rec[0];

	    for(size_t c = timestamp.find_first_of("\""); c != string::npos; c = c = timestamp.find_first_of("\"")){
	      timestamp.erase(c,1);
	    }
	    for(size_t c = timestamp.find_first_of("."); c != string::npos; c = c = timestamp.find_first_of(".")){
	      timestamp.erase(c,1);
	    }
	    for(size_t c = timestamp.find_first_of(" "); c != string::npos; c = c = timestamp.find_first_of(" ")){
	      timestamp.erase(c,1);
	    }
	    for(size_t c = timestamp.find_first_of(":"); c != string::npos; c = c = timestamp.find_first_of(":")){
	      timestamp.erase(c,1);
	    }
	    for(size_t c = timestamp.find_first_of("/"); c != string::npos; c = c = timestamp.find_first_of("/")){
	      timestamp.erase(c,1);
	    }

	    h_vec_1_0.push_back(std::atol(timestamp.c_str()));
	    h_vec_2_0.push_back(1);

	    h_vec_1_1.push_back(std::atol(timestamp.c_str()));
	    h_vec_2_1.push_back(1);
	    h_vec_1_2.push_back(std::atol(timestamp.c_str()));
	    h_vec_2_2.push_back(1);

	    /*
	    if(row<(N/2))
	    {
		h_vec_1_1.push_back(std::atol(timestamp.c_str()));
	    	h_vec_2_1.push_back(1);
	    }
	    else
	    {
		h_vec_1_2.push_back(std::atol(timestamp.c_str()));
	   	h_vec_2_2.push_back(1);
	    }
	    */
	    
	  }

	  tbb::tick_count mainStartTime = tbb::tick_count::now();
	  start_timer(&t); 
	  thrust::device_vector<long> key_in_0 = h_vec_1_0;
	  thrust::device_vector<long> value_in_0 = h_vec_2_0;

	  thrust::sort(key_in_0.begin(), key_in_0.end());
	  thrust::device_vector<long> dkey_out_0(N,0);
	  thrust::device_vector<long> dvalue_out_0(N,0);

	  auto new_end_0 = thrust::reduce_by_key(key_in_0.begin(),
					       key_in_0.end(),
					       value_in_0.begin(),
					       dkey_out_0.begin(),
	  				       dvalue_out_0.begin());

	  long new_size_0 = new_end_0.first - dkey_out_0.begin();

	  travdirtime = stop_timer(&t);
          print_timer(travdirtime); 
	  utility::report_elapsed_time((tbb::tick_count::now() - mainStartTime).seconds());

	  for(long i=0; i <new_size_0; i++)
	    {
		outputfile << dkey_out_0[i] << "," << dvalue_out_0[i] << endl;
	    }

	  /* streams */

          // tbb::tick_count mainStartTime = tbb::tick_count::now();

	  /*
	  start_timer(&t); 
	  mainStartTime = tbb::tick_count::now();
	  hipStream_t *stream = (hipStream_t *)malloc(sizeof(hipStream_t) * 2);

	  thrust::device_vector<long> key_in_1 = h_vec_1_1;
	  thrust::device_vector<long> value_in_1 = h_vec_2_1;
	  thrust::device_vector<long> key_in_2 = h_vec_1_2;
	  thrust::device_vector<long> value_in_2 = h_vec_2_2; 

	  for (int i = 0; i < 2; i++)
              	   hipStreamCreate(&stream[i]);

	  thrust::sort(thrust::cuda::par.on(stream[0]), key_in_1.begin(), key_in_1.end());
	  thrust::sort(thrust::cuda::par.on(stream[1]), key_in_2.begin(), key_in_2.end());

	  for (int i = 0; i < 2; i++)
	           hipStreamSynchronize(stream[i]);
 
	  for (int i = 0; i < 2; i++)
              	   hipStreamDestroy(stream[i]);

	  thrust::device_vector<long> dkey_out_1(N,0);
	  thrust::device_vector<long> dvalue_out_1(N,0);
	  thrust::device_vector<long> dkey_out_2(N,0);
	  thrust::device_vector<long> dvalue_out_2(N,0);
	  
	  for (int i = 0; i < 2; i++)
              	   hipStreamCreate(&stream[i]);
	  
	  auto new_end_1 = thrust::reduce_by_key(thrust::cuda::par.on(stream[0]), key_in_1.begin(),
					       key_in_1.end(),
					       value_in_1.begin(),
					       dkey_out_1.begin(),
	  				       dvalue_out_1.begin());

	  auto new_end_2 = thrust::reduce_by_key(thrust::cuda::par.on(stream[1]), key_in_2.begin(),
					       key_in_2.end(),
					       value_in_2.begin(),
					       dkey_out_2.begin(),
	  				       dvalue_out_2.begin());

	  for (int i = 0; i < 2; i++)
	           hipStreamSynchronize(stream[i]);
 
	  for (int i = 0; i < 2; i++)
              	   hipStreamDestroy(stream[i]);

	  travdirtime = stop_timer(&t);
          print_timer(travdirtime); 
	  utility::report_elapsed_time((tbb::tick_count::now() - mainStartTime).seconds());

	  long new_size_1 = new_end_1.first - dkey_out_1.begin();
	  long new_size_2 = new_end_2.first - dkey_out_2.begin();

	  std::remove("tmp2");
	  ofstream outputfile2("tmp2");
	  for(long i=0; i <new_size_1; i++)
	    {
		outputfile2 << dkey_out_1[i] << "," << dvalue_out_2[i] << endl;
	    }
	  outputfile2.close();

	  std::remove("tmp3");
	  ofstream outputfile3("tmp3");
	  for(long i=0; i <new_size_2; i++)
	    {
		outputfile3 << dkey_out_2[i] << "," << dvalue_out_2[i] << endl;
	    }
	  outputfile3.close();
	  */
	}
	
	catch (...) {
	  cout << "EXCEPTION!" << endl;
	  return 1;
	}
	
        delete[] Data;
        // utility::report_elapsed_time((tbb::tick_count::now() - mainStartTime).seconds());
       
        return 0;
	
    } catch(std::exception& e) {
        std::cerr<<"error occurred. error text is :\"" <<e.what()<<"\"\n";
    }
}
