#include<iostream>
#include<algorithm>
#include<iomanip>
#include<time.h>
#include<thrust/host_vector.h>
#include<thrust/device_vector.h>
#include<thrust/sort.h>
#include <thrust/iterator/permutation_iterator.h>

#define N (8<<27)
#define M N/10

template<class T>
class plusOne{
public:
    __device__ __host__ T operator() (T a){
        return a+1;
    }
};

int f()
{
	srand(time(NULL));
	return rand() % 1000;
}

int main(){
    printf("size %d \n",N);
    
    srand(time(NULL));
    thrust::host_vector<int> source(N);
    std::generate(source.begin(),source.end(),rand);
    thrust::device_vector<int> dsource=source;

    thrust::host_vector<int> map(M);

    /*
    map[0] = 3;
    map[1] = 1;
    map[2] = 0;
    map[3] = 5;
    */
    
    std::generate(map.begin(),map.end(),f);
    thrust::device_vector<int> dmap=map;
    
    hipEvent_t start,stop;
    float elapsed;
    
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    int sum = thrust::reduce(thrust::make_permutation_iterator(dsource.begin(), dmap.begin()), thrust::make_permutation_iterator(dsource.begin(), dmap.end()));
    std::cout << "sum :" << sum << std::endl;
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed,start,stop);

    std::cout<<"gpu :"<<elapsed<<"ms"<<std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
