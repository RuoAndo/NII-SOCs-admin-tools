#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define N 300000
#define NSTREAM 4

__global__ void kernel_1()
{
    double sum = 0.0;

    for(int i = 0; i < N; i++)
    {
        sum = sum + tan(0.1) * tan(0.1);
    }
}

__global__ void kernel_2()
{
    double sum = 0.0;

    for(int i = 0; i < N; i++)
    {
        sum = sum + tan(0.1) * tan(0.1);
    }
}

__global__ void kernel_3()
{
    double sum = 0.0;

    for(int i = 0; i < N; i++)
    {
        sum = sum + tan(0.1) * tan(0.1);
    }
}

__global__ void kernel_4()
{
    double sum = 0.0;

    for(int i = 0; i < N; i++)
    {
        sum = sum + tan(0.1) * tan(0.1);
    }
}

int main(int argc, char **argv)
{
    int n_streams = NSTREAM;
    int isize = 1;
    int iblock = 1;
    int bigcase = 0;

    if (argc > 1) n_streams = atoi(argv[1]);

    if (argc > 2) bigcase = atoi(argv[2]);

    float elapsed_time;

    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("> Using Device %d: %s with num_streams=%d\n", dev, deviceProp.name, n_streams);
    CHECK(hipSetDevice(dev));

    if (deviceProp.major < 3 || (deviceProp.major == 3 && deviceProp.minor < 5))
    {
        if (deviceProp.concurrentKernels == 0)
        {
            printf("> GPU does not support concurrent kernel execution (SM 3.5 or higher required)\n");
            printf("> CUDA kernel runs will be serialized\n");
        }
        else
        {
            printf("> GPU does not support HyperQ\n");
            printf("> CUDA kernel runs will have limited concurrency\n");
        }
    }

    printf("> Compute Capability %d.%d hardware with %d multi-processors\n",
           deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

    hipStream_t *streams = (hipStream_t *) malloc(n_streams * sizeof(hipStream_t));

    for (int i = 0; i < n_streams; i++)
    {
        CHECK(hipStreamCreate(&(streams[i])));
    }

    if (bigcase == 1)
    {
        iblock = 512;
        isize = 1 << 12;
    }

    dim3 block(iblock);
    dim3 grid(isize / iblock);

    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
    CHECK(hipEventRecord(start, 0));

    // breadth first 
    for (int i = 0; i < n_streams; i++)
        kernel_1<<<grid, block, 0, streams[i]>>>();

    for (int i = 0; i < n_streams; i++)
        kernel_2<<<grid, block, 0, streams[i]>>>();

    for (int i = 0; i < n_streams; i++)
        kernel_3<<<grid, block, 0, streams[i]>>>();

    for (int i = 0; i < n_streams; i++)
        kernel_4<<<grid, block, 0, streams[i]>>>();

    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));

    CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("Measured time for parallel execution = %f \n", elapsed_time);

    for (int i = 0; i < n_streams; i++)
    {
        CHECK(hipStreamDestroy(streams[i]));
    }

    free(streams);

    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));

    CHECK(hipDeviceReset());

    return 0;
}
