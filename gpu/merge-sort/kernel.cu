
#ifndef __CUDACC__
#define __CUDACC__
#endif
#include "hip/hip_runtime.h"


#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <iostream>
#include <cstdio>

#define N 1024
#define THREADS 32
#define BLOCKS 32


hipError_t sortCuda(unsigned int *data, const int size);

__global__ void sort(unsigned int *data) {
        int i = 2;
        __shared__ int temp[THREADS];

        while (i <= THREADS) {
          if ((threadIdx.x % i) == 0) {
                int index1 = threadIdx.x + (blockIdx.x * blockDim.x);
				int targetIndex = threadIdx.x;
                int endIndex1 = index1 + i/2;
                int index2 = endIndex1;
                int endIndex2 = index2 + i/2;
                
                while (!((index1==endIndex1) && (index2==endIndex2))) {
                        if ((index1 == endIndex1) && (index2 < endIndex2))
							temp[targetIndex++] = data[index2++];
                        else if ((index2 == endIndex2) && (index1 < endIndex1))
							temp[targetIndex++] = data[index1++];
                        else if (data[index1] < data[index2])
							temp[targetIndex++] = data[index1++];
                        else
							temp[targetIndex++] = data[index2++];
                }
          }
		  __syncthreads();
          data[threadIdx.x + (blockIdx.x*blockDim.x)] = temp[threadIdx.x];
          __syncthreads();
          i *= 2;
        }
}

__global__ void merge(unsigned int *data, unsigned int *final, int sortedsize) {
		int index1 = blockIdx.x * 2 * sortedsize;
		int targetIndex = blockIdx.x * 2 * sortedsize;
        int endIndex1 = index1 + sortedsize;
        int index2 = endIndex1;
        int endIndex2 = index2 + sortedsize;
   
        while (!((index1==endIndex1) && (index2==endIndex2))) {
                if ((index1 == endIndex1) && (index2 < endIndex2))
					final[targetIndex++] = data[index2++];
                else if ((index2 == endIndex2) && (index1 < endIndex1))
					final[targetIndex++] = data[index1++];
                else if (data[index1] < data[index2])
					final[targetIndex++] = data[index1++];
                else
					final[targetIndex++] = data[index2++];
        }
}


void init_data(unsigned int *data, unsigned int nitems) {
  for (unsigned i = 0 ; i < nitems ; i++)
    data[i] = rand() % nitems ;
}

int main() {
	unsigned int *h_data = 0;

	std::cout << "Initializing data:" << std::endl;
	h_data =(unsigned int *)malloc( N*sizeof(unsigned int));
	init_data(h_data, N);
	//for(int i=0 ; i<N ; i++)
	//	std::cout << "Data [" << i << "]: " << h_data[i] << std::endl;

    hipError_t cudaStatus = sortCuda(h_data, N);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "sortCuda failed!");
        return 1;
    }
	
	std::cout << "Results after sorting:" << std::endl;
	for(int i=0 ; i<N ; i++)
		std::cout << "Data [" << i << "]: " << h_data[i] << std::endl;

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t sortCuda(unsigned int *data, const int size) {
    unsigned int *dev_data = 0;
	unsigned int *dev_final = 0;
    hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        // goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_data, size * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        // goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_final, size * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        // goto Error;
    }

    cudaStatus = hipMemcpy(dev_data, data, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        // goto Error;
    }

	std::cout << "Launching kernel on the GPU" << std::endl;
	//quicksort<<< 1, size >>>(dev_data);
	sort<<<BLOCKS,THREADS>>>(dev_data);
	int blocks = BLOCKS/2;
    int sortedsize = THREADS;
    while (blocks > 0) {
		merge<<<blocks,1>>>(dev_data, dev_final, sortedsize);
		hipMemcpy(dev_data, dev_final, N*sizeof(int), hipMemcpyDeviceToDevice);
		blocks /= 2;
		sortedsize *= 2;
	}

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "sortKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        // goto Error;
    }
	    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching sortKernel!\n", cudaStatus);
        // goto Error;
    }

    cudaStatus = hipMemcpy(data, dev_data, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        // goto Error;
    }

Error:
    hipFree(dev_data);
	hipFree(dev_final);
    
    return cudaStatus;
}
