
#include <hip/hip_runtime.h>
#include <iostream>
//#include <helper_cuda.h>
#include <sys/time.h>

// data[], size, threads, blocks, 
void mergesort(long*, long, dim3, dim3);
// A[]. B[], size, width, slices, nThreads
__global__ void gpu_mergesort(long*, long*, long, long, long, dim3*, dim3*);
__device__ void gpu_bottomUpMerge(long*, long*, long, long, long);

// profiling
int tm();

#define min(a, b) (a < b ? a : b)

int main(int argc, char** argv) {

    dim3 threadsPerBlock;
    dim3 blocksPerGrid;

    threadsPerBlock.x = 32;
    threadsPerBlock.y = 1;
    threadsPerBlock.z = 1;

    blocksPerGrid.x = 8;
    blocksPerGrid.y = 1;
    blocksPerGrid.z = 1;

    //
    // Parse argv
    //
    tm();
    
    long size=100000000;
    size_t nBytes = size * sizeof(long);
    long *data;
    data = (long *)malloc(nBytes);     

    time_t t;
    srand((unsigned) time(&t));
    for(int i=0; i< 10;i++)
    {
	data[i] = rand();     
    }

    std::cout << "sorting " << size << " numbers\n\n";

    // merge-sort the data
    mergesort(data, size, threadsPerBlock, blocksPerGrid);

    tm();

    //
    // Print out the list
    //
    
    /*
    for (int i = 0; i < size; i++) {
        std::cout << data[i] << '\n';
    } 
    std::cout << "print list to stdout: " << tm() << " microseconds\n";
    */

}

void mergesort(long* data, long size, dim3 threadsPerBlock, dim3 blocksPerGrid) {

    // Allocate two arrays on the GPU
    // we switch back and forth between them during the sort
    long* D_data;
    long* D_swp;
    dim3* D_threads;
    dim3* D_blocks;
    
    // Actually allocate the two arrays
    tm();
    hipMalloc((void**) &D_data, size * sizeof(long));
    hipMalloc((void**) &D_swp, size * sizeof(long));
        std::cout << "hipMalloc device lists: " << tm() << " microseconds\n";

    // Copy from our input list into the first array
    hipMemcpy(D_data, data, size * sizeof(long), hipMemcpyHostToDevice);
        std::cout << "hipMemcpy list to device: " << tm() << " microseconds\n";
 
    // Copy the thread / block info to the GPU as well
    hipMalloc((void**) &D_threads, sizeof(dim3));
    hipMalloc((void**) &D_blocks, sizeof(dim3));

        std::cout << "hipMalloc device thread data: " << tm() << " microseconds\n";
    hipMemcpy(D_threads, &threadsPerBlock, sizeof(dim3), hipMemcpyHostToDevice);
    hipMemcpy(D_blocks, &blocksPerGrid, sizeof(dim3), hipMemcpyHostToDevice);

        std::cout << "hipMemcpy thread data to device: " << tm() << " microseconds\n";

    long* A = D_data;
    long* B = D_swp;

    long nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
                    blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;
	
    // Slice up the list and give pieces of it to each thread, letting the pieces grow
    // bigger and bigger until the whole list is sorted
    for (int width = 2; width < (size << 1); width <<= 1) {
        long slices = size / ((nThreads) * width) + 1;

            std::cout << "mergeSort - width: " << width 
                      << ", slices: " << slices 
                      << ", nThreads: " << nThreads << '\n';
            tm();

        // Actually call the kernel
        gpu_mergesort<<<blocksPerGrid, threadsPerBlock>>>(A, B, size, width, slices, D_threads, D_blocks);

            std::cout << "call mergesort kernel: " << tm() << " microseconds\n";

        // Switch the input / output arrays instead of copying them around
        A = A == D_data ? D_swp : D_data;
        B = B == D_data ? D_swp : D_data;
    }

    // Get the list back from the GPU
    tm();
    hipMemcpy(data, A, size * sizeof(long), hipMemcpyDeviceToHost);
        std::cout << "hipMemcpy list back to host: " << tm() << " microseconds\n";
    
    
    // Free the GPU memory
    hipFree(A);
    hipFree(B);
        std::cout << "cudaFree: " << tm() << " microseconds\n";
}

// GPU helper function
// calculate the id of the current thread
__device__ unsigned int getIdx(dim3* threads, dim3* blocks) {
    int x;
    return threadIdx.x +
           threadIdx.y * (x  = threads->x) +
           threadIdx.z * (x *= threads->y) +
           blockIdx.x  * (x *= threads->z) +
           blockIdx.y  * (x *= blocks->z) +
           blockIdx.z  * (x *= blocks->y);
}

//
// Perform a full mergesort on our section of the data.
//
__global__ void gpu_mergesort(long* source, long* dest, long size, long width, long slices, dim3* threads, dim3* blocks) {
    unsigned int idx = getIdx(threads, blocks);
    long start = width*idx*slices, 
         middle, 
         end;

    for (long slice = 0; slice < slices; slice++) {
        if (start >= size)
            break;

        middle = min(start + (width >> 1), size);
        end = min(start + width, size);
        gpu_bottomUpMerge(source, dest, start, middle, end);
        start += width;
    }
}

//
// Finally, sort something
// gets called by gpu_mergesort() for each slice
//
__device__ void gpu_bottomUpMerge(long* source, long* dest, long start, long middle, long end) {
    long i = start;
    long j = middle;
    for (long k = start; k < end; k++) {
        if (i < middle && (j >= end || source[i] < source[j])) {
            dest[k] = source[i];
            i++;
        } else {
            dest[k] = source[j];
            j++;
        }
    }
}

// 
// Get the time (in microseconds) since the last call to tm();
// the first value returned by this must not be trusted
//
timeval tStart;
int tm() {
    timeval tEnd;
    gettimeofday(&tEnd, 0);
    int t = (tEnd.tv_sec - tStart.tv_sec) * 1000000 + tEnd.tv_usec - tStart.tv_usec;
    tStart = tEnd;
    return t;
}


