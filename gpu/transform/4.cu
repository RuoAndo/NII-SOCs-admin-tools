#include<iostream>
#include<algorithm>
#include<iomanip>
#include<time.h>
#include<thrust/host_vector.h>
#include<thrust/device_vector.h>
#include<thrust/sort.h>

#define N (8<<10)
#define C1 10000

template<class T>
class plusOne{
public:
    __device__ __host__ T operator() (T a){
        return a+1;
    }
};

int main(){
    srand(time(NULL));
    thrust::host_vector<int> hv(N);
    std::generate(hv.begin(),hv.end(),rand);
    thrust::device_vector<int> dv=hv;

    hipEvent_t start,stop;
    float elapsed;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    hipEventRecord(start,0);
    for(int c=0;c<C1;c++){
        thrust::transform(dv.begin(),dv.end(),dv.begin(),plusOne<int>());
    }
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed,start,stop);

    std::cout<<"gpu :"<<elapsed<<"ms ["<<std::setprecision(8)<<C1/elapsed<<"/ms]"<<std::endl;

    std::generate(hv.begin(),hv.end(),rand);
    dv=hv;

    hipEventRecord(start,0);
    for(int c=0;c<C1;c++){
        thrust::transform(hv.begin(),hv.end(),hv.begin(),plusOne<int>());
    }
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed,start,stop);

    std::cout<<"cpu :"<<elapsed<<"ms ["<<std::setprecision(8)<<C1/elapsed<<"/ms]"<<std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
