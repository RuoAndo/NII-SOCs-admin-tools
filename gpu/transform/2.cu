#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <iostream>

struct saxpy_functor
{
    const float a;

    saxpy_functor(float _a) : a(_a) {}

    __host__ __device__
        float operator()(const float& x, const float& y) const { 
            return a * x + y;
        }
};

void saxpy_fast(float A, thrust::device_vector<float>& X, thrust::device_vector<float>& Y)
{
    // Y <- A * X + Y
    thrust::transform(X.begin(), X.end(), Y.begin(), Y.begin(), saxpy_functor(A));
}

void saxpy_slow(float A, thrust::device_vector<float>& X, thrust::device_vector<float>& Y)
{
    thrust::device_vector<float> temp(X.size());
   
    // temp <- A
    thrust::fill(temp.begin(), temp.end(), A);
    
    // temp <- A * X
    thrust::transform(X.begin(), X.end(), temp.begin(), temp.begin(), thrust::multiplies<float>());

    // Y <- A * X + Y
    thrust::transform(temp.begin(), temp.end(), Y.begin(), Y.begin(), thrust::plus<float>());
}

int main(void)
{
    // allocate three device_vectors with 10 elements
        thrust::device_vector<int> X(10);
	thrust::device_vector<int> Y(10);
	thrust::device_vector<int> Z(10);

    // initialize X to 0,1,2,3, ....
        thrust::sequence(X.begin(), X.end());

    // compute Y = -X
        thrust::transform(X.begin(), X.end(), Y.begin(), thrust::negate<int>());

    // fill Z with twos
        thrust::fill(Z.begin(), Z.end(), 2);

    // compute Y = X mod 2
        thrust::transform(X.begin(), X.end(), Z.begin(), Y.begin(), thrust::modulus<int>());

    // replace all the ones in Y with tens
        thrust::replace(Y.begin(), Y.end(), 1, 10);

    // print Y
        thrust::copy(Y.begin(), Y.end(), std::ostream_iterator<int>(std::cout, "\n"));

    return 0;
}