#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/transform.h>
#include <iostream>
#include<algorithm>
#include<iomanip>
#include<time.h>

struct sample_functor {
  double alpha;
  double beta;

  sample_functor(double _alpha, double _beta) {
    alpha = _alpha;
    beta = _beta;
  }

  __device__ double operator() (const double& x, const double& y) const {
    return alpha * x + (1.0 - alpha) * (beta * y);
  }
};

int main() {

  int N = 1024 << 20;

  std::cout << "vector size:" << N << std::endl;

  srand(time(NULL));
    
  thrust::host_vector<int> host_x(N);
  std::generate(host_x.begin(),host_x.end(),rand);
  thrust::device_vector<int> device_x=host_x;

  thrust::host_vector<int> host_y(N);
  std::generate(host_y.begin(),host_y.end(),rand);
  thrust::device_vector<int> device_y=host_y;

  /*
  thrust::host_vector<double> host_x{1.1, 3.3, 2.2};
  thrust::host_vector<double> host_y{6.6, 7.7, 8.8};
  thrust::host_vector<double> host_output(3);
  thrust::device_vector<double> device_x(3);
  thrust::device_vector<double> device_y(3);
  thrust::device_vector<double> device_output(3);
  */

  thrust::host_vector<double> host_output(N);
  thrust::device_vector<double> device_output(N);
  
  double alpha = 0.005;
  double beta = 0.1;

    hipEvent_t start,stop;
    float elapsed;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start,0);

  thrust::copy(host_x.begin(), host_x.end(), device_x.begin());
  thrust::copy(host_y.begin(), host_y.end(), device_y.begin());

  thrust::transform(device_x.begin(), device_x.end(), device_y.begin(), device_output.begin(), sample_functor(alpha, beta));

  thrust::copy(device_output.begin(), device_output.end(), host_output.begin());

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed,start,stop);

    std::cout<<"gpu :"<<elapsed<<"ms ["<<std::setprecision(8)<<elapsed<<"/ms]"<<std::endl;

  std::cout << host_output[0] << ", " << host_output[1] << ", " << host_output[2] << std::endl;
  // 0.6622, 0.78265, 0.8866000000000002

  return 0;
}
