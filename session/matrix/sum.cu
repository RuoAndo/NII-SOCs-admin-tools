
#include <hip/hip_runtime.h>
#include "stdio.h"
#define N 10

__global__ void add(int *a, int *b, int *c)
{
 int tID = blockIdx.x;
 if (tID < N)
 {
 c[tID] = a[tID] + b[tID];
 }
}

int main()
{
 int a[N], b[N], c[N];
 int *dev_a, *dev_b, *dev_c;
 hipMalloc((void **) &dev_a, N*sizeof(int));
 hipMalloc((void **) &dev_b, N*sizeof(int));
 hipMalloc((void **) &dev_c, N*sizeof(int));
 // Fill Arrays
 for (int i = 0; i < N; i++)
 {
 a[i] = i,
 b[i] = 1;
 }
 hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
 hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);
 
 add<<<N,1>>>(dev_a, dev_b, dev_c);
 hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
 for (int i = 0; i < N; i++)
 {
 printf("%d + %d = %d\n", a[i], b[i], c[i]);
 }
 return 0;
}
