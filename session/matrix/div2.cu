#include "hip/hip_runtime.h"
#if __linux__ && defined(__INTEL_COMPILER)
#define __sync_fetch_and_add(ptr,addend) _InterlockedExchangeAdd(const_cast<void*>(reinterpret_cast<volatile void*>(ptr)), addend)
#endif
#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>

#include <sys/socket.h>
#include <netinet/in.h>
#include <arpa/inet.h>

#include "tbb/concurrent_hash_map.h"
#include "tbb/blocked_range.h"
#include "tbb/parallel_for.h"
#include "tbb/tick_count.h"
#include "tbb/task_scheduler_init.h"
//  #include "tbb/tbb_allocator.hz"
#include "utility.h"
#include "timer.h"

#include "csv.hpp"
typedef std::basic_string<char,std::char_traits<char>,tbb::tbb_allocator<char> > MyString;

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include "util.h"

using namespace tbb;
using namespace std;

static bool verbose = false;
static bool silent = false;

// const int size_factor = 2;
// typedef concurrent_hash_map<MyString,int> StringTable;
static MyString* Data;

std::vector<std::string> split_string_2(std::string str, char del) {
  int first = 0;
  int last = str.find_first_of(del);

  std::vector<std::string> result;

  while (first < str.size()) {
    std::string subStr(str, first, last - first);

    result.push_back(subStr);

    first = last + 1;
    last = str.find_first_of(del, first);

    if (last == std::string::npos) {
      last = str.size();
    }
  }

  return result;
}

void sumArraysOnHost(float *A, float *B, float *C, const int N)
{
    for (int idx = 0; idx < N; idx++)
        C[idx] = A[idx] + B[idx];
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    // int i = blockIdx.x * blockDim.x + threadIdx.x;
     int i = blockIdx.x; // * blockDim.x + threadIdx.x;
    // C[i] = A[i] + B[i];
    C[i] = A[i] / B[i];
    // B[i] = B[i] + 1000;
    // C[i] = 3;
}

int main( int argc, char* argv[] ) {

  // int counter = 0;
  int N = atoi(argv[2]);  
  char* tmpchar;

  struct in_addr inaddr;
  char *some_addr;

  unsigned int t, travdirtime; 

    int dev = 0;
    hipSetDevice(dev);
    int i;
    int counter;

    int nElem = N;
    printf("Vector size %d\n", nElem);

    size_t nBytes = nElem * sizeof(float);

    // float h_D[N];
    float *h_A, *h_B, *h_C, *h_D, *hostRef, *gpuRef;
    h_A     = (float *)malloc(nBytes);
    h_B     = (float *)malloc(nBytes);
    h_C     = (float *)malloc(nBytes);
    h_D     = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef  = (float *)malloc(nBytes);

    try {
        tbb::tick_count mainStartTime = tbb::tick_count::now();
        srand(2);

        utility::thread_number_range threads(tbb::task_scheduler_init::default_num_threads,0);

        if ( silent ) verbose = false;

        Data = new MyString[N];

	const string csv_file = std::string(argv[1]); 
	vector<vector<string>> data; 

	const string targetIP = std::string(argv[3]); 

	std::string stringIP;
	std::string IPstring;

	char del2 = '.';

	for (const auto subStr : split_string_2(targetIP, del2)) {
	  unsigned long ipaddr_src;
	  ipaddr_src = atol(subStr.c_str());
	  std::bitset<8> trans =  std::bitset<8>(ipaddr_src);
	  std::string trans_string = trans.to_string();
	  IPstring = IPstring + trans_string;
	}
	
	unsigned long s = bitset<32>(IPstring).to_ulong();
        float f = (float)s;
	std::cout << f << endl;
	
	try {
	  Csv objCsv(csv_file);
	  if (!objCsv.getCsv(data)) {
	    cout << "read ERROR" << endl;
	    return 1;
	  }

         thrust::host_vector<int> h_vec_1(N);

          counter = 0;
	  for (unsigned int row = 0; row < data.size(); row++) {
	    vector<string> rec = data[row]; 
	    std::string pair = rec[0] + "," + rec[1];
	    char* cstr = new char[pair.size() + 1]; 
	    std::strcpy(cstr, pair.c_str());        

	    h_vec_1.push_back(atoi(rec[1].c_str()));
	    // std::cout << rec[1] << endl;
	    h_A[counter] = stof(rec[1]);
	    h_B[counter] = f;
	    h_C[counter] = 0;

	    Data[row] += cstr;	    
	    delete[] cstr;

	    counter = counter + 1;
	  }

    memset(hostRef, 0, nBytes);
    memset(gpuRef,  0, nBytes);

    float *d_A, *d_B, *d_C;
    hipMalloc((float**)&d_A, nBytes);
    hipMalloc((float**)&d_B, nBytes);
    hipMalloc((float**)&d_C, nBytes);

    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice);

    dim3 block (1);
    dim3 grid  (N);

    start_timer(&t);  

    sumArraysOnGPU<<<grid, block>>>(d_A, d_B, d_C, nElem);
    printf("Execution configure <<<%d, %d>>>\n", grid.x, block.x);

    travdirtime = stop_timer(&t);
    print_timer(travdirtime);

    hipMemcpy(h_D, d_C, nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_C, d_C, nBytes, hipMemcpyDeviceToHost);
    // hipMemcpy(h_A, d_A, nBytes, hipMemcpyDeviceToHost);

/*
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));
*/

    for(i=0;i<10;i++)
	printf("%f %f %f \n", h_A[i], h_B[i], h_D[i]);

    free(h_A);
    free(h_B);
    free(h_C);
    free(h_D);
    free(hostRef);
    free(gpuRef);

    hipDeviceReset();

   

	}
	catch (...) {
	  cout << "EXCEPTION!" << endl;
	  return 1;
	}
	
        delete[] Data;
        utility::report_elapsed_time((tbb::tick_count::now() - mainStartTime).seconds());
       
        return 0;
	
    } catch(std::exception& e) {
        std::cerr<<"error occurred. error text is :\"" <<e.what()<<"\"\n";
    }
}
