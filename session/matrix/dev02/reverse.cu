#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>

#include <hip/hip_runtime.h>
#include <stdio.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/sequence.h>

#include "csv.hpp"
#include "timer.h"
using namespace std;

struct is_eq_zero {
    __host__ __device__ bool operator()(long a) const {
                return (a==0);
		}
};

std::vector<std::string> split_string_2(std::string str, char del) {
  int first = 0;
  int last = str.find_first_of(del);

  std::vector<std::string> result;

  while (first < str.size()) {
    std::string subStr(str, first, last - first);

    result.push_back(subStr);

    first = last + 1;
    last = str.find_first_of(del, first);

    if (last == std::string::npos) {
      last = str.size();
    }
  }

  return result;
}

int main(int argc, char **argv)
{
    int N = atoi(argv[2]);

    unsigned int t, travdirtime; 

    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("%s starting reduction at ", argv[0]);
    printf("device %d: %s \n", dev, deviceProp.name);
    hipSetDevice(dev);


    const string csv_file = std::string(argv[1]); 
    vector<vector<string>> data; 

    Csv objCsv(csv_file);
    if (!objCsv.getCsv(data)) {
     std::cout << "read ERROR" << std::endl;
     return 1;
    }

    std::remove("tmp");
    ofstream outputfile("tmp"); 

    for (int row = 0; row < data.size(); row++) {
      vector<string> rec = data[row]; 

      std::string tmpstring = rec[0];

      char del = '.';
      std::string IPstring;
      std::string stringIP = rec[1];

      // std::cout << stringIP << endl;
      std::bitset<32> s = std::bitset<32>(stoull(stringIP));
      string bs = s.to_string();
      // std::cout << ss << endl;

      string bs1 = bs.substr(0,8);
      int bi1 =  bitset<8>(bs1).to_ulong();
    
      string bs2 = bs.substr(8,8);
      int bi2 =  bitset<8>(bs2).to_ulong();

      string bs3 = bs.substr(16,8);
      int bi3 =  bitset<8>(bs3).to_ulong();

      string bs4 = bs.substr(24,8);
      int bi4 =  bitset<8>(bs4).to_ulong();

      string sourceIP = to_string(bi1) + "." + to_string(bi2) + "." + to_string(bi3) + "." + to_string(bi4);
    
      outputfile << tmpstring.substr( 0, 4 )
	     	<< "-"
		<< tmpstring.substr( 4, 2 ) 
		<< "-"
		<< tmpstring.substr( 6, 2 )
		<< " "
		<< tmpstring.substr( 8, 2 )
		<< ":"
		<< tmpstring.substr( 10, 2 )
		<< ":"
		<< tmpstring.substr( 12, 2 )
		<< "\."
		<< tmpstring.substr( 14, 3 )
		<< "," << sourceIP << endl;      
    }

    outputfile.close();

    return EXIT_SUCCESS;
}
