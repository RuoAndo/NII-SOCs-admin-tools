#include <algorithm>
#include <cfloat>
#include <chrono>
#include <fstream>
#include <iostream>
#include <random>
#include <sstream>
#include <vector>
#include <boost/tokenizer.hpp>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>
#include <random>
#include "timer.h"

using namespace std;

string now() {
    time_t t = time(0);
        char buffer[9] = {0};

    strftime(buffer, 9, "%H:%M:%S", localtime(&t));
    return string(buffer);
}

void discern(unsigned long *IPaddress, unsigned long *netmask, unsigned long address_to_match, double *result, size_t data_size, int thread_id)
{
    // int GPU_number = thread_id % 1;
    int GPU_number = 0;
    hipSetDevice(GPU_number);

    struct timespec startTime, endTime, sleepTime;

    clock_gettime(CLOCK_REALTIME, &startTime);
    sleepTime.tv_sec = 0;
    sleepTime.tv_nsec = 123;

    // thrust::host_vector<double> IPaddress_hv(data_size);
    /*
    thrust::host_vector<double> netmask_hv(data_size);
    thrust::host_vector<double> masked_IPaddress_hv(data_size);
    thrust::host_vector<double> address_to_match_hv(data_size);
    thrust::host_vector<double> result_hv(data_size);
    */

    thrust::host_vector<double> IPaddress_hv(IPaddress, IPaddress + data_size);
    thrust::host_vector<double> netmask_hv(netmask, netmask + data_size);
    thrust::host_vector<double> result_hv(result, result + data_size);
    
    thrust::host_vector<double> address_to_match_hv(data_size);
    thrust::fill(address_to_match_hv.begin(), address_to_match_hv.end(), address_to_match);
    thrust::host_vector<double> masked_IPaddress_hv(data_size);
    thrust::fill(masked_IPaddress_hv.begin(), masked_IPaddress_hv.end(), 0);

    /*
    for(int i=0; i < data_size; i++)
    {
	IPaddress_hv[i] = (double)IPaddress[i];
	netmask_hv[i] = (double)netmask[i];
	masked_IPaddress_hv[i] = 0;
	address_to_match_hv[i] = (double)address_to_match;
	result_hv[i] = (double)result[i];
    }
    */

    clock_gettime(CLOCK_REALTIME, &endTime);
    printf("[insertion] ");
    if (endTime.tv_nsec < startTime.tv_nsec) {
       printf("%10ld.%09ld", endTime.tv_sec - startTime.tv_sec - 1 ,endTime.tv_nsec + 1000000000 - startTime.tv_nsec);
    } else {
       printf("%10ld.%09ld", endTime.tv_sec - startTime.tv_sec, endTime.tv_nsec - startTime.tv_nsec);
    }
    printf(" sec\n");

    clock_gettime(CLOCK_REALTIME, &startTime);
    sleepTime.tv_sec = 0;
    sleepTime.tv_nsec = 123;

    thrust::device_vector<double> IPaddress_dv = IPaddress_hv;
    thrust::device_vector<double> netmask_dv = netmask_hv;
    thrust::device_vector<double> masked_IPaddress_dv = masked_IPaddress_hv;
    thrust::device_vector<double> address_to_match_dv = address_to_match_hv;
    thrust::device_vector<double> result_dv = result_hv;

    clock_gettime(CLOCK_REALTIME, &endTime);
    printf("[tranfering] ");
    if (endTime.tv_nsec < startTime.tv_nsec) {
       printf("%10ld.%09ld", endTime.tv_sec - startTime.tv_sec - 1 ,endTime.tv_nsec + 1000000000 - startTime.tv_nsec);
    } else {
       printf("%10ld.%09ld", endTime.tv_sec - startTime.tv_sec, endTime.tv_nsec - startTime.tv_nsec);
    }
    printf(" sec\n");

    clock_gettime(CLOCK_REALTIME, &startTime);
    sleepTime.tv_sec = 0;
    sleepTime.tv_nsec = 123;

    thrust::transform(IPaddress_dv.begin(), IPaddress_dv.end(),
    		      netmask_dv.begin(), masked_IPaddress_dv.begin(),
		      thrust::bit_and<unsigned long>());
		      
    clock_gettime(CLOCK_REALTIME, &endTime);
    printf("[transform1] ");
    if (endTime.tv_nsec < startTime.tv_nsec) {
       printf("%10ld.%09ld", endTime.tv_sec - startTime.tv_sec - 1 ,endTime.tv_nsec + 1000000000 - startTime.tv_nsec);
    } else {
       printf("%10ld.%09ld", endTime.tv_sec - startTime.tv_sec, endTime.tv_nsec - startTime.tv_nsec);
    }
    printf(" sec\n");

    // thrust::equal<double> op;

    clock_gettime(CLOCK_REALTIME, &startTime);
    sleepTime.tv_sec = 0;
    sleepTime.tv_nsec = 123;
    thrust::transform(masked_IPaddress_dv.begin(), masked_IPaddress_dv.end(),
    		      address_to_match_dv.begin(), result_dv.begin(),
		      thrust::minus<double>());
    clock_gettime(CLOCK_REALTIME, &endTime);
    printf("[transform2] ");
    if (endTime.tv_nsec < startTime.tv_nsec) {
       printf("%10ld.%09ld", endTime.tv_sec - startTime.tv_sec - 1 ,endTime.tv_nsec + 1000000000 - startTime.tv_nsec);
    } else {
       printf("%10ld.%09ld", endTime.tv_sec - startTime.tv_sec, endTime.tv_nsec - startTime.tv_nsec);
    }
    printf(" sec\n");


    clock_gettime(CLOCK_REALTIME, &startTime);
    sleepTime.tv_sec = 0;
    sleepTime.tv_nsec = 123;
    
    thrust::host_vector<double> result_hv_2(data_size);
    thrust::copy(result_dv.begin(), result_dv.end(),result_hv_2.begin());

    for(int i = 0; i < data_size; i++)
    {
    	result[i] =  result_hv_2[i];
    }

    clock_gettime(CLOCK_REALTIME, &endTime);
    printf("[transfer2] ");
    if (endTime.tv_nsec < startTime.tv_nsec) {
       printf("%10ld.%09ld", endTime.tv_sec - startTime.tv_sec - 1 ,endTime.tv_nsec + 1000000000 - startTime.tv_nsec);
    } else {
       printf("%10ld.%09ld", endTime.tv_sec - startTime.tv_sec, endTime.tv_nsec - startTime.tv_nsec);
    }
    printf(" sec\n");
}

void sort(unsigned long long *key, long *value, unsigned long long *key_out, long *value_out, int kBytes, int vBytes, size_t data_size, int thread_id)
{
    // int GPU_number = thread_id % 1;
    int GPU_number = 0;
    // struct timespec startTime, endTime, sleepTime;
    hipSetDevice(GPU_number);

    thrust::host_vector<unsigned long long> h_vec_key(data_size);
    thrust::host_vector<long> h_vec_value(data_size);

    for(int i=0; i < data_size; i++)
    {
	h_vec_key[i] = key[i];
	h_vec_value[i] = value[i];
    }
    
    thrust::device_vector<unsigned long long> d_vec_key(data_size);
    thrust::device_vector<long> d_vec_value(data_size);

    thrust::copy(h_vec_key.begin(), h_vec_key.end(), d_vec_key.begin());
    thrust::copy(h_vec_value.begin(), h_vec_value.end(), d_vec_value.begin());
    
    thrust::sort_by_key(d_vec_key.begin(), d_vec_key.end(), d_vec_value.begin());

    thrust::host_vector<unsigned long long> h_vec_key_2(data_size);
    thrust::host_vector<long> h_vec_value_2(data_size);

    thrust::copy(d_vec_value.begin(),d_vec_value.end(),h_vec_value_2.begin());
    thrust::copy(d_vec_key.begin(),d_vec_key.end(),h_vec_key_2.begin());
    
    for(int i = 0; i < data_size; i++)
    {
    	key_out[i] =  h_vec_key_2[i];
	value_out[i] =  h_vec_value_2[i];
    }
}

void transfer(unsigned long long *key, long *value, unsigned long long *key_out, long *value_out, int kBytes, int vBytes, size_t data_size, int *new_size, int thread_id)
{
    int GPU_number = thread_id % 4;
    //int GPU_number = 0;
    hipSetDevice(GPU_number);

    struct timespec startTime, endTime, sleepTime;
    // struct timespec startTime, endTime;
    clock_gettime(CLOCK_REALTIME, &startTime);
    sleepTime.tv_sec = 0;
    sleepTime.tv_nsec = 123;

    cout << "GPU" << GPU_number << ":thread_id" << thread_id << " - memory allocation start at: " << now() << endl;

    thrust::host_vector<unsigned long long> h_vec_key(data_size);
    thrust::host_vector<long> h_vec_value(data_size);

    for(int i=0; i < data_size; i++)
    {
	h_vec_key[i] = key[i];
	h_vec_value[i] = value[i];
    }

    thrust::device_vector<unsigned long long> d_vec_key(data_size);
    thrust::device_vector<long> d_vec_value(data_size);
    thrust::copy(h_vec_key.begin(), h_vec_key.end(), d_vec_key.begin());
    thrust::copy(h_vec_value.begin(), h_vec_value.end(), d_vec_value.begin());

    /*
    hipStream_t stream;
    hipStreamCreate(&stream);
    thrust::copy(thrust::cuda::par.on(stream), h_vec_key.begin(), h_vec_key.end(), d_vec_key.begin());
    thrust::copy(thrust::cuda::par.on(stream), h_vec_value.begin(), h_vec_value.end(), d_vec_value.begin());
    */

    cout << "GPU" << GPU_number << ":thread_id" << thread_id << " - memory allocation finished at: " << now() << endl;

    clock_gettime(CLOCK_REALTIME, &endTime);
    if (endTime.tv_nsec < startTime.tv_nsec) {
            printf("ThreadID:%d - [kernel][memory transfer hostToDevice] GPU[%d] - %ld.%ld", thread_id, GPU_number, 
	    			    	      			   endTime.tv_sec - startTime.tv_sec - 1 ,endTime.tv_nsec + 1000000000 - startTime.tv_nsec);
    } else {
      	    printf("ThreadID:%d - [kernel][memory transfer hostToDevice] GPU[%d] - %ld.%09ld ", thread_id, GPU_number,
	    			    	      			   	     	endTime.tv_sec - startTime.tv_sec ,endTime.tv_nsec - startTime.tv_nsec);
	    // printf("%10ld.%09ld", endTime.tv_sec - startTime.tv_sec ,endTime.tv_nsec - startTime.tv_nsec);
    }
    printf(" sec \n");

    clock_gettime(CLOCK_REALTIME, &startTime);
    sleepTime.tv_sec = 0;
    sleepTime.tv_nsec = 123;

    thrust::sort_by_key(d_vec_key.begin(), d_vec_key.end(), d_vec_value.begin());
    thrust::device_vector<unsigned long long> d_vec_key_out(data_size);
    thrust::device_vector<long> d_vec_value_out(data_size);

    auto new_end = thrust::reduce_by_key(d_vec_key.begin(), d_vec_key.end(), d_vec_value.begin(), d_vec_key_out.begin(), d_vec_value_out.begin());
    int new_size_r = new_end.first - d_vec_key_out.begin();

    // thrust::sort_by_key(key, key + data_size, value);
    // auto new_end = thrust::reduce_by_key(key, key + data_size, value, key_out, value_out);

    clock_gettime(CLOCK_REALTIME, &endTime);
    if (endTime.tv_nsec < startTime.tv_nsec) {
            printf("ThreadID:%d - [kernel][reduction] GPU[%d] - %ld.%09ld", thread_id, GPU_number, 
	    						     		 endTime.tv_sec - startTime.tv_sec - 1 ,endTime.tv_nsec + 1000000000 - startTime.tv_nsec);
    } else {
      	    printf("ThreadID:%d - [kernel][reduction] GPU[%d] - %ld.%09ld ", thread_id, GPU_number, 
	    			    	      			         endTime.tv_sec - startTime.tv_sec ,endTime.tv_nsec - startTime.tv_nsec);
	    					    		 
	    // printf("%10ld.%09ld", endTime.tv_sec - startTime.tv_sec ,endTime.tv_nsec - startTime.tv_nsec);
    }
    printf(" sec \n");

    clock_gettime(CLOCK_REALTIME, &startTime);
    sleepTime.tv_sec = 0;
    sleepTime.tv_nsec = 123;

    thrust::host_vector<unsigned long long> h_vec_key_2(data_size);
    thrust::host_vector<long> h_vec_value_2(data_size);

    thrust::copy(d_vec_value_out.begin(),d_vec_value_out.end(),h_vec_value_2.begin());
    thrust::copy(d_vec_key_out.begin(),d_vec_key_out.end(),h_vec_key_2.begin());

    for(int i = 0; i < new_size_r; i++)
    {
    	key_out[i] =  h_vec_key_2[i];
	value_out[i] =  h_vec_value_2[i];
    }

    clock_gettime(CLOCK_REALTIME, &endTime);
    if (endTime.tv_nsec < startTime.tv_nsec) {
            printf("ThreadID:%d - [kernel][memory transfer deviceToHost] GPU[%d] - %ld.%ld", thread_id, GPU_number,
	    			    	      			   endTime.tv_sec - startTime.tv_sec - 1 ,endTime.tv_nsec + 1000000000 - startTime.tv_nsec);
    } else {
      	    printf("ThreadID:%d - [kernel][memory transfer deviceToHost] GPU[%d] - %ld.%09ld ", thread_id, GPU_number,
	    			    	      			   	     	endTime.tv_sec - startTime.tv_sec ,endTime.tv_nsec - startTime.tv_nsec);
	    // printf("%10ld.%09ld", endTime.tv_sec - startTime.tv_sec ,endTime.tv_nsec - startTime.tv_nsec);
    }
    printf(" sec \n");

    (*new_size) = new_size_r;
    // (*new_size) = sizeof(*new_end.first);

    // printf("開始時刻　 = %10ld.%09ld\n", startTime.tv_sec, startTime.tv_nsec);
    // printf("終了時刻　 = %10ld.%09ld\n", endTime.tv_sec, endTime.tv_nsec);
    // printf("経過実時間 = ");

    /*
    clock_gettime(CLOCK_REALTIME, &endTime);
    if (endTime.tv_nsec < startTime.tv_nsec) {
            printf("%10ld.%09ld", endTime.tv_sec - startTime.tv_sec - 1 ,endTime.tv_nsec + 1000000000 - startTime.tv_nsec);
    } else {
      	    printf("GPU%d:thread_id:%d - [kernel][reduction] %ld.%09ld", GPU_number, thread_id,
	    			 endTime.tv_sec - startTime.tv_sec ,endTime.tv_nsec - startTime.tv_nsec);
	    // printf("%10ld.%09ld", endTime.tv_sec - startTime.tv_sec ,endTime.tv_nsec - startTime.tv_nsec);
    }
    printf(" sec \n");
    */

}

void kernel(long *h_key, long *h_value_1, long *h_value_2, string filename, int size)
{
  int N = size;

  cout << "kernel" << endl;

  for(int i = 0; i < 5; i++)
  {
	cout << h_key[i] << "," << h_value_1[i] << endl;
  }

  thrust::host_vector<int> h_vec_1(N);
  std::generate(h_vec_1.begin(), h_vec_1.end(), rand); 

  thrust::device_vector<int> key_in(N);
  thrust::copy(h_vec_1.begin(), h_vec_1.end(), key_in.begin()); 

  thrust::host_vector<unsigned long long> h_vec_key_1(N);
  thrust::host_vector<unsigned long long> h_vec_key_2(N);

  thrust::host_vector<long> h_vec_value_1(N);
  thrust::host_vector<long> h_vec_value_2(N);

  cout << N << endl;

  for(int i=0; i < N; i++)
  {
	// cout << h_key[i] << endl;
	h_vec_key_1[i] = h_key[i];
	h_vec_key_2[i] = h_key[i];
	h_vec_value_1[i] = h_value_1[i];
	h_vec_value_2[i] = h_value_2[i];
  }

  /* 1 -> 3 */

  thrust::device_vector<unsigned long long> d_vec_key_1(N);
  thrust::device_vector<long> d_vec_value_1(N);
  thrust::copy(h_vec_key_1.begin(), h_vec_key_1.end(), d_vec_key_1.begin());
  thrust::copy(h_vec_value_1.begin(), h_vec_value_1.end(), d_vec_value_1.begin());

  // thrust::sort_by_key(d_vec_key_1.begin(), d_vec_key_1.end(), d_vec_value_1.begin(), thrust::greater<unsigned long long>());

  thrust::sort_by_key(d_vec_key_1.begin(), d_vec_key_1.end(), d_vec_value_1.begin());

  thrust::host_vector<unsigned long long> h_vec_key_3(N);
  thrust::host_vector<long> h_vec_value_3(N);

  thrust::copy(d_vec_value_1.begin(),d_vec_value_1.end(),h_vec_value_3.begin());
  thrust::copy(d_vec_key_1.begin(),d_vec_key_1.end(),h_vec_key_3.begin());

  /* 2 -> 4 */

  thrust::device_vector<unsigned long long> d_vec_key_2(N);
  thrust::device_vector<long> d_vec_value_2(N);
  thrust::copy(h_vec_key_2.begin(), h_vec_key_2.end(), d_vec_key_2.begin());
  thrust::copy(h_vec_value_2.begin(), h_vec_value_2.end(), d_vec_value_2.begin());

  // thrust::sort_by_key(d_vec_key_2.begin(), d_vec_key_2.end(), d_vec_value_2.begin(), thrust::greater<unsigned long long>());

  thrust::sort_by_key(d_vec_key_2.begin(), d_vec_key_2.end(), d_vec_value_2.begin());

  thrust::host_vector<unsigned long long> h_vec_key_4(N);
  thrust::host_vector<long> h_vec_value_4(N);

  thrust::copy(d_vec_value_2.begin(),d_vec_value_2.end(),h_vec_value_4.begin());
  thrust::copy(d_vec_key_2.begin(),d_vec_key_2.end(),h_vec_key_4.begin());

  cout << "1 -> 3" << endl;
  for(int i = 0; i < 5; i++)
  {
	cout << h_vec_key_3[i] << "," << h_vec_value_3[i] << endl;
  }

  cout << "2 -> 4" << endl;
  for(int i = 0; i < 5; i++)
  {
	cout << h_vec_key_4[i] << "," << h_vec_value_4[i] << endl;
  }

  thrust::device_vector<unsigned long long> d_vec_key_1_out(N);
  thrust::device_vector<long> d_vec_value_1_out(N);

  thrust::device_vector<unsigned long long> d_vec_key_2_out(N);
  thrust::device_vector<long> d_vec_value_2_out(N);

  auto new_end_1 = thrust::reduce_by_key(d_vec_key_1.begin(), d_vec_key_1.end(), d_vec_value_1.begin(),
       	       	 		       d_vec_key_1_out.begin(), d_vec_value_1_out.begin());

  int new_size_1 = new_end_1.first - d_vec_key_1_out.begin() + 1; 

  thrust::host_vector<unsigned long long> h_vec_key_3_out(N);
  thrust::host_vector<long> h_vec_value_3_out(N);

  thrust::copy(d_vec_value_1_out.begin(),d_vec_value_1_out.end(),h_vec_value_3_out.begin());
  thrust::copy(d_vec_key_1_out.begin(),d_vec_key_1_out.end(),h_vec_key_3_out.begin());

  auto new_end_2 = thrust::reduce_by_key(d_vec_key_2.begin(), d_vec_key_2.end(), d_vec_value_2.begin(),
       	       	 		       d_vec_key_2_out.begin(), d_vec_value_2_out.begin());      

  int new_size_2 = new_end_2.first - d_vec_key_2_out.begin();// + 1; 

  thrust::host_vector<unsigned long long> h_vec_key_4_out(N);
  thrust::host_vector<long> h_vec_value_4_out(N);

  thrust::copy(d_vec_value_2_out.begin(),d_vec_value_2_out.end(),h_vec_value_4_out.begin());
  thrust::copy(d_vec_key_2_out.begin(),d_vec_key_2_out.end(),h_vec_key_4_out.begin());

  cout << "1 -> 3" << endl;
  for(int i = 0; i < 5; i++)
  {
	cout << h_vec_key_3_out[i] << "," << h_vec_value_3_out[i] << endl;
  }

  cout << "2 -> 4" << endl;
  for(int i = 0; i < 5; i++)
  {
	cout << h_vec_key_4_out[i] << "," << h_vec_value_4_out[i] << endl;
  }

  ofstream outputfile(filename);
    
  cout << "all" << endl;

  outputfile << "timestamp, counted, bytes" << endl;
  
  for(int i = 0; i < new_size_2; i++)
  {
	// cout << h_vec_key_3_out[i] << "," << h_vec_value_3_out[i] << "," << h_vec_value_4_out[i] << endl;

	/*
	if(h_vec_key_3_out[i] != 0)
		outputfile << h_vec_key_3_out[i] << "," << h_vec_value_3_out[i] << "," << h_vec_value_4_out[i] << endl;
	*/
	
	std::string timestamp = to_string(h_vec_key_3_out[i]);

	outputfile << timestamp.substr(0,4) << "-" << timestamp.substr(4,2) << "-" << timestamp.substr(6,2) << " "
	     	   << timestamp.substr(8,2) << ":" << timestamp.substr(10,2) << ":" << timestamp.substr(12,2)
	     	   << "." << timestamp.substr(14,3) << "," 
		   << h_vec_value_3_out[i] << "," << h_vec_value_4_out[i] << endl;
  }

  outputfile.close();
  
}


