#include "hip/hip_runtime.h"
#if __linux__ && defined(__INTEL_COMPILER)
#define __sync_fetch_and_add(ptr,addend) _InterlockedExchangeAdd(const_cast<void*>(reinterpret_cast<volatile void*>(ptr)), addend)
#endif
#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>

#include <sys/socket.h>
#include <netinet/in.h>
#include <arpa/inet.h>   

#include "tbb/concurrent_hash_map.h"
#include "tbb/blocked_range.h"
#include "tbb/parallel_for.h"
#include "tbb/tick_count.h"
#include "tbb/task_scheduler_init.h"
#include "tbb/concurrent_vector.h"
#include "utility.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "csv.hpp"
#include "timer.h"

using namespace tbb;
using namespace std;

std::vector<string> timestamp;

__global__ void sumArraysOnGPU(unsigned long long *A, long *B, long *C, const int N)
{
    // extern __shared__ long *shared_data[];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // int idx = threadIdx.x;
    
    for (int i = 0; i < N; ++i) {
    	if(A[idx]==A[i])
	{
	  C[idx]++;
	}
    }
    __syncthreads();
}

int main( int argc, char* argv[] ) {

  int counter = 0;
  int N = atoi(argv[2]);  

  const size_t ullBytes = N * sizeof(unsigned long long);  
  const size_t lBytes = N * sizeof(long);  

  unsigned long long *h_A;
  long *h_B;  

  h_A = (unsigned long long *)malloc(ullBytes);
  h_B = (long *)malloc(lBytes);

  thrust::host_vector<unsigned long long> h_key_in_all(N);
  thrust::host_vector<long> h_value_in_all(N);

  unsigned int t, travdirtime; 

  int ngpus = 4;

    try {
        tbb::tick_count mainStartTime = tbb::tick_count::now();
        srand(2);

        utility::thread_number_range threads(tbb::task_scheduler_init::default_num_threads,0);

	const string csv_file = std::string(argv[1]); 
	vector<vector<string>> data; 

 	unsigned long long **d_k_i = (unsigned long long **)malloc(sizeof(unsigned long long *) * ngpus);
  	long **d_v_i = (long **)malloc(sizeof(long *) * ngpus);

 	unsigned long long **h_k_i = (unsigned long long **)malloc(sizeof(unsigned long long *) * ngpus);
  	long **h_v_i = (long **)malloc(sizeof(long *) * ngpus);
	
  	hipStream_t *stream = (hipStream_t *)malloc(sizeof(hipStream_t) * ngpus); 

  	for (int i = 0; i < ngpus; i++)
  	{
	   hipSetDevice(i);
           hipMalloc((void **) &d_k_i[i], ullBytes);
	   hipMalloc((void **) &d_v_i[i], lBytes);
	   hipHostMalloc((void **) &h_k_i[i], ullBytes);
	   hipHostMalloc((void **) &h_v_i[i], lBytes);	
	   hipStreamCreate(&stream[i]);
  	}                  

	try {
	  Csv objCsv(csv_file);
	  if (!objCsv.getCsv(data)) {
	    cout << "read ERROR" << endl;
	    return 1;
	  }

	  counter = 0;
	  for (unsigned int row = 0; row < data.size(); row++) {
	    vector<string> rec = data[row]; 
	    std::string tms = rec[0];

	    for(size_t c = tms.find_first_of("\""); c != string::npos; c = c = tms.find_first_of("\"")){
	      tms.erase(c,1);
	    }

	    for(size_t c = tms.find_first_of("/"); c != string::npos; c = c = tms.find_first_of("/")){
	      tms.erase(c,1);
	    }

	    for(size_t c = tms.find_first_of("."); c != string::npos; c = c = tms.find_first_of(".")){
	      tms.erase(c,1);
	    }

	    for(size_t c = tms.find_first_of(" "); c != string::npos; c = c = tms.find_first_of(" ")){
	      tms.erase(c,1);
	    }

	    for(size_t c = tms.find_first_of(":"); c != string::npos; c = c = tms.find_first_of(":")){
	      tms.erase(c,1);
	    }

	   h_A[row] = stoull(tms);
      	   h_B[row] = 1;

	   if(counter < N/2) {
		h_k_i[0][counter] = stoull(tms);
      	   	h_v_i[0][counter] = 1;
	   }

	   if(counter > N/2) {
		h_k_i[1][counter] = stoull(tms);
      	   	h_v_i[1][counter] = 1;
	   }

	   counter = counter + 1;

	  }

	  int iLen = 1024;
	  dim3 block (iLen);
	  dim3 grid  ((N + block.x - 1) / block.x); 

	  hipSetDevice(0);
	  hipMemcpyAsync(d_k_i[0], h_k_i[0], N/2, hipMemcpyHostToDevice, stream[0]);
	  hipMemcpyAsync(d_v_i[0], h_v_i[0], N/2, hipMemcpyHostToDevice, stream[0]);
	  hipStreamSynchronize(stream[0]);

	  unsigned long long *d_A;
	  long *d_B;
	  hipMalloc((unsigned long long**)&d_A, ullBytes);
	  hipMalloc((long**)&d_B, lBytes);

	  hipMemcpy(d_A, h_k_i[0], ullBytes, hipMemcpyHostToDevice);
	  hipMemcpy(d_B, h_v_i[0], lBytes, hipMemcpyHostToDevice);

	  long *sum, *gpuRef;
	  
	  hipMalloc((long**)&sum, lBytes);

	  sumArraysOnGPU<<<grid, block>>>(d_A, d_B, sum, N);

	  gpuRef  = (long *)malloc(lBytes);
	  hipMemcpy(gpuRef, sum, lBytes, hipMemcpyDeviceToHost); 

	  for(int i = 0; i < 10; i++)
	  {
		    cout << h_k_i[0][i] << "," << gpuRef[i] <<"," << h_v_i[0][i] << endl;
          }

	}
	catch (...) {
	  cout << "EXCEPTION!" << endl;
	  return 1;
	}

        return 0;
	
    } catch(std::exception& e) {
        std::cerr<<"error occurred. error text is :\"" <<e.what()<<"\"\n";
    }
}
