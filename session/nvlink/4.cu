#include "hip/hip_runtime.h"
#if __linux__ && defined(__INTEL_COMPILER)
#define __sync_fetch_and_add(ptr,addend) _InterlockedExchangeAdd(const_cast<void*>(reinterpret_cast<volatile void*>(ptr)), addend)
#endif
#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>

#include <sys/socket.h>
#include <sys/time.h>
#include <netinet/in.h>
#include <arpa/inet.h>   

#include "tbb/concurrent_hash_map.h"
#include "tbb/blocked_range.h"
#include "tbb/parallel_for.h"
#include "tbb/tick_count.h"
#include "tbb/task_scheduler_init.h"
#include "tbb/concurrent_vector.h"
#include "utility.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/system/cuda/experimental/pinned_allocator.h>
#include <thrust/system/cuda/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <thrust/for_each.h>

#include "csv.hpp"
#include "timer.h"

using namespace tbb;
using namespace std;

std::vector<string> timestamp;

#define min(a, b) (a < b ? a : b)

typedef long mytype;
typedef thrust::host_vector<mytype, thrust::cuda::experimental::pinned_allocator<mytype> > pinnedVector1;

typedef thrust::host_vector<mytype, thrust::cuda::experimental::pinned_allocator<mytype> > pinnedVector2;

#define TV1 1
#define TV2 2




int main(int argc, char** argv) {

    int N = atoi(argv[2]);
    unsigned int t, travdirtime; 

    hipSetDevice(0);
    pinnedVector1 hi1(N);
    pinnedVector2 ho1(N);

    hipSetDevice(1);
    pinnedVector1 hi2(N);
    pinnedVector2 ho2(N);

    const string csv_file = std::string(argv[1]); 
    vector<vector<string>> data; 

    Csv objCsv(csv_file);
    if (!objCsv.getCsv(data)) {
       cout << "read ERROR" << endl;
       return 1;
       }

    long size=atoi(argv[2]);

    for (unsigned int row = 0; row < data.size(); row++) {
    	vector<string> rec = data[row];

	std::string tms = rec[0];
	std::string bytes = rec[20];

	for(size_t c = tms.find_first_of("\""); c != string::npos; c = c = tms.find_first_of("\"")){
    	      tms.erase(c,1);
	}

	for(size_t c = tms.find_first_of("/"); c != string::npos; c = c = tms.find_first_of("/")){
	      tms.erase(c,1);
	}

        for(size_t c = tms.find_first_of("."); c != string::npos; c = c = tms.find_first_of(".")){
	      tms.erase(c,1);
	}

	for(size_t c = tms.find_first_of(" "); c != string::npos; c = c = tms.find_first_of(" ")){
	      tms.erase(c,1);
	}

	for(size_t c = tms.find_first_of(":"); c != string::npos; c = c = tms.find_first_of(":")){
	      tms.erase(c,1);
	}

	for(size_t c = bytes.find_first_of("\""); c != string::npos; c = c = bytes.find_first_of("\"")){
	      bytes.erase(c,1);
	}

	hi1.push_back(stol(tms));
	hi2.push_back(stol(tms));

	ho1.push_back(stol(bytes));
	ho2.push_back(stol(bytes));
    }

    hipSetDevice(0);
    thrust::device_vector<mytype> di1(N);
    thrust::device_vector<mytype> do1(N);

    hipSetDevice(1);
    thrust::device_vector<mytype> di2(N);
    thrust::device_vector<mytype> do2(N);

    hipSetDevice(0);
    thrust::fill(hi1.begin(), hi1.end(),  TV1);
    thrust::sequence(do1.begin(), do1.end());

    hipSetDevice(1);
    thrust::fill(hi2.begin(), hi2.end(),  TV2);
    thrust::sequence(do2.begin(), do2.end());

    hipStream_t s1, s2;
  
    hipSetDevice(0);
    hipStreamCreate(&s1);
    hipSetDevice(1);
    hipStreamCreate(&s2);

    start_timer(&t);
    hipSetDevice(0);
    hipMemcpyAsync(thrust::raw_pointer_cast(di1.data()), thrust::raw_pointer_cast(hi1.data()), di1.size()*sizeof(mytype), hipMemcpyHostToDevice, s1);

    hipMemcpyAsync(thrust::raw_pointer_cast(di2.data()), thrust::raw_pointer_cast(hi2.data()), di2.size()*sizeof(mytype), hipMemcpyHostToDevice, s2);

    hipDeviceSynchronize();

    travdirtime = stop_timer(&t);
    print_timer(travdirtime);

    

}



