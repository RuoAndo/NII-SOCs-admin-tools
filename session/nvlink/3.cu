#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/system/cuda/experimental/pinned_allocator.h>
#include <thrust/system/cuda/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <thrust/for_each.h>
#include <iostream>
#include "timer.h"

// DSIZE determines duration of H2D and D2H transfers
#define DSIZE (1048576*64)
// SSIZE,LSIZE determine duration of kernel launched by thrust
#define SSIZE (512*512)
#define LSIZE 1
// KSIZE determines size of thrust kernels (number of threads per block)
#define KSIZE 64
#define TV1 1
#define TV2 2

typedef int mytype;
typedef thrust::host_vector<mytype, thrust::cuda::experimental::pinned_allocator<mytype> > pinnedVector;

struct sum_functor
{
  mytype *dptr;
  sum_functor(mytype* _dptr) : dptr(_dptr) {};
  __host__ __device__ void operator()(mytype &data) const
    {
      mytype result = data;
      for (int j = 0; j < LSIZE; j++)
        for (int i = 0; i < SSIZE; i++)
          result += dptr[i];
      data = result;
    }
};

int main(){

  unsigned int t, travdirtime; 

  hipSetDevice(0);
  pinnedVector hi1(DSIZE);
  pinnedVector ho1(DSIZE);

  hipSetDevice(1);
  pinnedVector hi2(DSIZE);
  pinnedVector ho2(DSIZE);

  hipSetDevice(0);
  thrust::device_vector<mytype> di1(DSIZE);
  thrust::device_vector<mytype> do1(DSIZE);
  thrust::device_vector<mytype> dc1(KSIZE);

  hipSetDevice(1);
  thrust::device_vector<mytype> di2(DSIZE);
  thrust::device_vector<mytype> do2(DSIZE);
  thrust::device_vector<mytype> dc2(KSIZE);

  hipSetDevice(0);
  thrust::fill(hi1.begin(), hi1.end(),  TV1);
  thrust::sequence(do1.begin(), do1.end());

  hipSetDevice(1);
  thrust::fill(hi2.begin(), hi2.end(),  TV2);
  thrust::sequence(do2.begin(), do2.end());

  hipStream_t s1, s2;
  
  hipSetDevice(0);
  hipStreamCreate(&s1);
  
  hipSetDevice(1);
  hipStreamCreate(&s2);

  start_timer(&t);
  hipSetDevice(0);
  hipMemcpyAsync(thrust::raw_pointer_cast(di1.data()), thrust::raw_pointer_cast(hi1.data()), di1.size()*sizeof(mytype), hipMemcpyHostToDevice, s1);
  thrust::for_each(thrust::cuda::par.on(s1), do1.begin(), do1.begin()+KSIZE, sum_functor(thrust::raw_pointer_cast(di1.data())));
  hipMemcpyAsync(thrust::raw_pointer_cast(ho1.data()), thrust::raw_pointer_cast(do1.data()), do1.size()*sizeof(mytype), hipMemcpyDeviceToHost, s1);
  travdirtime = stop_timer(&t);
  print_timer(travdirtime);

  start_timer(&t); 
  hipSetDevice(1);
  hipMemcpyAsync(thrust::raw_pointer_cast(di2.data()), thrust::raw_pointer_cast(hi2.data()), di2.size()*sizeof(mytype), hipMemcpyHostToDevice, s2);
  thrust::for_each(thrust::cuda::par.on(s2), do2.begin(), do2.begin()+KSIZE, sum_functor(thrust::raw_pointer_cast(di2.data())));
  hipMemcpyAsync(thrust::raw_pointer_cast(ho2.data()), thrust::raw_pointer_cast(do2.data()), do2.size()*sizeof(mytype), hipMemcpyDeviceToHost, s2);
  travdirtime = stop_timer(&t);
  print_timer(travdirtime);

  hipDeviceSynchronize();


  for (int i=0; i < KSIZE; i++){
    if (ho1[i] != ((LSIZE*SSIZE*TV1) + i)) { std::cout << "mismatch on stream 1 at " << i << " was: " << ho1[i] << " should be: " << ((DSIZE*TV1)+i) << std::endl; return 1;}
    if (ho2[i] != ((LSIZE*SSIZE*TV2) + i)) { std::cout << "mismatch on stream 2 at " << i << " was: " << ho2[i] << " should be: " << ((DSIZE*TV2)+i) << std::endl; return 1;}
    }
  std::cout << "Success!" << std::endl;
  return 0;
}
