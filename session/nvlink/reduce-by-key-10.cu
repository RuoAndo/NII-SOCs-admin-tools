#include "hip/hip_runtime.h"
#if __linux__ && defined(__INTEL_COMPILER)
#define __sync_fetch_and_add(ptr,addend) _InterlockedExchangeAdd(const_cast<void*>(reinterpret_cast<volatile void*>(ptr)), addend)
#endif
#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>

#include <sys/socket.h>
#include <sys/time.h>
#include <netinet/in.h>
#include <arpa/inet.h>   

#include "tbb/concurrent_hash_map.h"
#include "tbb/blocked_range.h"
#include "tbb/parallel_for.h"
#include "tbb/tick_count.h"
#include "tbb/task_scheduler_init.h"
#include "tbb/concurrent_vector.h"
#include "utility.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "csv.hpp"
#include "timer.h"

using namespace tbb;
using namespace std;

std::vector<string> timestamp;

// data[], size, threads, blocks, 
void mergesort(long*, long*, long, dim3, dim3);
// A[]. B[], size, width, slices, nThreads
__global__ void gpu_mergesort(long*, long*, long*, long*, long, long, long, dim3*, dim3*);
__device__ void gpu_bottomUpMerge(long*, long*, long*, long*, long, long, long);

#define min(a, b) (a < b ? a : b)

__global__ void sumArraysOnGPU(long *A, long *B, long *C, long *D, const int N)
{
    // extern __shared__ long *shared_data[];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // int idx = threadIdx.x;
    
    for (int i = 0; i < N; ++i) {
    	if(A[idx]==A[i])
	{
	  // D[idx]++;
	  D[idx] += B[i];
	}
    }
    __syncthreads();

    for (int i = 0; i < N; ++i) {
    	if( A[idx]==A[i] && idx > i)
	{
	  C[idx]=0;
	}
    }
    __syncthreads();
}


int main(int argc, char** argv) {

    int N = atoi(argv[2]);

    dim3 threadsPerBlock;
    dim3 blocksPerGrid;

    threadsPerBlock.x = 32;
    threadsPerBlock.y = 1;
    threadsPerBlock.z = 1;

    blocksPerGrid.x = 8;
    blocksPerGrid.y = 1;
    blocksPerGrid.z = 1;

    const string csv_file = std::string(argv[1]); 
    vector<vector<string>> data; 

    // tm();

    Csv objCsv(csv_file);
    if (!objCsv.getCsv(data)) {
       cout << "read ERROR" << endl;
       return 1;
       }

    long size=atoi(argv[2]);

    size_t nBytes = size * sizeof(long);
    long *data2;
    data2 = (long *)malloc(nBytes);

    size_t ullBytes = size * sizeof(unsigned long long);

    unsigned long long *data3;
    data3 = (unsigned long long *)malloc(ullBytes);

    long *value;
    value = (long *)malloc(nBytes);

    long *value_sorted;
    value_sorted = (long *)malloc(nBytes);

    long *value2;
    value2 = (long *)malloc(nBytes);     

    // counter = 0;
    for (unsigned int row = 0; row < data.size(); row++) {
    	vector<string> rec = data[row];
	std::string tms = rec[0];
	std::string bytes = rec[20];

	for(size_t c = tms.find_first_of("\""); c != string::npos; c = c = tms.find_first_of("\"")){
    	      tms.erase(c,1);
	}

	for(size_t c = tms.find_first_of("/"); c != string::npos; c = c = tms.find_first_of("/")){
	      tms.erase(c,1);
	}

        for(size_t c = tms.find_first_of("."); c != string::npos; c = c = tms.find_first_of(".")){
	      tms.erase(c,1);
	}

	for(size_t c = tms.find_first_of(" "); c != string::npos; c = c = tms.find_first_of(" ")){
	      tms.erase(c,1);
	}

	for(size_t c = tms.find_first_of(":"); c != string::npos; c = c = tms.find_first_of(":")){
	      tms.erase(c,1);
	}

	for(size_t c = bytes.find_first_of("\""); c != string::npos; c = c = bytes.find_first_of("\"")){
	      bytes.erase(c,1);
	}

	data2[row] = stol(tms);
	value[row] = stol(bytes);
    }

    for(int i = 0; i < 5; i++)
    	    cout << data2[i] << endl;
    
    std::cout << "sorting " << size << " numbers\n\n";
    
    // merge-sort the data
    mergesort(data2, value, size, threadsPerBlock, blocksPerGrid);

    /*
    for(int i = 0; i < 5; i++)
    	    cout << data2[i] << "," << value[i] << endl;
    */

    long *d_A;
    long *d_B;
    hipMalloc((long**)&d_A, nBytes);
    hipMalloc((long**)&d_B, nBytes);
    hipMemcpy(d_A, data2, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, value, nBytes, hipMemcpyHostToDevice);
    
    long *d_new_key, *d_new_value;
    hipMalloc((long**)&d_new_key, nBytes);
    hipMalloc((long**)&d_new_value, nBytes);
    hipMemcpy(d_new_key, data2, nBytes, hipMemcpyHostToDevice);

    int iLen = 1024;
    dim3 block (iLen);
    dim3 grid  ((N + block.x - 1) / block.x); 

    sumArraysOnGPU<<<grid, block>>>(d_A, d_B, d_new_key, d_new_value, size);

    long *gpuRef, *gpuRef2;
    
    gpuRef  = (long *)malloc(nBytes);
    hipMemcpy(gpuRef, d_new_key, nBytes, hipMemcpyDeviceToHost); 

    gpuRef2  = (long *)malloc(nBytes);
    hipMemcpy(gpuRef2, d_new_value, nBytes, hipMemcpyDeviceToHost); 

    int new_size = 0;
    for(int i = 0; i < N; i++)
    {
	if(gpuRef[i] != 0)
	{
	    cout << gpuRef[i] << "," << gpuRef2[i] << endl;     
	    new_size++;
	}
    }
}

void mergesort(long* data, long *value, long size, dim3 threadsPerBlock, dim3 blocksPerGrid) {

    //
    // Allocate two arrays on the GPU
    // we switch back and forth between them during the sort
    //
    long* D_data;
    long* D_swp;

    long* V_data;
    long* V_swp;
    
    dim3* D_threads;
    dim3* D_blocks;
    
    // Actually allocate the two arrays
    //tm();
    hipMalloc((void**) &D_data, size * sizeof(long));
    hipMalloc((void**) &D_swp, size * sizeof(long));

    hipMalloc((void**) &V_data, size * sizeof(long));
    hipMalloc((void**) &V_swp, size * sizeof(long));
    
        //std::cout << "hipMalloc device lists: " << tm() << " microseconds\n";

    // Copy from our input list into the first array
    hipMemcpy(D_data, data, size * sizeof(long), hipMemcpyHostToDevice);
    hipMemcpy(V_data, value, size * sizeof(long), hipMemcpyHostToDevice);

        // std::cout << "hipMemcpy list to device: " << tm() << " microseconds\n";
 
    //
    // Copy the thread / block info to the GPU as well
    //
    hipMalloc((void**) &D_threads, sizeof(dim3));
    hipMalloc((void**) &D_blocks, sizeof(dim3));

        // std::cout << "hipMalloc device thread data: " << tm() << " microseconds\n";
    hipMemcpy(D_threads, &threadsPerBlock, sizeof(dim3), hipMemcpyHostToDevice);
    hipMemcpy(D_blocks, &blocksPerGrid, sizeof(dim3), hipMemcpyHostToDevice);

        // std::cout << "hipMemcpy thread data to device: " << tm() << " microseconds\n";

    long* A = D_data;
    long* B = D_swp;

    long* C = V_data;
    long* D = V_swp;

    long nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
                    blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;

    //
    // Slice up the list and give pieces of it to each thread, letting the pieces grow
    // bigger and bigger until the whole list is sorted
    //
    for (int width = 2; width < (size << 1); width <<= 1) {
        long slices = size / ((nThreads) * width) + 1;

            std::cout << "mergeSort - width: " << width 
                      << ", slices: " << slices 
                      << ", nThreads: " << nThreads << '\n';
            // tm();

        // Actually call the kernel
        gpu_mergesort<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, D, size, width, slices, D_threads, D_blocks);

            //std::cout << "call mergesort kernel: " << tm() << " microseconds\n";

        // Switch the input / output arrays instead of copying them around
        A = A == D_data ? D_swp : D_data;
        B = B == D_data ? D_swp : D_data;
    }

    //
    // Get the list back from the GPU
    //
    // tm();
    hipMemcpy(data, A, size * sizeof(long), hipMemcpyDeviceToHost);
    hipMemcpy(value, D, size * sizeof(long), hipMemcpyDeviceToHost);
        // std::cout << "hipMemcpy list back to host: " << tm() << " microseconds\n";
    
    // Free the GPU memory
    hipFree(A);
    hipFree(B);
        //std::cout << "hipFree: " << tm() << " microseconds\n";
}

// GPU helper function
// calculate the id of the current thread
__device__ unsigned int getIdx(dim3* threads, dim3* blocks) {
    int x;
    return threadIdx.x +
           threadIdx.y * (x  = threads->x) +
           threadIdx.z * (x *= threads->y) +
           blockIdx.x  * (x *= threads->z) +
           blockIdx.y  * (x *= blocks->z) +
           blockIdx.z  * (x *= blocks->y);
}

//
// Perform a full mergesort on our section of the data.
//
__global__ void gpu_mergesort(long* source, long* dest, long* value_src, long* value_dest, long size, long width, long slices, dim3* threads, dim3* blocks) {
    unsigned int idx = getIdx(threads, blocks);
    long start = width*idx*slices, 
         middle, 
         end;

    for (long slice = 0; slice < slices; slice++) {
        if (start >= size)
            break;

        middle = min(start + (width >> 1), size);
        end = min(start + width, size);
        gpu_bottomUpMerge(source, dest, value_src, value_dest, start, middle, end);
        start += width;
    }
}

//
// Finally, sort something
// gets called by gpu_mergesort() for each slice
//
__device__ void gpu_bottomUpMerge(long* source, long* dest, long* value_src, long* value_dest, long start, long middle, long end) {
    long i = start;
    long j = middle;
    for (long k = start; k < end; k++) {
        if (i < middle && (j >= end || source[i] < source[j])) {
            dest[k] = source[i];
	    value_dest[k] = value_src[i];
            i++;
        } else {
            dest[k] = source[j];
	    value_dest[k] = value_src[i];
            j++;
        }
    }
}

// read data into a minimal linked list
typedef struct {
    int v;
    void* next;
} LinkNode;

// helper function for reading numbers from stdin
// it's 'optimized' not to check validity of the characters it reads in..
long readList(long** list) {
    //tm();
    long v, size = 0;
    LinkNode* node = 0;
    LinkNode* first = 0;
    while (std::cin >> v) {
        LinkNode* next = new LinkNode();
        next->v = v;
        if (node)
            node->next = next;
        else 
            first = next;
        node = next;
        size++;
    }


    if (size) {
        *list = new long[size]; 
        LinkNode* node = first;
        long i = 0;
        while (node) {
            (*list)[i++] = node->v;
            node = (LinkNode*) node->next;
        }

    }

        //std::cout << "read stdin: " << tm() << " microseconds\n";

    return size;
}


// 
// Get the time (in microseconds) since the last call to tm();
// the first value returned by this must not be trusted
//
timeval tStart;
int tm() {
    timeval tEnd;
    gettimeofday(&tEnd, 0);
    int t = (tEnd.tv_sec - tStart.tv_sec) * 1000000 + tEnd.tv_usec - tStart.tv_usec;
    tStart = tEnd;
    return t;
}


