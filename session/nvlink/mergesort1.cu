#include "hip/hip_runtime.h"
#if __linux__ && defined(__INTEL_COMPILER)
#define __sync_fetch_and_add(ptr,addend) _InterlockedExchangeAdd(const_cast<void*>(reinterpret_cast<volatile void*>(ptr)), addend)
#endif
#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>

#include <sys/socket.h>
#include <sys/time.h>
#include <netinet/in.h>
#include <arpa/inet.h>   

#include "tbb/concurrent_hash_map.h"
#include "tbb/blocked_range.h"
#include "tbb/parallel_for.h"
#include "tbb/tick_count.h"
#include "tbb/task_scheduler_init.h"
#include "tbb/concurrent_vector.h"
#include "utility.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "csv.hpp"
#include "timer.h"


using namespace tbb;
using namespace std;

std::vector<string> timestamp;

// data[], size, threads, blocks, 
void mergesort(long*, long, dim3, dim3);
// A[]. B[], size, width, slices, nThreads
__global__ void gpu_mergesort(long*, long*, long, long, long, dim3*, dim3*);
__device__ void gpu_bottomUpMerge(long*, long*, long, long, long);

#define min(a, b) (a < b ? a : b)

int main(int argc, char** argv) {

    dim3 threadsPerBlock;
    dim3 blocksPerGrid;

    threadsPerBlock.x = 32;
    threadsPerBlock.y = 1;
    threadsPerBlock.z = 1;

    blocksPerGrid.x = 8;
    blocksPerGrid.y = 1;
    blocksPerGrid.z = 1;


    const string csv_file = std::string(argv[1]); 
    vector<vector<string>> data; 

    // tm();

    Csv objCsv(csv_file);
    if (!objCsv.getCsv(data)) {
       cout << "read ERROR" << endl;
       return 1;
       }

    long size=atoi(argv[2]);
    size_t nBytes = size * sizeof(long);
    long *data2;
    data2 = (long *)malloc(nBytes);     

    // counter = 0;
    for (unsigned int row = 0; row < data.size(); row++) {
    	vector<string> rec = data[row];
	std::string tms = rec[0];

	for(size_t c = tms.find_first_of("\""); c != string::npos; c = c = tms.find_first_of("\"")){
    		          tms.erase(c,1);
	}

	for(size_t c = tms.find_first_of("/"); c != string::npos; c = c = tms.find_first_of("/")){
	             tms.erase(c,1);
	}

        for(size_t c = tms.find_first_of("."); c != string::npos; c = c = tms.find_first_of(".")){
	      tms.erase(c,1);
	}

	for(size_t c = tms.find_first_of(" "); c != string::npos; c = c = tms.find_first_of(" ")){
	      tms.erase(c,1);
	}

	for(size_t c = tms.find_first_of(":"); c != string::npos; c = c = tms.find_first_of(":")){
	      tms.erase(c,1);
	}

	data2[row] = stol(tms);

    }

    /*
    for(int i = 0; i < 10; i++)
    	    cout << data2[i] << endl;
    */

    //
    // Parse argv
    //
    /*
    //tm();
    
    long size=100000000;
    size_t nBytes = size * sizeof(long);
    long *data;
    data = (long *)malloc(nBytes);     

    time_t t;
    srand((unsigned) time(&t));
    for(int i=0; i< 10;i++)
    {
	data[i] = rand();     
    }
    */

    /*
    long size = readList(&data);
    if (!size) return -1;
    */

    std::cout << "sorting " << size << " numbers\n\n";
    
    // merge-sort the data
    mergesort(data2, size, threadsPerBlock, blocksPerGrid);

    /*
    for(int i = 0; i < 10; i++)
    	    cout << data2[i] << endl;
    */

    //tm();

    //
    // Print out the list
    //
    
    /*
    for (int i = 0; i < size; i++) {
        std::cout << data[i] << '\n';
    } 
    std::cout << "print list to stdout: " << tm() << " microseconds\n";
    */

}

void mergesort(long* data, long size, dim3 threadsPerBlock, dim3 blocksPerGrid) {

    //
    // Allocate two arrays on the GPU
    // we switch back and forth between them during the sort
    //
    long* D_data;
    long* D_swp;
    dim3* D_threads;
    dim3* D_blocks;
    
    // Actually allocate the two arrays
    //tm();
    hipMalloc((void**) &D_data, size * sizeof(long));
    hipMalloc((void**) &D_swp, size * sizeof(long));
        //std::cout << "hipMalloc device lists: " << tm() << " microseconds\n";

    // Copy from our input list into the first array
    hipMemcpy(D_data, data, size * sizeof(long), hipMemcpyHostToDevice);
        // std::cout << "hipMemcpy list to device: " << tm() << " microseconds\n";
 
    //
    // Copy the thread / block info to the GPU as well
    //
    hipMalloc((void**) &D_threads, sizeof(dim3));
    hipMalloc((void**) &D_blocks, sizeof(dim3));

        // std::cout << "hipMalloc device thread data: " << tm() << " microseconds\n";
    hipMemcpy(D_threads, &threadsPerBlock, sizeof(dim3), hipMemcpyHostToDevice);
    hipMemcpy(D_blocks, &blocksPerGrid, sizeof(dim3), hipMemcpyHostToDevice);

        // std::cout << "hipMemcpy thread data to device: " << tm() << " microseconds\n";

    long* A = D_data;
    long* B = D_swp;

    long nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
                    blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;

    //
    // Slice up the list and give pieces of it to each thread, letting the pieces grow
    // bigger and bigger until the whole list is sorted
    //
    for (int width = 2; width < (size << 1); width <<= 1) {
        long slices = size / ((nThreads) * width) + 1;

            std::cout << "mergeSort - width: " << width 
                      << ", slices: " << slices 
                      << ", nThreads: " << nThreads << '\n';
            // tm();

        // Actually call the kernel
        gpu_mergesort<<<blocksPerGrid, threadsPerBlock>>>(A, B, size, width, slices, D_threads, D_blocks);

            //std::cout << "call mergesort kernel: " << tm() << " microseconds\n";

        // Switch the input / output arrays instead of copying them around
        A = A == D_data ? D_swp : D_data;
        B = B == D_data ? D_swp : D_data;
    }

    //
    // Get the list back from the GPU
    //
    // tm();
    hipMemcpy(data, A, size * sizeof(long), hipMemcpyDeviceToHost);
        // std::cout << "hipMemcpy list back to host: " << tm() << " microseconds\n";
    
    
    // Free the GPU memory
    hipFree(A);
    hipFree(B);
        //std::cout << "hipFree: " << tm() << " microseconds\n";
}

// GPU helper function
// calculate the id of the current thread
__device__ unsigned int getIdx(dim3* threads, dim3* blocks) {
    int x;
    return threadIdx.x +
           threadIdx.y * (x  = threads->x) +
           threadIdx.z * (x *= threads->y) +
           blockIdx.x  * (x *= threads->z) +
           blockIdx.y  * (x *= blocks->z) +
           blockIdx.z  * (x *= blocks->y);
}

//
// Perform a full mergesort on our section of the data.
//
__global__ void gpu_mergesort(long* source, long* dest, long size, long width, long slices, dim3* threads, dim3* blocks) {
    unsigned int idx = getIdx(threads, blocks);
    long start = width*idx*slices, 
         middle, 
         end;

    for (long slice = 0; slice < slices; slice++) {
        if (start >= size)
            break;

        middle = min(start + (width >> 1), size);
        end = min(start + width, size);
        gpu_bottomUpMerge(source, dest, start, middle, end);
        start += width;
    }
}

//
// Finally, sort something
// gets called by gpu_mergesort() for each slice
//
__device__ void gpu_bottomUpMerge(long* source, long* dest, long start, long middle, long end) {
    long i = start;
    long j = middle;
    for (long k = start; k < end; k++) {
        if (i < middle && (j >= end || source[i] < source[j])) {
            dest[k] = source[i];
            i++;
        } else {
            dest[k] = source[j];
            j++;
        }
    }
}

// read data into a minimal linked list
typedef struct {
    int v;
    void* next;
} LinkNode;

// helper function for reading numbers from stdin
// it's 'optimized' not to check validity of the characters it reads in..
long readList(long** list) {
    //tm();
    long v, size = 0;
    LinkNode* node = 0;
    LinkNode* first = 0;
    while (std::cin >> v) {
        LinkNode* next = new LinkNode();
        next->v = v;
        if (node)
            node->next = next;
        else 
            first = next;
        node = next;
        size++;
    }


    if (size) {
        *list = new long[size]; 
        LinkNode* node = first;
        long i = 0;
        while (node) {
            (*list)[i++] = node->v;
            node = (LinkNode*) node->next;
        }

    }

        //std::cout << "read stdin: " << tm() << " microseconds\n";

    return size;
}


// 
// Get the time (in microseconds) since the last call to tm();
// the first value returned by this must not be trusted
//
timeval tStart;
int tm() {
    timeval tEnd;
    gettimeofday(&tEnd, 0);
    int t = (tEnd.tv_sec - tStart.tv_sec) * 1000000 + tEnd.tv_usec - tStart.tv_usec;
    tStart = tEnd;
    return t;
}


