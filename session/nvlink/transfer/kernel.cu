#include <algorithm>
#include <cfloat>
#include <chrono>
#include <fstream>
#include <iostream>
#include <random>
#include <sstream>
#include <vector>
#include <boost/tokenizer.hpp>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>
#include <random>
#include "timer.h"

using namespace std;

void transfer(unsigned long long *key, long *value, int kBytes, int vBytes, int thread_id)
{
    unsigned long long *d_A;
    long *d_B;
    unsigned int t, travdirtime;

    start_timer(&t);

    hipSetDevice(thread_id);
    hipMalloc((unsigned long long**)&d_A, kBytes);
    hipMalloc((long**)&d_B, vBytes);
    hipMemcpy(d_A, key, kBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, value, vBytes, hipMemcpyHostToDevice);

    travdirtime = stop_timer(&t);
    print_timer(travdirtime);

}

void kernel(long *h_key, long *h_value_1, long *h_value_2, string filename, int size)
{
  int N = size;

  cout << "kernel" << endl;

  for(int i = 0; i < 5; i++)
  {
	cout << h_key[i] << "," << h_value_1[i] << endl;
  }

  thrust::host_vector<int> h_vec_1(N);
  std::generate(h_vec_1.begin(), h_vec_1.end(), rand); 

  thrust::device_vector<int> key_in(N);
  thrust::copy(h_vec_1.begin(), h_vec_1.end(), key_in.begin()); 

  thrust::host_vector<unsigned long long> h_vec_key_1(N);
  thrust::host_vector<unsigned long long> h_vec_key_2(N);

  thrust::host_vector<long> h_vec_value_1(N);
  thrust::host_vector<long> h_vec_value_2(N);

  cout << N << endl;

  for(int i=0; i < N; i++)
  {
	// cout << h_key[i] << endl;
	h_vec_key_1[i] = h_key[i];
	h_vec_key_2[i] = h_key[i];
	h_vec_value_1[i] = h_value_1[i];
	h_vec_value_2[i] = h_value_2[i];
  }

  /* 1 -> 3 */

  thrust::device_vector<unsigned long long> d_vec_key_1(N);
  thrust::device_vector<long> d_vec_value_1(N);
  thrust::copy(h_vec_key_1.begin(), h_vec_key_1.end(), d_vec_key_1.begin());
  thrust::copy(h_vec_value_1.begin(), h_vec_value_1.end(), d_vec_value_1.begin());

  // thrust::sort_by_key(d_vec_key_1.begin(), d_vec_key_1.end(), d_vec_value_1.begin(), thrust::greater<unsigned long long>());

  thrust::sort_by_key(d_vec_key_1.begin(), d_vec_key_1.end(), d_vec_value_1.begin());

  thrust::host_vector<unsigned long long> h_vec_key_3(N);
  thrust::host_vector<long> h_vec_value_3(N);

  thrust::copy(d_vec_value_1.begin(),d_vec_value_1.end(),h_vec_value_3.begin());
  thrust::copy(d_vec_key_1.begin(),d_vec_key_1.end(),h_vec_key_3.begin());

  /* 2 -> 4 */

  thrust::device_vector<unsigned long long> d_vec_key_2(N);
  thrust::device_vector<long> d_vec_value_2(N);
  thrust::copy(h_vec_key_2.begin(), h_vec_key_2.end(), d_vec_key_2.begin());
  thrust::copy(h_vec_value_2.begin(), h_vec_value_2.end(), d_vec_value_2.begin());

  // thrust::sort_by_key(d_vec_key_2.begin(), d_vec_key_2.end(), d_vec_value_2.begin(), thrust::greater<unsigned long long>());

  thrust::sort_by_key(d_vec_key_2.begin(), d_vec_key_2.end(), d_vec_value_2.begin());

  thrust::host_vector<unsigned long long> h_vec_key_4(N);
  thrust::host_vector<long> h_vec_value_4(N);

  thrust::copy(d_vec_value_2.begin(),d_vec_value_2.end(),h_vec_value_4.begin());
  thrust::copy(d_vec_key_2.begin(),d_vec_key_2.end(),h_vec_key_4.begin());

  cout << "1 -> 3" << endl;
  for(int i = 0; i < 5; i++)
  {
	cout << h_vec_key_3[i] << "," << h_vec_value_3[i] << endl;
  }

  cout << "2 -> 4" << endl;
  for(int i = 0; i < 5; i++)
  {
	cout << h_vec_key_4[i] << "," << h_vec_value_4[i] << endl;
  }

  thrust::device_vector<unsigned long long> d_vec_key_1_out(N);
  thrust::device_vector<long> d_vec_value_1_out(N);

  thrust::device_vector<unsigned long long> d_vec_key_2_out(N);
  thrust::device_vector<long> d_vec_value_2_out(N);

  auto new_end_1 = thrust::reduce_by_key(d_vec_key_1.begin(), d_vec_key_1.end(), d_vec_value_1.begin(),
       	       	 		       d_vec_key_1_out.begin(), d_vec_value_1_out.begin());

  int new_size_1 = new_end_1.first - d_vec_key_1_out.begin() + 1; 

  thrust::host_vector<unsigned long long> h_vec_key_3_out(N);
  thrust::host_vector<long> h_vec_value_3_out(N);

  thrust::copy(d_vec_value_1_out.begin(),d_vec_value_1_out.end(),h_vec_value_3_out.begin());
  thrust::copy(d_vec_key_1_out.begin(),d_vec_key_1_out.end(),h_vec_key_3_out.begin());

  auto new_end_2 = thrust::reduce_by_key(d_vec_key_2.begin(), d_vec_key_2.end(), d_vec_value_2.begin(),
       	       	 		       d_vec_key_2_out.begin(), d_vec_value_2_out.begin());      

  int new_size_2 = new_end_2.first - d_vec_key_2_out.begin();// + 1; 

  thrust::host_vector<unsigned long long> h_vec_key_4_out(N);
  thrust::host_vector<long> h_vec_value_4_out(N);

  thrust::copy(d_vec_value_2_out.begin(),d_vec_value_2_out.end(),h_vec_value_4_out.begin());
  thrust::copy(d_vec_key_2_out.begin(),d_vec_key_2_out.end(),h_vec_key_4_out.begin());

  cout << "1 -> 3" << endl;
  for(int i = 0; i < 5; i++)
  {
	cout << h_vec_key_3_out[i] << "," << h_vec_value_3_out[i] << endl;
  }

  cout << "2 -> 4" << endl;
  for(int i = 0; i < 5; i++)
  {
	cout << h_vec_key_4_out[i] << "," << h_vec_value_4_out[i] << endl;
  }

  ofstream outputfile(filename);
    
  cout << "all" << endl;

  outputfile << "timestamp, counted, bytes" << endl;
  
  for(int i = 0; i < new_size_2; i++)
  {
	// cout << h_vec_key_3_out[i] << "," << h_vec_value_3_out[i] << "," << h_vec_value_4_out[i] << endl;

	/*
	if(h_vec_key_3_out[i] != 0)
		outputfile << h_vec_key_3_out[i] << "," << h_vec_value_3_out[i] << "," << h_vec_value_4_out[i] << endl;
	*/
	
	std::string timestamp = to_string(h_vec_key_3_out[i]);

	outputfile << timestamp.substr(0,4) << "-" << timestamp.substr(4,2) << "-" << timestamp.substr(6,2) << " "
	     	   << timestamp.substr(8,2) << ":" << timestamp.substr(10,2) << ":" << timestamp.substr(12,2)
	     	   << "." << timestamp.substr(14,3) << "," 
		   << h_vec_value_3_out[i] << "," << h_vec_value_4_out[i] << endl;
  }

  outputfile.close();
  
}


