#include <algorithm>
#include <cfloat>
#include <chrono>
#include <fstream>
#include <iostream>
#include <random>
#include <sstream>
#include <vector>
#include <boost/tokenizer.hpp>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>
#include <random>

#include "timer.h"

using namespace std;

std::vector < std::vector< std::string > > parse_csv(const char* filepath)
{
    std::vector< std::vector< std::string > > cells;
    std::string line;
    std::ifstream ifs(filepath);

    // csvを走査
    while (std::getline(ifs, line)) {

        std::vector< std::string > data;

        // 1行を走査
        boost::tokenizer< boost::escaped_list_separator< char > > tokens(line);
        for (const std::string& token : tokens) {
            data.push_back(token);
        }

        // 1行読み込んだ結果を入れる
        cells.push_back(data);
    }

    return cells;
}

int main(int argc, const char* argv[])
{
  int N = atoi(argv[2]);
  
  int counter = 0;
  int ngpus = 4;
  const size_t ullBytes = N * sizeof(unsigned long long);  
  const size_t lBytes = N * sizeof(long);  

  int RATIO = 100000;
  unsigned int t, travdirtime; 

  unsigned long long **d_A = (unsigned long long **)malloc(sizeof(unsigned long long *) * ngpus);
  long **d_B = (long **)malloc(sizeof(long *) * ngpus);
  
  unsigned long long *h_A;
  long *h_B;  

  h_A = (unsigned long long *)malloc(ullBytes);
  h_B = (long *)malloc(lBytes);

  hipStream_t *stream = (hipStream_t *)malloc(sizeof(hipStream_t) * ngpus); 

  for (int i = 0; i < ngpus; i++)
  {
	hipSetDevice(i);
        hipMalloc((void **) &d_A[i], ullBytes);
	hipMalloc((void **) &d_B[i], lBytes);
        hipStreamCreate(&stream[i]);
  }

  const auto cells = parse_csv(argv[1]);

  counter = 0;
  for (const auto& rows : cells) {
      h_A[counter] = std::stoull(rows[0]);
      h_B[counter] = 1;

      if(counter % RATIO == 0)
      {
	cout << "stored " << counter / RATIO << "..." << endl;
      }

      counter = counter + 1;
  }

  thrust::host_vector<unsigned long long> h_key_in(N);
  thrust::host_vector<long> h_value_in(N);

  for(int i = 0; i < N; i++)
  {
      h_key_in[i] = h_A[i];
      h_value_in[i] = h_B[i];
  }

  hipSetDevice(0);

  thrust::device_vector<unsigned long long> key_in_0(N); 
  thrust::device_vector<long> value_in_0(N); 

  thrust::device_vector<unsigned long long> key_in_1(N); 
  thrust::device_vector<long> value_in_1(N); 

  thrust::copy(h_key_in.begin(), h_key_in.end(), key_in_0.begin());
  thrust::copy(h_value_in.begin(), h_value_in.end(), value_in_0.begin());    

  thrust::device_vector<unsigned long long> dkey_out_0(h_key_in.size());
  thrust::device_vector<long> dvalue_out_0(h_value_in.size());

  thrust::sort(thrust::cuda::par.on(stream[0]), key_in_0.begin(), key_in_0.end());
  auto new_end_0 = thrust::reduce_by_key(thrust::cuda::par.on(stream[0]), key_in_0.begin(), key_in_0.end(),
       	       	 							value_in_0.begin(),
       	       	 				       			dkey_out_0.begin(),
									dvalue_out_0.begin());
									
  long new_size_0 = new_end_0.first - dkey_out_0.begin();

  const size_t ullBytes2 = new_size_0 * sizeof(unsigned long long);  
  const size_t lBytes2 = new_size_0 * sizeof(long);  

  unsigned long long *h_C;
  long *h_D;  

  h_C = (unsigned long long *)malloc(ullBytes2);
  h_D = (long *)malloc(lBytes2);

  for(int i = 0; i < new_size_0; i++)
  {
	h_C[i] = dkey_out_0[i];
	h_D[i] = dvalue_out_0[i];
  }

  unsigned long long **d_C = (unsigned long long **)malloc(sizeof(unsigned long long *) * ngpus);
  long **d_D = (long **)malloc(sizeof(long *) * ngpus);

  for (int i = 0; i < ngpus; i++)
  {
	hipSetDevice(i);
        hipMalloc((void **) &d_C[i], ullBytes2);
	hipMalloc((void **) &d_D[i], lBytes2);
  }

  hipMemcpy(d_C[0], h_C, ullBytes2, hipMemcpyHostToDevice);
  hipMemcpy(d_D[0], h_D, ullBytes2, hipMemcpyHostToDevice);

  hipDeviceEnablePeerAccess(0, 1);  

  cout << "[Device0][transfer] Device0 to Device1" << endl;
  start_timer(&t);
  hipMemcpy(d_C[1], d_D[0], ullBytes2, hipMemcpyDeviceToDevice);
  hipMemcpy(d_D[1], d_D[0], lBytes2, hipMemcpyDeviceToDevice);
  travdirtime = stop_timer(&t);
  print_timer(travdirtime);

  hipSetDevice(1);

  cout << "[Device1][store] array to device_vector" << endl;
  start_timer(&t);
  for(int i = 0; i < new_size_0; i++)
  {
	key_in_1[i] = d_C[1][i];
	value_in_1[i] = d_D[1][i];
  }
  travdirtime = stop_timer(&t);
  print_timer(travdirtime);

}
