// #include "../common/common.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

inline bool isCapableP2P(int ngpus)
{
    //cudaDeviceProp prop[ngpus];
    hipDeviceProp_t *prop = (hipDeviceProp_t *)malloc(ngpus * sizeof(hipDeviceProp_t));

    int iCount = 0;

    for (int i = 0; i < ngpus; i++)
    {
        hipGetDeviceProperties(&prop[i], i);

        if (prop[i].major >= 2) iCount++;

        printf("> GPU%d: %s %s capable of Peer-to-Peer access\n",
               i, prop[i].name, (prop[i].major >= 2 ? "is" : "not"));
    }

    if (iCount != ngpus)
    {
        printf("> no enough device to run this application\n");
    }

    return (iCount == ngpus);
}

inline void enableP2P(int ngpus)
{
    for (int i = 0; i < ngpus; i++)
    {
        hipSetDevice(i);

        for (int j = 0; j < ngpus; j++)
        {
            if (i == j) continue;

            int peer_access_available = 0;
            hipDeviceCanAccessPeer(&peer_access_available, i, j);

            if (peer_access_available)
            {
                hipDeviceEnablePeerAccess(j, 0);
                printf("> GPU%d enabled direct access to GPU%d\n", i, j);
            }
            else
            {
                printf("(%d, %d)\n", i, j );
            }
        }
    }
}

inline void disableP2P(int ngpus)
{
    for (int i = 0; i < ngpus; i++)
    {
        hipSetDevice(i);

        for (int j = 0; j < ngpus; j++)
        {
            if (i == j) continue;

            int peer_access_available = 0;
            hipDeviceCanAccessPeer(&peer_access_available, i, j);

            if (peer_access_available)
            {
                hipDeviceDisablePeerAccess(j);
                printf("> GPU%d disabled direct access to GPU%d\n", i, j);
            }
        }
    }
}

void initialData(float *ip, int size)
{
    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)rand() / (float)RAND_MAX;
    }
}

int main(int argc, char **argv)
{
    int ngpus;

    int N;
    N = atoi(argv[1]);

    // check device count
    // CHECK(cudaGetDeviceCount(&ngpus));
    // printf("> CUDA-capable device count: %i\n", ngpus);

    ngpus = 2;

    // check p2p capability
    isCapableP2P(ngpus);

    // get ngpus from command line
    /*
    if (argc > 1)
    {
        if (atoi(argv[1]) > ngpus)
        {
            fprintf(stderr, "Invalid number of GPUs specified: %d is greater "
                    "than the total number of GPUs in this platform (%d)\n",
                    atoi(argv[1]), ngpus);
            return 1;
        }

        ngpus = atoi(argv[1]);
    }
    */

    if (ngpus > 2)
    {
        fprintf(stderr, "No more than 2 GPUs supported\n");
        return 1;
    }

    if (ngpus > 1) enableP2P(ngpus);

    // Allocate buffers
    // int iSize = 1024 * 1024 * 16;
    int iSize = 1024 * 1024 * N;
    const size_t iBytes = iSize * sizeof(float);
    printf("\nAllocating buffers (%iMB on each GPU and CPU Host)...\n",
           int(iBytes / 1024 / 1024));

    float **d_src = (float **)malloc(sizeof(float) * ngpus);
    float **d_rcv = (float **)malloc(sizeof(float) * ngpus);
    float **h_src = (float **)malloc(sizeof(float) * ngpus);
    hipStream_t *stream = (hipStream_t *)malloc(sizeof(hipStream_t) * ngpus);

    // Create CUDA event handles
    hipEvent_t start, stop;
    hipSetDevice(0);
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for (int i = 0; i < ngpus; i++)
    {
        hipSetDevice(i);
        hipMalloc(&d_src[i], iBytes);
        hipMalloc(&d_rcv[i], iBytes);
        hipHostMalloc((void **) &h_src[i], iBytes, hipHostMallocDefault);

        hipStreamCreate(&stream[i]);
    }

    for (int i = 0; i < ngpus; i++)
    {
        initialData(h_src[i], iSize);
    }

    // unidirectional gmem copy
    hipSetDevice(0);
    hipEventRecord(start, 0);

    for (int i = 0; i < 100; i++)
    {
        if (i % 2 == 0)
        {
            hipMemcpy(d_src[1], d_src[0], iBytes, hipMemcpyDeviceToDevice);
        }
        else
        {
            hipMemcpy(d_src[0], d_src[1], iBytes, hipMemcpyDeviceToDevice);
        }
    }

    hipSetDevice(0);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsed_time_ms;
    hipEventElapsedTime(&elapsed_time_ms, start, stop);

    elapsed_time_ms /= 100.0f;
    printf("Ping-pong unidirectional cudaMemcpy:\t %8.2f ms ", elapsed_time_ms);
    printf("performance: %8.2f GB/s\n", (float)iBytes / (elapsed_time_ms * 1e6f));

    //  bidirectional asynchronous gmem copy
    hipEventRecord(start, 0);

    for (int i = 0; i < 100; i++)
    {
        hipMemcpyAsync(d_src[1], d_src[0], iBytes, hipMemcpyDeviceToDevice, stream[0]);
        hipMemcpyAsync(d_rcv[0], d_rcv[1], iBytes, hipMemcpyDeviceToDevice, stream[1]);
    }

    hipSetDevice(0);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    elapsed_time_ms = 0.0f;
    hipEventElapsedTime(&elapsed_time_ms, start, stop);

    elapsed_time_ms /= 100.0f;
    printf("Ping-pong bidirectional cudaMemcpyAsync:\t %8.2f ms ", elapsed_time_ms);
    printf("performance: %8.2f GB/s\n", (float)2.0f * iBytes / (elapsed_time_ms * 1e6f));

    disableP2P(ngpus);

    // free
    hipSetDevice(0);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    for (int i = 0; i < ngpus; i++)
    {
        hipSetDevice(i);
        hipFree(d_src[i]);
        hipFree(d_rcv[i]);
        hipStreamDestroy(stream[i]);
        hipDeviceReset();
    }

    exit(EXIT_SUCCESS);
}
