#include "hip/hip_runtime.h"
#if __linux__ && defined(__INTEL_COMPILER)
#define __sync_fetch_and_add(ptr,addend) _InterlockedExchangeAdd(const_cast<void*>(reinterpret_cast<volatile void*>(ptr)), addend)
#endif
#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>

#include <sys/socket.h>
#include <sys/time.h>
#include <netinet/in.h>
#include <arpa/inet.h>   

#include "tbb/concurrent_hash_map.h"
#include "tbb/blocked_range.h"
#include "tbb/parallel_for.h"
#include "tbb/tick_count.h"
#include "tbb/task_scheduler_init.h"
#include "tbb/concurrent_vector.h"
#include "utility.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/system/cuda/experimental/pinned_allocator.h>
#include <thrust/system/cuda/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <thrust/for_each.h>

#include "csv.hpp"
#include "timer.h"

using namespace tbb;
using namespace std;

std::vector<string> timestamp;

#define min(a, b) (a < b ? a : b)

typedef long mytype;
typedef thrust::host_vector<mytype, thrust::cuda::experimental::pinned_allocator<mytype> > pinnedVector1;

typedef thrust::host_vector<mytype, thrust::cuda::experimental::pinned_allocator<mytype> > pinnedVector2;

#define TV1 1
#define TV2 2

inline bool isCapableP2P(int ngpus)
{
    //hipDeviceProp_t prop[ngpus];
    hipDeviceProp_t *prop = (hipDeviceProp_t *)malloc(ngpus * sizeof(hipDeviceProp_t));

    int iCount = 0;

    for (int i = 0; i < ngpus; i++)
    {
        hipGetDeviceProperties(&prop[i], i);

        if (prop[i].major >= 2) iCount++;

        printf("> GPU%d: %s %s capable of Peer-to-Peer access\n",
               i, prop[i].name, (prop[i].major >= 2 ? "is" : "not"));
    }

    if (iCount != ngpus)
    {
        printf("> no enough device to run this application\n");
    }

    return (iCount == ngpus);
}

/*
 * enable P2P memcopies between GPUs (all GPUs must be compute capability 2.0 or
 * later (Fermi or later)).
 */
inline void enableP2P(int ngpus)
{
    for (int i = 0; i < ngpus; i++)
    {
        hipSetDevice(i);

        for (int j = 0; j < ngpus; j++)
        {
            if (i == j) continue;

            int peer_access_available = 0;
            hipDeviceCanAccessPeer(&peer_access_available, i, j);

            if (peer_access_available)
            {
                hipDeviceEnablePeerAccess(j, 0);
                printf("> GPU%d enabled direct access to GPU%d\n", i, j);
            }
            else
            {
                printf("(%d, %d)\n", i, j );
            }
        }
    }
}

inline void disableP2P(int ngpus)
{
    for (int i = 0; i < ngpus; i++)
    {
        hipSetDevice(i);

        for (int j = 0; j < ngpus; j++)
        {
            if (i == j) continue;

            int peer_access_available = 0;
            hipDeviceCanAccessPeer(&peer_access_available, i, j);

            if (peer_access_available)
            {
                hipDeviceDisablePeerAccess(j);
                printf("> GPU%d disabled direct access to GPU%d\n", i, j);
            }
        }
    }
}

int main(int argc, char** argv) {

    int N = atoi(argv[2]);
    unsigned int t, travdirtime; 

    hipSetDevice(0);
    pinnedVector1 hi1(N);
    pinnedVector2 ho1(N);

    hipSetDevice(1);
    pinnedVector1 hi2(N);
    pinnedVector2 ho2(N);

    enableP2P(2);

    const string csv_file = std::string(argv[1]); 
    vector<vector<string>> data; 

    Csv objCsv(csv_file);
    if (!objCsv.getCsv(data)) {
       cout << "read ERROR" << endl;
       return 1;
       }

    long size=atoi(argv[2]);

    for (unsigned int row = 0; row < data.size(); row++) {
    	vector<string> rec = data[row];

	std::string tms = rec[0];
	std::string bytes = rec[20];

	for(size_t c = tms.find_first_of("\""); c != string::npos; c = c = tms.find_first_of("\"")){
    	      tms.erase(c,1);
	}

	for(size_t c = tms.find_first_of("/"); c != string::npos; c = c = tms.find_first_of("/")){
	      tms.erase(c,1);
	}

        for(size_t c = tms.find_first_of("."); c != string::npos; c = c = tms.find_first_of(".")){
	      tms.erase(c,1);
	}

	for(size_t c = tms.find_first_of(" "); c != string::npos; c = c = tms.find_first_of(" ")){
	      tms.erase(c,1);
	}

	for(size_t c = tms.find_first_of(":"); c != string::npos; c = c = tms.find_first_of(":")){
	      tms.erase(c,1);
	}

	for(size_t c = bytes.find_first_of("\""); c != string::npos; c = c = bytes.find_first_of("\"")){
	      bytes.erase(c,1);
	}

	hi1.push_back(stol(tms));
	hi2.push_back(stol(tms));

	ho1.push_back(stol(bytes));
	ho2.push_back(stol(bytes));
    }

    hipSetDevice(0);
    thrust::device_vector<mytype> di1(N);
    thrust::device_vector<mytype> do1(N);

    hipSetDevice(1);
    thrust::device_vector<mytype> di2(N);
    thrust::device_vector<mytype> do2(N);

    thrust::device_vector<mytype> dNVLNK(N);

    hipSetDevice(0);
    thrust::fill(hi1.begin(), hi1.end(),  TV1);
    thrust::sequence(do1.begin(), do1.end());

    hipSetDevice(1);
    thrust::fill(hi2.begin(), hi2.end(),  TV2);
    thrust::sequence(do2.begin(), do2.end());

    hipStream_t s1, s2, s3;
  
    hipSetDevice(0);
    hipStreamCreate(&s1);
    
    hipSetDevice(1);
    hipStreamCreate(&s2);
    hipStreamCreate(&s3);

    cout << "H1 -> D1: Host -> GPU0" << endl;
    start_timer(&t);
    hipSetDevice(0);
    hipMemcpyAsync(thrust::raw_pointer_cast(di1.data()), thrust::raw_pointer_cast(hi1.data()), di1.size()*sizeof(mytype), hipMemcpyHostToDevice, s1);

    cout << "H2 -> D2: Host -> GPU1" << endl;
    hipMemcpyAsync(thrust::raw_pointer_cast(di2.data()), thrust::raw_pointer_cast(hi2.data()), di2.size()*sizeof(mytype), hipMemcpyHostToDevice, s2);

    hipDeviceSynchronize();

    travdirtime = stop_timer(&t);
    print_timer(travdirtime);

    cout << "D1-> GPU1: GPU0->GPU1" << endl;
    start_timer(&t);
    hipSetDevice(0);
    hipMemcpyAsync(thrust::raw_pointer_cast(dNVLNK.data()), thrust::raw_pointer_cast(di1.data()), dNVLNK.size()*sizeof(mytype), hipMemcpyDeviceToDevice, s2);

    travdirtime = stop_timer(&t);
    print_timer(travdirtime);
}



