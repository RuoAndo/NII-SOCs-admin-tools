
#include <omp.h>
#include <stdio.h> 
#include <stdlib.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <time.h>
#include <sys/time.h>

#define DSIZE 20000

using namespace std;

int main(int argc, char *argv[])
{
    timeval t1, t2;
    int num_gpus = 2;   // number of CUDA GPUs

    printf("%s Starting...\n\n", argv[0]);

    // determine the number of CUDA capable GPUs
    // hipGetDeviceCount(&num_gpus);

    if (num_gpus < 1)
    {
        printf("no CUDA capable devices were detected\n");
        return 1;
    }

    // display CPU and GPU configuration
    printf("number of host CPUs:\t%d\n", omp_get_num_procs());
    printf("number of CUDA devices:\t%d\n", num_gpus);

    for (int i = 0; i < num_gpus; i++)
    {
        hipDeviceProp_t dprop;
        hipGetDeviceProperties(&dprop, i);
        printf("   %d: %s\n", i, dprop.name);
    }

    printf("initialize data\n");


    // initialize data
    typedef thrust::device_vector<int> dvec;
    typedef dvec *p_dvec;
    std::vector<p_dvec> dvecs;

    for(unsigned int i = 0; i < num_gpus; i++) {
      hipSetDevice(i);
      p_dvec temp = new dvec(DSIZE);
      dvecs.push_back(temp);
      }

    thrust::host_vector<int> data(DSIZE);
    thrust::generate(data.begin(), data.end(), rand);

    // copy data
    for (unsigned int i = 0; i < num_gpus; i++) {
      hipSetDevice(i);
      thrust::copy(data.begin(), data.end(), (*(dvecs[i])).begin());
      }

    printf("start sort\n");
    gettimeofday(&t1,NULL);

    // run as many CPU threads as there are CUDA devices
    omp_set_num_threads(num_gpus);  // create as many CPU threads as there are CUDA devices
    #pragma omp parallel
    {
        unsigned int cpu_thread_id = omp_get_thread_num();
        hipSetDevice(cpu_thread_id);
        thrust::sort((*(dvecs[cpu_thread_id])).begin(), (*(dvecs[cpu_thread_id])).end());
        hipDeviceSynchronize();
    }
    gettimeofday(&t2,NULL);
    printf("finished\n");
    unsigned long et = ((t2.tv_sec * 1000000)+t2.tv_usec) - ((t1.tv_sec * 1000000) + t1.tv_usec);
    if (hipSuccess != hipGetLastError())
        printf("%s\n", hipGetErrorString(hipGetLastError()));
    printf("sort time = %fs\n", (float)et/(float)(1000000));
    // check results
    thrust::host_vector<int> result(DSIZE);
    thrust::sort(data.begin(), data.end());
    for (int i = 0; i < num_gpus; i++)
    {
        hipSetDevice(i);
        thrust::copy((*(dvecs[i])).begin(), (*(dvecs[i])).end(), result.begin());
        for (int j = 0; j < DSIZE; j++)
          if (data[j] != result[j]) { printf("mismatch on device %d at index %d, host: %d, device: %d\n", i, j, data[j], result[j]); return 1;}
    }
    printf("Success\n");
    return 0;

}
