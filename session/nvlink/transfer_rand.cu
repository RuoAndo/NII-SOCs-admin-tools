#include <algorithm>
#include <cfloat>
#include <chrono>
#include <fstream>
#include <iostream>
#include <random>
#include <sstream>
#include <vector>
#include <boost/tokenizer.hpp>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>

#include <random>
#include <functional> //for std::function
#include <algorithm>  //for std::generate_n

#include "timer.h"

using namespace std;

std::vector < std::vector< std::string > > parse_csv(const char* filepath)
{
    std::vector< std::vector< std::string > > cells;
    std::string line;
    std::ifstream ifs(filepath);

    while (std::getline(ifs, line)) {

        std::vector< std::string > data;

        boost::tokenizer< boost::escaped_list_separator< char > > tokens(line);
        for (const std::string& token : tokens) {
            data.push_back(token);
        }

        cells.push_back(data);
    }

    return cells;
}

int main(int argc, const char* argv[])
{

  std::random_device rnd;
  std::mt19937 mt(rnd());
  std::uniform_int_distribution<unsigned long long> randN(20190501000000000, 20190501235959000);
  
  int N = atoi(argv[1]);
  
  int counter = 0;
  int ngpus = 4;
  const size_t ullBytes = N * sizeof(unsigned long long);  
  const size_t lBytes = N * sizeof(long);  

  int RATIO = 100000;
  unsigned int t, travdirtime; 

  unsigned long long **d_A = (unsigned long long **)malloc(sizeof(unsigned long long *) * ngpus);
  long **d_B = (long **)malloc(sizeof(long *) * ngpus);
  
  unsigned long long *h_A;
  long *h_B;  

  h_A = (unsigned long long *)malloc(ullBytes);
  h_B = (long *)malloc(lBytes);

  hipStream_t *stream = (hipStream_t *)malloc(sizeof(hipStream_t) * ngpus); 

  for (int i = 0; i < ngpus; i++)
  {
	hipSetDevice(i);
        hipMalloc((void **) &d_A[i], ullBytes);
	hipMalloc((void **) &d_B[i], lBytes);
	
        hipStreamCreate(&stream[i]);
  }                  


  start_timer(&t);
  cout << "generating the random number " << N << "..." << endl;
  unsigned long long r;
  long r2;
  for (int i = 0; i < N; ++i) {    
      r = randN(mt);
      h_A[counter] = r;
      r = randN(mt);
      h_B[counter] = r2;
  }
  travdirtime = stop_timer(&t);
  print_timer(travdirtime);  

  /*
  thrust::device_vector<unsigned long long> d_key_in_0(N,0);
  thrust::device_vector<long> d_value_in_0(N,0);

  thrust::device_vector<unsigned long long> d_key_in_1(N,0);
  thrust::device_vector<long> d_value_in_1(N,0);
  */

  hipSetDevice(0);
  hipDeviceEnablePeerAccess(0, 1);  

  cout << "[transfer CPU->GPU] Host(h_A, h_B) to Device0(d_A[0], d_B[0])" << endl;
  start_timer(&t);
  hipMemcpy(d_A[0], h_A, ullBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_B[0], h_B, lBytes, hipMemcpyHostToDevice);
  travdirtime = stop_timer(&t);
  print_timer(travdirtime);  

  hipSetDevice(0);

  cout << "[transfer GPU->GPU] GPU0(d_A[0],d_B[0]) -> GPU1(d_A[1],d_B[1])" << endl;
  start_timer(&t);
  hipMemcpy(d_A[1], d_A[0], ullBytes, hipMemcpyDeviceToDevice);
  hipMemcpy(d_B[1], d_B[0], lBytes, hipMemcpyDeviceToDevice);
  travdirtime = stop_timer(&t);
  print_timer(travdirtime);  

  hipSetDevice(1);
  hipDeviceEnablePeerAccess(1, 2);  

  cout << "[transfer GPU->GPU] GPU1(d_A[1],d_B[1]) -> GPU2(d_A[2],d_B[2])" << endl;
  start_timer(&t);
  hipMemcpy(d_A[2], d_A[1], ullBytes, hipMemcpyDeviceToDevice);
  hipMemcpy(d_B[2], d_B[1], lBytes, hipMemcpyDeviceToDevice);
  travdirtime = stop_timer(&t);
  print_timer(travdirtime);  

  /*
  start_timer(&t);
  cout << "[copy] Array to device_vector in device 0" << endl;
  for(int i = 0; i < N; i++)
  {
  	  d_key_in_0[i] = d_A[0][i];
	  d_value_in_0[i] = d_B[0][i];
  }
  travdirtime = stop_timer(&t);
  print_timer(travdirtime);  
  */

  /*
  hipSetDevice(1);

  cout << "[copy] Array to device_vector in device 1" << endl;
  start_timer(&t);
  for(int i = 0; i < N; i++)
  {
  	  d_key_in_1[i] = d_A[1][i];
	  d_value_in_1[i] = d_B[1][i];
  }
  travdirtime = stop_timer(&t);
  print_timer(travdirtime);
  */

}
