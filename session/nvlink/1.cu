#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <random>
#include <iostream>
#include <sys/time.h>
#include <thrust/system/cuda/vector.h>
#include <thrust/system/cuda/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/generate.h>
#include <thrust/pair.h>
#include <cstdlib>
#include <iostream>
#include <map>
#include <cassert>


// This example demonstrates how to intercept calls to get_temporary_buffer
// and return_temporary_buffer to control how Thrust allocates temporary storage
// during algorithms such as thrust::sort. The idea will be to create a simple
// cache of allocations to search when temporary storage is requested. If a hit
// is found in the cache, we quickly return the cached allocation instead of
// resorting to the more expensive thrust::cuda::malloc.
//
// Note: this implementation cached_allocator is not thread-safe. If multiple
// (host) threads use the same cached_allocator then they should gain exclusive
// access to the allocator before accessing its methods.


// cached_allocator: a simple allocator for caching allocation requests
class cached_allocator
{
  public:
    // just allocate bytes
    typedef char value_type;

    cached_allocator() {}

    ~cached_allocator()
    {
      // free all allocations when cached_allocator goes out of scope
      free_all();
    }

    char *allocate(std::ptrdiff_t num_bytes)
    {
      char *result = 0;

      // search the cache for a free block
      free_blocks_type::iterator free_block = free_blocks.find(num_bytes);

      if(free_block != free_blocks.end())
      {
        std::cout << "cached_allocator::allocator(): found a hit" << std::endl;

        // get the pointer
        result = free_block->second;

        // erase from the free_blocks map
        free_blocks.erase(free_block);
      }
      else
      {
        // no allocation of the right size exists
        // create a new one with cuda::malloc
        // throw if cuda::malloc can't satisfy the request
        try
        {
          std::cout << "cached_allocator::allocator(): no free block found; calling cuda::malloc" << std::endl;

          // allocate memory and convert cuda::pointer to raw pointer
          result = thrust::cuda::malloc<char>(num_bytes).get();
        }
        catch(std::runtime_error &e)
        {
          throw;
        }
      }

      // insert the allocated pointer into the allocated_blocks map
      allocated_blocks.insert(std::make_pair(result, num_bytes));

      return result;
    }

    void deallocate(char *ptr, size_t n)
    {
      // erase the allocated block from the allocated blocks map
      allocated_blocks_type::iterator iter = allocated_blocks.find(ptr);
      std::ptrdiff_t num_bytes = iter->second;
      allocated_blocks.erase(iter);

      // insert the block into the free blocks map
      free_blocks.insert(std::make_pair(num_bytes, ptr));
    }

  private:
    typedef std::multimap<std::ptrdiff_t, char*> free_blocks_type;
    typedef std::map<char *, std::ptrdiff_t>     allocated_blocks_type;

    free_blocks_type      free_blocks;
    allocated_blocks_type allocated_blocks;

    void free_all()
    {
      std::cout << "cached_allocator::free_all(): cleaning up after ourselves..." << std::endl;

      // deallocate all outstanding blocks in both lists
      for(free_blocks_type::iterator i = free_blocks.begin();
          i != free_blocks.end();
          ++i)
      {
        // transform the pointer to cuda::pointer before calling cuda::free
        thrust::cuda::free(thrust::cuda::pointer<char>(i->second));
      }

      for(allocated_blocks_type::iterator i = allocated_blocks.begin();
          i != allocated_blocks.end();
          ++i)
      {
        // transform the pointer to cuda::pointer before calling cuda::free
        thrust::cuda::free(thrust::cuda::pointer<char>(i->first));
      }
    }

};




int main() {
    cached_allocator alloc;
    // size of arrays
    const int n = 300000000;

    // random number generator
    std::mt19937 rng;

    // key/val on host
    uint32_t * key = new uint32_t[n];
    uint32_t * val = new uint32_t[n];

    // fill key val
    for(int i = 0; i < n; i++) {
        key[i] = rng();
        val[i] = i;
    }

   // key/val on device
   uint32_t * dev_key;
   uint32_t * dev_val;

   // allocate memory on GPU for key/val
   hipMalloc((void**)&dev_key, n*sizeof(uint32_t));
   hipMalloc((void**)&dev_val, n*sizeof(uint32_t));

   // copy key/val onto the device
   hipMemcpy(dev_key, key, n*sizeof(uint32_t), hipMemcpyHostToDevice);
   hipMemcpy(dev_val, val, n*sizeof(uint32_t), hipMemcpyHostToDevice);

   // get thrust device pointers to key/val on device
   thrust::device_ptr<uint32_t> dev_key_ptr = thrust::device_pointer_cast(dev_key);
   thrust::device_ptr<uint32_t> dev_val_ptr = thrust::device_pointer_cast(dev_val);

   // data on host
   uint32_t * data;

   // allocate pinned memory for data on host
   hipHostMalloc((void**)&data, n*sizeof(uint32_t));

   // fill data with random numbers
   for(int i = 0; i < n; i++) {
       data[i] = rng();
   }

   // data on device
   uint32_t * dev_data;

   // allocate memory for data on the device
   hipMalloc((void**)&dev_data, n*sizeof(uint32_t));

   // for timing
   struct timeval t1, t2;

   // two streams
   hipStream_t stream1;
   hipStream_t stream2;

   // create streams
   hipStreamCreate(&stream1);
   hipStreamCreate(&stream2);

   for(int i = 0; i < 10; i++) {

       // Copy data into dev_data on stream 1 (nothing happening on stream 2 for now)
       gettimeofday(&t1, NULL);
           hipMemcpyAsync(dev_data, data, n*sizeof(uint32_t), hipMemcpyHostToDevice, stream1);
           hipDeviceSynchronize();
       gettimeofday(&t2, NULL);
       double t_copy = double(t2.tv_sec-t1.tv_sec)*1000.0 + double(t2.tv_usec-t1.tv_usec)/1000.0;


       // Sort_by_key on stream 2 (nothing hapenning on stream 1 for now)
       gettimeofday(&t1, NULL);
           thrust::sort_by_key(thrust::cuda::par(alloc).on(stream2), dev_key_ptr, dev_key_ptr + n, dev_val_ptr);
           hipDeviceSynchronize();
       gettimeofday(&t2, NULL);
       double t_sort = double(t2.tv_sec-t1.tv_sec)*1000.0 + double(t2.tv_usec-t1.tv_usec)/1000.0;

       // Overlap both operations
       gettimeofday(&t1, NULL);
           thrust::sort_by_key(thrust::cuda::par(alloc).on(stream2), dev_key_ptr, dev_key_ptr + n, dev_val_ptr);
           hipMemcpyAsync(dev_data, data, n*sizeof(uint32_t), hipMemcpyHostToDevice, stream1);
           hipDeviceSynchronize();
       gettimeofday(&t2, NULL);
       double t_both = double(t2.tv_sec-t1.tv_sec)*1000.0 + double(t2.tv_usec-t1.tv_usec)/1000.0;

       std::cout << "t_copy: " << t_copy << ", t_sort: " << t_sort << ", t_both1: " << t_both << std::endl;

   }

   // clean up
   hipStreamDestroy(stream1);
   hipStreamDestroy(stream2);
   hipHostFree(data);
   hipFree(dev_data);
   hipFree(dev_key);
   hipFree(dev_val);
   delete [] key;
   delete [] val;

}
