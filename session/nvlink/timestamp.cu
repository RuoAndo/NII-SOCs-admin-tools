#if __linux__ && defined(__INTEL_COMPILER)
#define __sync_fetch_and_add(ptr,addend) _InterlockedExchangeAdd(const_cast<void*>(reinterpret_cast<volatile void*>(ptr)), addend)
#endif
#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>

#include <sys/socket.h>
#include <netinet/in.h>
#include <arpa/inet.h>   

#include "tbb/concurrent_hash_map.h"
#include "tbb/blocked_range.h"
#include "tbb/parallel_for.h"
#include "tbb/tick_count.h"
#include "tbb/task_scheduler_init.h"
#include "tbb/concurrent_vector.h"
//  #include "tbb/tbb_allocator.hz"
#include "utility.h"

#include "csv.hpp"
#include "timer.h"

using namespace tbb;
using namespace std;

std::vector<string> timestamp;

std::vector<std::string> split_string_2(std::string str, char del) {
  int first = 0;
  int last = str.find_first_of(del);

  std::vector<std::string> result;

  while (first < str.size()) {
    std::string subStr(str, first, last - first);

    result.push_back(subStr);

    first = last + 1;
    last = str.find_first_of(del, first);

    if (last == std::string::npos) {
      last = str.size();
    }
  }

  return result;
}

int main( int argc, char* argv[] ) {

  int counter = 0;
  int N = atoi(argv[2]);  

  int ngpus = 4;
  int RATIO = 1000000;
  const size_t iBytes = N * sizeof(float);  

  unsigned int t, travdirtime; 
  
  struct in_addr inaddr;
  char *some_addr;

    try {
        tbb::tick_count mainStartTime = tbb::tick_count::now();
        srand(2);

        utility::thread_number_range threads(tbb::task_scheduler_init::default_num_threads,0);
        // Data = new MyString[N];

	float **d_A = (float **)malloc(sizeof(float *) * ngpus);
	float **d_B = (float **)malloc(sizeof(float *) * ngpus);
	float **d_C = (float **)malloc(sizeof(float *) * ngpus);

	float **h_A = (float **)malloc(sizeof(float *) * ngpus);
	float **h_B = (float **)malloc(sizeof(float *) * ngpus);    

	hipStream_t *stream = (hipStream_t *)malloc(sizeof(hipStream_t) * ngpus); 

	for (int i = 0; i < ngpus; i++)
	  {
	    hipSetDevice(i);
	    hipMalloc((void **) &d_A[i], iBytes);
	    hipMalloc((void **) &d_B[i], iBytes);
	    hipMalloc((void **) &d_C[i], iBytes);

	    hipHostMalloc((void **) &h_A[i], iBytes);
	    hipHostMalloc((void **) &h_B[i], iBytes);

	    hipStreamCreate(&stream[i]);
	  }                  

	const string csv_file = std::string(argv[1]); 
	vector<vector<string>> data; 

	start_timer(&t);
	
	try {
	  Csv objCsv(csv_file);
	  if (!objCsv.getCsv(data)) {
	    cout << "read ERROR" << endl;
	    return 1;
	  }

	  for (unsigned int row = 0; row < data.size(); row++) {
	    vector<string> rec = data[row]; 
	    std::string tms = rec[0];

	    for(size_t c = tms.find_first_of("\""); c != string::npos; c = c = tms.find_first_of("\"")){
	      tms.erase(c,1);
	    }

	    for(size_t c = tms.find_first_of("/"); c != string::npos; c = c = tms.find_first_of("/")){
	      tms.erase(c,1);
	    }

	    for(size_t c = tms.find_first_of("."); c != string::npos; c = c = tms.find_first_of(".")){
	      tms.erase(c,1);
	    }

	    for(size_t c = tms.find_first_of(" "); c != string::npos; c = c = tms.find_first_of(" ")){
	      tms.erase(c,1);
	    }

	    for(size_t c = tms.find_first_of(":"); c != string::npos; c = c = tms.find_first_of(":")){
	      tms.erase(c,1);
	    }

	    h_A[0][row] = stof(tms);

	    if(row % RATIO == 0)
	      {
		cout << "stored " << row / RATIO << "..." << endl; 
	      }
		  
	    // timestamp.push_back(tms);
	  }
	}
	catch (...) {
	  cout << "EXCEPTION!" << endl;
	  return 1;
	}

	travdirtime = stop_timer(&t);
	print_timer(travdirtime);  
	
	hipSetDevice(0);
        hipDeviceEnablePeerAccess(0, 1);
	
	start_timer(&t);
	hipMemcpy(d_A[0], h_A[0], iBytes, hipMemcpyHostToDevice);
	travdirtime = stop_timer(&t);
	print_timer(travdirtime);  

	start_timer(&t);
	hipMemcpy(d_A[1], d_A[0], iBytes, hipMemcpyDeviceToDevice);
	travdirtime = stop_timer(&t);
	print_timer(travdirtime);  
  
  
	/*
	std::remove("writethrough-timestamp");
	ofstream outputfile("writethrough-timestamp");
	
	for(auto itr = timestamp.begin(); itr != timestamp.end(); ++itr) {
	  outputfile  << *itr << std::endl;
	}
      
	outputfile.close();
	*/	


        utility::report_elapsed_time((tbb::tick_count::now() - mainStartTime).seconds());
       
        return 0;
	
    } catch(std::exception& e) {
        std::cerr<<"error occurred. error text is :\"" <<e.what()<<"\"\n";
    }
}
