// #include "../common/common.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

inline bool isCapableP2P(int ngpus)
{
    hipDeviceProp_t *prop = (hipDeviceProp_t *)malloc(ngpus * sizeof(hipDeviceProp_t));

    int iCount = 0;

    for (int i = 0; i < ngpus; i++)
    {
        hipGetDeviceProperties(&prop[i], i);

        if (prop[i].major >= 2) iCount++;

        printf("> GPU%d: %s %s capable of Peer-to-Peer access\n",
               i, prop[i].name, (prop[i].major >= 2 ? "is" : "not"));
    }

    if (iCount != ngpus)
    {
        printf("> no enough device to run this application\n");
    }

    return (iCount == ngpus);
}

inline void enableP2P(int ngpus)
{
    for (int i = 0; i < ngpus; i++)
    {
        hipSetDevice(i);

        for (int j = 0; j < ngpus; j++)
        {
            if (i == j) continue;

            int peer_access_available = 0;
            hipDeviceCanAccessPeer(&peer_access_available, i, j);

            if (peer_access_available)
            {
                hipDeviceEnablePeerAccess(j, 0);
                printf("> GPU%d enabled direct access to GPU%d\n", i, j);
            }
            else
            {
                printf("(%d, %d)\n", i, j );
            }
        }
    }
}

inline void disableP2P(int ngpus)
{
    for (int i = 0; i < ngpus; i++)
    {
        hipSetDevice(i);

        for (int j = 0; j < ngpus; j++)
        {
            if (i == j) continue;

            int peer_access_available = 0;
            hipDeviceCanAccessPeer(&peer_access_available, i, j);

            if (peer_access_available)
            {
                hipDeviceDisablePeerAccess(j);
                printf("> GPU%d disabled direct access to GPU%d\n", i, j);
            }
        }
    }
}

void initialData(float *ip, int size)
{
    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)rand() / (float)RAND_MAX;
    }
}

int main(int argc, char **argv)
{
    int ngpus;

    int N;
    N = atoi(argv[1]);

    ngpus = 3;
    isCapableP2P(ngpus);

    printf("ngpus %d \n", ngpus);

    if (ngpus > 1) enableP2P(ngpus);

    int iSize = 1024 * 1024 * N;
    const size_t iBytes = iSize * sizeof(float);
    printf("\nAllocating buffers (%iMB on each GPU and CPU Host)...\n",
           int(iBytes / 1024 / 1024));

    float **d_src = (float **)malloc(sizeof(float) * ngpus);
    float **d_rcv = (float **)malloc(sizeof(float) * ngpus);
    float **h_src = (float **)malloc(sizeof(float) * ngpus);
    hipStream_t *stream = (hipStream_t *)malloc(sizeof(hipStream_t) * ngpus);

    hipEvent_t start, stop;
    hipSetDevice(0);
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for (int i = 0; i < ngpus; i++)
    {
        hipSetDevice(i);
        hipMalloc(&d_src[i], iBytes);
        hipMalloc(&d_rcv[i], iBytes);
        hipHostMalloc((void **) &h_src[i], iBytes, hipHostMallocDefault);
        hipStreamCreate(&stream[i]);
    }

    /*
    for (int i = 0; i < ngpus; i++)
    {
        initialData(h_src[i], iSize);
    }
    */

    hipSetDevice(0);
    hipEventRecord(start, 0);

    /*
    for (int i = 0; i < 100; i++)
    {
        if (i % 2 == 0)
        {
            cudaMemcpy(d_src[1], d_src[2], iBytes, cudaMemcpyDeviceToDevice);
        }
        else
        {
            cudaMemcpy(d_src[2], d_src[1], iBytes, cudaMemcpyDeviceToDevice);
        }
    }
    */

    float elapsed_time_ms;

    for (int i = 0; i < ngpus; i++)
    {
        for (int j = 0; j < ngpus; j++)
	{
		hipEventRecord(start, 0);
		// printf("GPU%d -> GPU%d \n", i, j);
		hipMemcpy(d_src[i], d_src[j], iBytes, hipMemcpyDeviceToDevice);
		hipEventRecord(stop, 0);
    		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed_time_ms, start, stop);
		elapsed_time_ms /= 100.0f;
		printf("GPU%d->GPU%d: performance: %8.2f GB/s\n", i, j, (float)iBytes / (elapsed_time_ms * 1e6f));		
	}
    }

    // cudaEventRecord(stop, 0);
    // cudaEventSynchronize(stop);

    // float elapsed_time_ms;
    // cudaEventElapsedTime(&elapsed_time_ms, start, stop);

    // elapsed_time_ms /= 100.0f;
    // printf("Ping-pong unidirectional cudaMemcpy:\t %8.2f ms ", elapsed_time_ms);
    // printf("performance: %8.2f GB/s\n\n", (float)iBytes / (elapsed_time_ms * 1e6f));

    hipSetDevice(0);
    disableP2P(ngpus);

    // free
    hipSetDevice(0);

    for (int i = 0; i < ngpus; i++)
    {
        hipSetDevice(i);
        hipFree(d_src[i]);
        hipFree(d_rcv[i]);
        hipStreamDestroy(stream[i]);
        // cudaDeviceReset();
    }

    exit(EXIT_SUCCESS);
}
