#include "hip/hip_runtime.h"
#if __linux__ && defined(__INTEL_COMPILER)
#define __sync_fetch_and_add(ptr,addend) _InterlockedExchangeAdd(const_cast<void*>(reinterpret_cast<volatile void*>(ptr)), addend)
#endif
#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>
#include <map>

#include <sys/socket.h>
#include <netinet/in.h>
#include <arpa/inet.h>

#include "csv.hpp"
#include "timer.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include "util.h"

using namespace std;
// static MyString* Data;

void sumArraysOnHost(int *A, int *B, float *C, const int N)
{
    for (int idx = 0; idx < N; idx++)
    {
        C[idx] = A[idx] / B[idx];
    }
}

__global__ void sumArraysOnGPU(int *A, int *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) C[i] = A[i] / B[i];
}

int main( int argc, char* argv[] ) {

  int N = atoi(argv[2]);  
  unsigned int t, travdirtime; 

  size_t nBytes = N * sizeof(int);
  size_t fBytes = N * sizeof(float);

  int *h_Recv, *h_nRecv;
  float *h_C;
  
  h_Recv = (int *)malloc(nBytes);
  h_nRecv = (int *)malloc(nBytes);
  h_C = (float *)malloc(fBytes);

  const string csv_file = std::string(argv[1]); 
  vector<vector<string>> data; 

  try {
      	  Csv objCsv(csv_file);
	  if (!objCsv.getCsv(data)) {
	    cout << "read ERROR" << endl;
	    return 1;
	    }

	  for (int row = 0; row < data.size(); row++) {
	    vector<string> rec = data[row];

	    for(size_t c = rec[17].find_first_of("\""); c != string::npos; c = c = rec[17].find_first_of("\"")){
	      rec[17].erase(c,1);
	      }

	    for(size_t c = rec[20].find_first_of("\""); c != string::npos; c = c = rec[20].find_first_of("\"")){
	      rec[20].erase(c,1);
	      }

	    h_nRecv[row] = atoi(rec[17].c_str());
	    h_Recv[row] = atoi(rec[20].c_str());
	  }

	  int *d_Recv, *d_nRecv;
	  float *d_C;
    	  hipMalloc((int**)&d_Recv, nBytes);
    	  hipMalloc((int**)&d_nRecv, nBytes);
    	  hipMalloc((float**)&d_C, fBytes);

	  hipMemcpy(d_Recv, h_Recv, nBytes, hipMemcpyHostToDevice);
    	  hipMemcpy(d_nRecv, h_nRecv, nBytes, hipMemcpyHostToDevice);

          start_timer(&t);  
	  sumArraysOnHost(h_Recv, h_nRecv, h_C, N);
          travdirtime = stop_timer(&t);
          print_timer(travdirtime);

	  for(int i=0;i<5;i++)
		printf("%d %d %f \n", h_Recv[i], h_nRecv[i], h_C[i]);

	  dim3 block (1);
          dim3 grid  (N);
	  
          start_timer(&t);  
          sumArraysOnGPU<<<grid, block>>>(d_Recv, d_nRecv, d_C, N);
          printf("Execution configure <<<%d, %d>>>\n", grid.x, block.x);
          travdirtime = stop_timer(&t);
          print_timer(travdirtime);

          hipMemcpy(h_C, d_C, nBytes, hipMemcpyDeviceToHost);

    	  hipFree(d_Recv);
    	  hipFree(d_nRecv);
    	  hipFree(d_C);

	  for(int i=0;i<5;i++)
		printf("%d %d %f \n", h_Recv[i], h_nRecv[i], h_C[i]);

          free(h_Recv);
          free(h_nRecv);

	  hipDeviceReset();


   }
   catch (...) {
	  cout << "EXCEPTION!" << endl;
	  return 1;
   }
	
   // delete[] Data;       
   return 0;
}
