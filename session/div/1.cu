#include<iostream>
#include<algorithm>
#include<iomanip>
#include<time.h>
#include<thrust/host_vector.h>
#include<thrust/device_vector.h>
#include<thrust/sort.h>

#define N (8<<10)
#define COUNT_LIMIT 10000

template<class T>
class plusOne{
public:
    __device__ __host__ T operator() (T a){
        return a+1;
    }
};

int main(){
    srand(time(NULL));
    thrust::host_vector<int> host_vector(N);
    std::generate(host_vector.begin(),host_vector.end(),rand);
    thrust::device_vector<int> device_vector=host_vector;

    hipEvent_t start,stop;
    float elapsed;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    hipEventRecord(start,0);
    for(int c=0;c<COUNT_LIMIT;c++){
        thrust::transform(device_vector.begin(),device_vector.end(),device_vector.begin(),plusOne<int>());
    }
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed,start,stop);

    std::cout<<"gpu :"<<elapsed<<"ms ["<<std::setprecision(8)<<COUNT_LIMIT/elapsed<<"/ms]"<<std::endl;

    std::generate(host_vector.begin(),host_vector.end(),rand);
    device_vector=host_vector;

    hipEventRecord(start,0);
    for(int c=0;c<COUNT_LIMIT;c++){
        thrust::transform(host_vector.begin(),host_vector.end(),host_vector.begin(),plusOne<int>());
    }
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed,start,stop);

    std::cout<<"cpu :"<<elapsed<<"ms ["<<std::setprecision(8)<<COUNT_LIMIT/elapsed<<"/ms]"<<std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
