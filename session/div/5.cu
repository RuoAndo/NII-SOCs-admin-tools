#include "hip/hip_runtime.h"
#if __linux__ && defined(__INTEL_COMPILER)
#define __sync_fetch_and_add(ptr,addend) _InterlockedExchangeAdd(const_cast<void*>(reinterpret_cast<volatile void*>(ptr)), addend)
#endif
#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>
#include <map>

#include <sys/socket.h>
#include <netinet/in.h>
#include <arpa/inet.h>

#include "csv.hpp"
#include "timer.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include "util.h"

using namespace std;
// static MyString* Data;

__global__ void bitonic_sort(unsigned long long *data, int data_size, int chunk_size, int sub_chunk_size, int *key)
{
    // int thread_index = blockDim.x * blockIdx.x + threadIdx.x;
    int thread_index = threadIdx.x;

    if (thread_index < data_size / 2) {
            int half_chunk_size = chunk_size / 2;
	            int chunk_index = thread_index / half_chunk_size;

        int half_sub_chunk_size = sub_chunk_size / 2;
	        int sub_chunk_index = thread_index / half_sub_chunk_size;

        bool up = (chunk_index % 2 == 0);
	        int a = sub_chunk_size * sub_chunk_index + thread_index % half_sub_chunk_size;
		int b = a + half_sub_chunk_size;

        unsigned long long va = data[a];
	        unsigned long long vb = data[b];
		        if (va > vb == up) {
			            data[a] = vb;
				    data[b] = va;
				    key[a] = key[b];
		}
	}
}

__global__ void sumArraysOnGPU(unsigned long long *A, int *B, const int N)
{
    // __shared__ extern int F[];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // int idx = threadIdx.x;

    for(int i=0; i < N; i++)
    {
	if(A[idx] == A[i])
	{
		B[idx]++;
	}
    }

    // __syncthreads();
    // B[idx] = F[idx];
}

__device__ void gpu_bottomUpMerge(unsigned long long* source, unsigned long long* dest,
                                  long start, long middle,
				  long end, int *skeys, int *dkeys) {
     int idx = blockIdx.x * blockDim.x + threadIdx.x;
     long i = start;
     long j = middle;

     for (long k = start; k < end; k++) {
     	 if (i < middle && (j >= end || source[i] < source[j])) {
	    dest[k] = source[i];
	    dkeys[k] = skeys[i];
	    i++;
	    }
	 else {
     	    dest[k] = source[j];
     	    j++;
     	    }
     }
}

__global__ void gpu_mergesort(unsigned long long* source, unsigned long long* dest, long size,
                              long width, long slices, int *dkeys, int *skeys) {
			      
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   long start = width*idx*slices, middle, end;

    for (long slice = 0; slice < slices; slice++) {
            if (start >= size)
	                break;
	    middle = min(start + (width >> 1), size);
	    end = min(start + width, size);
	    gpu_bottomUpMerge(source, dest, start, middle, end, dkeys, skeys);
	    start += width;
	    }
}

int main( int argc, char* argv[] ) {

  int N = atoi(argv[2]);  
  unsigned int t, travdirtime; 
  
  size_t nBytes = N * sizeof(unsigned long long);
  size_t iBytes = N * sizeof(int);

  unsigned long long *h_timestamp;
  int *h_count;
  h_timestamp = (unsigned long long *)malloc(nBytes);
  h_count = (int *)malloc(iBytes);

  unsigned long long *s_timestamp;
  int *s_count;
  s_timestamp = (unsigned long long *)malloc(nBytes);
  s_count = (int *)malloc(iBytes);

  const string csv_file = std::string(argv[1]); 
  vector<vector<string>> data; 

  thrust::host_vector<unsigned long long> h_vec_timestamp(N);
  thrust::host_vector<int> h_vec_timestamp_count(N);

  try {
      	  Csv objCsv(csv_file);
	  if (!objCsv.getCsv(data)) {
	    cout << "read ERROR" << endl;
	    return 1;
	    }

	  for (int row = 0; row < data.size(); row++) {
	    vector<string> rec = data[row];

	    std::string timestamp = rec[0];

	    for(size_t c = timestamp.find_first_of("\""); c != string::npos; c = c = timestamp.find_first_of("\"")){
	      timestamp.erase(c,1);
	    }
	    for(size_t c = timestamp.find_first_of("."); c != string::npos; c = c = timestamp.find_first_of(".")){
	      timestamp.erase(c,1);
	    }
	    for(size_t c = timestamp.find_first_of(" "); c != string::npos; c = c = timestamp.find_first_of(" ")){
	      timestamp.erase(c,1);
	    }
	    for(size_t c = timestamp.find_first_of(":"); c != string::npos; c = c = timestamp.find_first_of(":")){
	      timestamp.erase(c,1);
	    }
	    for(size_t c = timestamp.find_first_of("/"); c != string::npos; c = c = timestamp.find_first_of("/")){
	      timestamp.erase(c,1);
	    }

	    h_timestamp[row] = stoull(timestamp.c_str());
	    h_count[row] = 1;

	    h_vec_timestamp[row] = stoull(timestamp.c_str());
	    h_vec_timestamp_count[row] = 1;
	  }

	  unsigned long long *d_timestamp, *timestamp_s;
	  int *d_count, *count_s;
	  
    	  hipMalloc((unsigned long long**)&d_timestamp, nBytes);
	  hipMalloc((unsigned long long**)&timestamp_s, nBytes);
    	  hipMalloc((int**)&d_count, iBytes);
	  hipMalloc((int**)&count_s, iBytes);
 
	  hipMemcpy(d_timestamp, h_timestamp, nBytes, hipMemcpyHostToDevice);
    	  hipMemcpy(d_count, h_count, iBytes, hipMemcpyHostToDevice);

	  dim3 block(1);
          dim3 grid(N);

          start_timer(&t);  
          sumArraysOnGPU<<<grid, block>>>(d_timestamp, d_count, N);
	  printf("Execution configure <<<%d, %d>>>\n", grid.x, block.x);
          travdirtime = stop_timer(&t);
          print_timer(travdirtime);

          hipMemcpy(h_timestamp, d_timestamp, nBytes, hipMemcpyDeviceToHost);
	  hipMemcpy(h_count, d_count, iBytes, hipMemcpyDeviceToHost);

	  /*
	  for(int i = 0; i < 5; i++)
	  	  cout << h_timestamp[i] << "," << h_count[i] << endl;	
          */

	  gpu_mergesort<<<grid, block>>>(d_timestamp, timestamp_s, N, N/10, N/10, d_count, count_s);
          hipMemcpy(s_timestamp, timestamp_s, nBytes, hipMemcpyDeviceToHost);
	  hipMemcpy(s_count, count_s, iBytes, hipMemcpyDeviceToHost);

	  for(int i = 0; i < 10; i++)
	  	  cout << s_timestamp[i] << "," << s_count[i] << endl;
		  
	  /*
	  bitonic_sort<<<grid, block>>>(d_timestamp, N, 1000, 100, d_count);	  
          hipMemcpy(h_timestamp, d_timestamp, nBytes, hipMemcpyDeviceToHost);
	  hipMemcpy(h_count, d_count, iBytes, hipMemcpyDeviceToHost);

	  for(int i = 0; i < 3; i++)
	  	  cout << h_timestamp[i] << "," << h_count[i] << endl;
          */

    	  hipFree(d_timestamp);
    	  hipFree(d_count);

          free(h_timestamp);
          free(h_count);
		
	  /*
          start_timer(&t);  

	  thrust::device_vector<unsigned long long> key_in = h_vec_timestamp;
	  thrust::device_vector<int> value_in = h_vec_timestamp_count;

	  thrust::sort(key_in.begin(), key_in.end());

	  thrust::device_vector<unsigned long long> dkey_out(N,0);
	  thrust::device_vector<int> dvalue_out(N,0);
	  
	  auto new_end = thrust::reduce_by_key(key_in.begin(),
					       key_in.end(),
					       value_in.begin(),
					       dkey_out.begin(),
	  				       dvalue_out.begin());

	  long new_size = new_end.first - dkey_out.begin();

          travdirtime = stop_timer(&t);
	  print_timer(travdirtime);
	  
	  for(int i = 0; i < 3; i++)
		cout << dkey_out[i] << "," << dvalue_out[i] << endl;
	  */

	  /*
	  std::remove("tmp2");
	  ofstream outputfile2("tmp2");
	  
	  for(long i=0; i < new_size; i++)
	    {
		outputfile2 << dkey_out[i] << "," << dvalue_out[i] << endl;
	    }

	  outputfile2.close();
	  */

	  // hipDeviceReset();
   }
	  
   catch (...) {
	  cout << "EXCEPTION!" << endl;
	  return 1;
   }
	
   // delete[] Data;       
   return 0;
}
