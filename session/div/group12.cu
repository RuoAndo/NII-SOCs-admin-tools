#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <pthread.h>

#include <string>
#include <iostream>
#include <fstream>
#include <sstream>
#include <bitset>

#include <random>
#include <map>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include "timer.h" 

#include <cstdio>
#include <cctype>
#include <iostream>
#include "tbb/task_scheduler_init.h"
#include "tbb/concurrent_hash_map.h"
#include "tbb/blocked_range.h"
#include "tbb/parallel_for.h"

using namespace tbb;
using namespace std;

#define THREAD_NUM 50

using namespace std;

struct HashCompare {
    static size_t hash( const char& x ) {
        return (size_t)x;
    }
    static bool equal( const char& x, const char& y ) {
        return x==y;
    }
};

typedef concurrent_hash_map<unsigned long, std::vector<unsigned long>> CharTable;
static CharTable table;

struct is_smaller_than_10 {
    __host__ __device__ bool operator()(const int a) const {
                return (a==0);
    }
};

template<class T>
struct saxpi{
    T k1;
    T k2;
    saxpi(T _k1, T _k2){
        k1=_k1;
        k2=_k2;
    }
    __host__ __device__ T operator()(T &x) const{
        return x*k1+k2;
    }
};

typedef struct _thread_arg {
    int id;
} thread_arg_t;

typedef struct _result {
  map<long, long> m;
  pthread_mutex_t mutex;
} result_t;
result_t result;

std::vector<std::string> split_string_2(std::string str, char del) {
  int first = 0;
  int last = str.find_first_of(del);

  std::vector<std::string> result;

  while (first < str.size()) {
    std::string subStr(str, first, last - first);

    result.push_back(subStr);

    first = last + 1;
    last = str.find_first_of(del, first);

    if (last == std::string::npos) {
      last = str.size();
    }
  }

  return result;
}

void *thread_func(void *arg) {
    thread_arg_t* targ = (thread_arg_t *)arg;
    int i, j, k;
    int label = 0;
    unsigned int key = 0;
    unsigned int value = 0;
    int counter = 0;
    string src;
    string dst;
    int progress = 0;

    string fname;

    if(std::to_string(targ->id).length()==1)
        fname = "x0" + std::to_string(targ->id);

    if(std::to_string(targ->id).length()==2)
        fname = "x" + std::to_string(targ->id);
	
    std::cout << "thread ID: " << targ->id << " - start." << std::endl;

    ifstream ifs(fname);
 
    string str;
    while(getline(ifs,str)){
      string token;
      istringstream stream(str);
      counter = 0;
      while(getline(stream,token,',')){
	
	if(counter==0)
	  src = token;

	if(counter==4)
	  dst = token;

        counter = counter + 1;
       }

	string tmp_string_first = src;
	string tmp_string_second = dst;

	for(size_t c = tmp_string_first.find_first_of("\""); c != string::npos; c = c = tmp_string_first.find_first_of("\"")){
	      tmp_string_first.erase(c,1);
	}

	for(size_t c = tmp_string_first.find_first_of(":"); c != string::npos; c = c = tmp_string_first.find_first_of(":")){
	      tmp_string_first.erase(c,1);
	}

	for(size_t c = tmp_string_first.find_first_of(" "); c != string::npos; c = c = tmp_string_first.find_first_of(" ")){
	      tmp_string_first.erase(c,1);
	}

	for(size_t c = tmp_string_first.find_first_of("/"); c != string::npos; c = c = tmp_string_first.find_first_of("/")){
	      tmp_string_first.erase(c,1);
	}

	for(size_t c = tmp_string_first.find_first_of("\""); c != string::npos; c = c = tmp_string_first.find_first_of("\"")){
	      tmp_string_first.erase(c,1);
	}

	for(size_t c = tmp_string_first.find_first_of("."); c != string::npos; c = c = tmp_string_first.find_first_of(".")){
	      tmp_string_first.erase(c,1);
	}

	for(size_t c = tmp_string_second.find_first_of("\""); c != string::npos; c = c = tmp_string_second.find_first_of("\"")){
	      tmp_string_second.erase(c,1);
	}

	char del = '.';

	std::string stringIP;
	std::string IPstring;
	    	    
        stringIP = tmp_string_second;
	for (const auto subStr : split_string_2(stringIP, del)) {
	      unsigned long ipaddr_src;
	      ipaddr_src = atoi(subStr.c_str());
	      std::bitset<8> trans =  std::bitset<8>(ipaddr_src);
	      std::string trans_string = trans.to_string();
	      IPstring = IPstring + trans_string;
	}  

	unsigned long n = bitset<32>(IPstring).to_ullong();
	unsigned long m = atol(tmp_string_first.c_str());
	
	// cout << "CHECK:" << m << "," << n << ":" << endl;

	CharTable::accessor a;
	table.insert(a, m);
	a->second.push_back(n);

	if(progress%100000==0)
	{
		cout <<  "progress count:" << targ->id << "," << progress << "," << table.size() << endl;
        }
	progress = progress + 1;
    }

    std::cout << "thread ID: " << targ->id << " - done." << std::endl;
    return;
}

__global__ void sumArraysOnGPU(unsigned long *A, unsigned long *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    C[i] = A[i] % B[i];
}

int main(int argc, char *argv[])
{
    pthread_t handle[THREAD_NUM];
    thread_arg_t targ[THREAD_NUM];

    string tmp_string_first;
    string tmp_string_second;

    unsigned int t, travdirtime;   
    int nData = atoi(argv[2]);

    int i;
    int counter;

    cout << "#1 concurrent hash map:" << endl;
    start_timer(&t);

    /* �����J�n */
    for (i = 0; i < THREAD_NUM; i++) {
        targ[i].id = i;
        pthread_create(&handle[i], NULL, thread_func, (void*)&targ[i]);
	// pthread_create(&handle[i], NULL, thread_func, (void*)&targ[i]);
    }

    /* �I����҂� */
    for (i = 0; i < THREAD_NUM; i++) 
        pthread_join(handle[i], NULL);

    travdirtime = stop_timer(&t);
    print_timer(travdirtime);       

    thrust::host_vector<unsigned long> h_vec_1(nData);
    thrust::host_vector<unsigned long> h_vec_2(nData);

    cout << "#2 concurrent hash map to host_vector:" << endl; 
    start_timer(&t);

    counter = 0;
    for( CharTable::iterator i=table.begin(); i!=table.end(); ++i )
    {
      for(auto itr = i->second.begin(); itr != i->second.end(); ++itr) {
	       h_vec_1[counter] = (unsigned long)i->first;
   	       h_vec_2[counter] = (unsigned long)*itr;
	      
      	       if(counter%1000000==0)
      	       {
	       std::cout << "counter:" << counter << endl;
	       std::cout << h_vec_1[counter] << "," << h_vec_2[counter] << endl;
	       std::cout << i->first << "," << *itr << endl;
      	       }
	       counter = counter + 1;
      }
    }

    cout << "counter:" << counter << endl;

    travdirtime = stop_timer(&t);
    print_timer(travdirtime);       

    for(i=0; i<counter; i++)
    {
	if(i%3000000==0)
	     cout << "CHECK:" << (unsigned long)h_vec_1[i] << "," << (unsigned long)h_vec_2[i] << endl;
    }

    /***/

    const string targetIP = std::string(argv[1]); 

    std::string stringIP;
    std::string IPstring;

    char del2 = '.';

    for (const auto subStr : split_string_2(targetIP, del2)) {
    	unsigned long ipaddr_src;
	ipaddr_src = atol(subStr.c_str());
	std::bitset<8> trans =  std::bitset<8>(ipaddr_src);
	std::string trans_string = trans.to_string();
	IPstring = IPstring + trans_string;
	}

    unsigned long s = bitset<32>(IPstring).to_ulong();
    unsigned long f = s;
    std::cout << targetIP << "," << f << endl;

    thrust::device_vector<unsigned long> d_vec_srcIP(nData);
    thrust::device_vector<unsigned long> d_vec_timestamp(nData);
    thrust::device_vector<int> d_vec_substract(nData);

    thrust::copy(h_vec_2.begin(), h_vec_2.end(), d_vec_srcIP.begin());
    thrust::copy(h_vec_1.begin(), h_vec_1.end(), d_vec_timestamp.begin());

    auto ff = [=]  __device__ (int x) {return x - f ;};
    thrust::transform(d_vec_srcIP.begin(),d_vec_srcIP.end(),d_vec_substract.begin(),ff);

    /*
    counter = 0;
    for(i=0; i<nData; i++)
    {
        if(d_vec_substract[i] == 0)
		cout << d_vec_timestamp[i] << "," << d_vec_srcIP[i] << "," << d_vec_substract[i] << endl;
		
	counter = counter + 1;
    }
    */

    cout << "#3 sort on GPU:" << endl;
    start_timer(&t);
    cout << "sort before:" << d_vec_timestamp.size() << endl;
    thrust::sort_by_key(d_vec_substract.begin(), d_vec_substract.end(), d_vec_timestamp.begin());
    cout << "sort after:" << d_vec_timestamp.size() << endl;
    travdirtime = stop_timer(&t);
    print_timer(travdirtime);       

    cout << "#4 copy_if on GPU:" << endl;
    start_timer(&t);
    std::cout << d_vec_timestamp.size() << std::endl;
    int N_prime = thrust::count_if(d_vec_substract.begin(), d_vec_substract.end(), is_smaller_than_10());
    std::cout << N_prime << std::endl;
    thrust::device_vector<int> d_b(N_prime);
    thrust::copy_if(d_vec_substract.begin(), d_vec_substract.end(), d_b.begin(), is_smaller_than_10());
    std::cout << "d_b size:" << d_b.size() << std::endl;
    travdirtime = stop_timer(&t);
    print_timer(travdirtime);       

    thrust::host_vector<int> h_b(N_prime);
    thrust::host_vector<unsigned long> h_vec_timestamp(N_prime);

    for(i=0; i<N_prime; i++)
    {
	h_vec_timestamp[i] = d_vec_timestamp[i];
	h_b[i] = d_b[i];
    }

    std::remove("tmp");
    ofstream outputfile("tmp");

    for(i=0; i<N_prime; i++)
    {
	cout << h_vec_timestamp[i] << "," << h_b[i] << endl;

	std::string tmpstring = std::to_string(h_vec_timestamp[i]);
	outputfile << tmpstring.substr( 0, 4 ) << "-" << tmpstring.substr( 4, 2 ) << "-" << tmpstring.substr( 6, 2 ) << " " << tmpstring.substr( 8, 2) << ":" << tmpstring.substr( 10, 2 ) << ":" << tmpstring.substr( 12, 2 ) << "," << argv[1] << endl;;                  
    }

    outputfile.close();

    /*
    for(i=0; i<d_b.size(); i++)
    {
	cout << d_vec_timestamp[i] << "," << d_b[i] << endl;
    }
    */

}

