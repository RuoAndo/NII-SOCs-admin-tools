#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <pthread.h>

#include <string>
#include <iostream>
#include <fstream>
#include <sstream>
#include <bitset>

// #include <eigen3/Eigen/Dense>
// #include <eigen3/Eigen/Core>
// #include <eigen3/Eigen/SVD>

#include <random>
#include <map>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
// #include "util.h"
#include "timer.h" 

#include <cstdio>
#include <cctype>
#include <iostream>
#include "tbb/task_scheduler_init.h"
#include "tbb/concurrent_hash_map.h"
#include "tbb/blocked_range.h"
#include "tbb/parallel_for.h"

using namespace tbb;
using namespace std;

#define THREAD_NUM 46

// using namespace Eigen;
using namespace std;

struct HashCompare {
    static size_t hash( const char& x ) {
        return (size_t)x;
    }
    static bool equal( const char& x, const char& y ) {
        return x==y;
    }
};

// typedef concurrent_hash_map<unsigned long, unsigned long, HashCompare> CharTable;
// typedef concurrent_hash_map<unsigned long, unsigned long> CharTable;
typedef concurrent_hash_map<unsigned long, std::vector<unsigned long>> CharTable;
// typedef concurrent_hash_map<MyString,std::vector<string>> StringTable; 
static CharTable table;

typedef struct _thread_arg {
    int id;
} thread_arg_t;

typedef struct _result {
  map<long, long> m;
  pthread_mutex_t mutex;
} result_t;
result_t result;

std::vector<std::string> split_string_2(std::string str, char del) {
  int first = 0;
  int last = str.find_first_of(del);

  std::vector<std::string> result;

  while (first < str.size()) {
    std::string subStr(str, first, last - first);

    result.push_back(subStr);

    first = last + 1;
    last = str.find_first_of(del, first);

    if (last == std::string::npos) {
      last = str.size();
    }
  }

  return result;
}

void *thread_func(void *arg) {
    thread_arg_t* targ = (thread_arg_t *)arg;
    int i, j, k;
    int label = 0;
    unsigned int key = 0;
    unsigned int value = 0;
    int counter = 0;
    string src;
    string dst;
    int progress = 0;

    string fname;

    if(std::to_string(targ->id).length()==1)
        fname = "x0" + std::to_string(targ->id);

    if(std::to_string(targ->id).length()==2)
        fname = "x" + std::to_string(targ->id);
	
    // string fname = std::to_string(targ->id);
    std::cout << "thread ID: " << targ->id << " - start." << std::endl;

    ifstream ifs(fname);
 
    string str;
    while(getline(ifs,str)){
      string token;
      istringstream stream(str);
      counter = 0;
      while(getline(stream,token,',')){
	
	if(counter==0)
	  src = token;

	if(counter==4)
	  dst = token;

        counter = counter + 1;
       }

	string tmp_string_first = src;
	string tmp_string_second = dst;

	for(size_t c = tmp_string_first.find_first_of("\""); c != string::npos; c = c = tmp_string_first.find_first_of("\"")){
	      tmp_string_first.erase(c,1);
	}

	for(size_t c = tmp_string_first.find_first_of(":"); c != string::npos; c = c = tmp_string_first.find_first_of(":")){
	      tmp_string_first.erase(c,1);
	}

	for(size_t c = tmp_string_first.find_first_of(" "); c != string::npos; c = c = tmp_string_first.find_first_of(" ")){
	      tmp_string_first.erase(c,1);
	}

	for(size_t c = tmp_string_first.find_first_of("/"); c != string::npos; c = c = tmp_string_first.find_first_of("/")){
	      tmp_string_first.erase(c,1);
	}

	for(size_t c = tmp_string_first.find_first_of("\""); c != string::npos; c = c = tmp_string_first.find_first_of("\"")){
	      tmp_string_first.erase(c,1);
	}

	for(size_t c = tmp_string_first.find_first_of("."); c != string::npos; c = c = tmp_string_first.find_first_of(".")){
	      tmp_string_first.erase(c,1);
	}

	for(size_t c = tmp_string_second.find_first_of("\""); c != string::npos; c = c = tmp_string_second.find_first_of("\"")){
	      tmp_string_second.erase(c,1);
	}

	char del = '.';

	std::string stringIP;
	std::string IPstring;
	    	    
        stringIP = tmp_string_second;
	for (const auto subStr : split_string_2(stringIP, del)) {
	      unsigned long ipaddr_src;
	      ipaddr_src = atoi(subStr.c_str());
	      std::bitset<8> trans =  std::bitset<8>(ipaddr_src);
	      std::string trans_string = trans.to_string();
	      IPstring = IPstring + trans_string;
	}  

	unsigned long n = bitset<64>(IPstring).to_ullong();
	long m = atol(tmp_string_first.c_str());
	
	// cout << src << "," << n << "," << dst << "," << m << endl;

	CharTable::accessor a;
	table.insert(a, m);
	// table.insert(a, n);
	a->second.push_back(n);
	// a->second.push_back( n );

	/*
	pthread_mutex_lock(&result.mutex);
	result.m.insert(pair<long, long>(m,n));
	pthread_mutex_unlock(&result.mutex);
	*/
    
	if(progress%100000==0)
	{
		cout <<  "progress count:" << targ->id << "," << progress << "," << table.size() << endl;
		// cout <<  "progress count:" << targ->id << "," << progress << "," << endl;
        }
	progress = progress + 1;
	/*
	for( CharTable::iterator i=table.begin(); i!=table.end(); ++i )
             cout << i->first << "," << i->second;
        cout << endl;
	*/
    }

    /*    
    for( CharTable::iterator i=table.begin(); i!=table.end(); ++i )
        cout << i->first << "(" << i->second << ") ";
    cout << endl;
    */

    std::cout << "thread ID: " << targ->id << " - done." << std::endl;
    return;
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    // int i = blockIdx.x * blockDim.x + threadIdx.x;
     int i = blockIdx.x; // * blockDim.x + threadIdx.x;
    // C[i] = A[i] + B[i];
    C[i] = A[i] / B[i];
    // B[i] = B[i] + 1000;
    // C[i] = 3;
}

int main(int argc, char *argv[])
{
    pthread_t handle[THREAD_NUM];
    thread_arg_t targ[THREAD_NUM];

    string tmp_string_first;
    string tmp_string_second;

    unsigned int t, travdirtime;   

    int i;
    int counter;

    cout << "threads:" << endl;
    start_timer(&t);

    /* �����J�n */
    for (i = 0; i < THREAD_NUM; i++) {
        targ[i].id = i;
        pthread_create(&handle[i], NULL, thread_func, (void*)&targ[i]);
	// pthread_create(&handle[i], NULL, thread_func, (void*)&targ[i]);
    }

    /* �I����҂� */
    for (i = 0; i < THREAD_NUM; i++) 
        pthread_join(handle[i], NULL);

    travdirtime = stop_timer(&t);
    print_timer(travdirtime);       

    // std::remove("tmp");
    // ofstream outputfile("tmp");

    thrust::host_vector<long> h_vec_1(table.size());
    thrust::host_vector<long> h_vec_2(table.size());

    cout << "host_vector" << endl; 
    start_timer(&t);

    counter = 0;
    for( CharTable::iterator i=table.begin(); i!=table.end(); ++i )
    {

      for(auto itr = i->second.begin(); itr != i->second.end(); ++itr) {
      	       // outputfile << i->first << "," << *itr << endl;
	       h_vec_1[counter] = long(i->first);
   	       h_vec_2[counter] = long(*itr);
      }
      
      if(counter%1000000==0)
      {
	std::cout << "counter:" << counter << endl;
      }
      
      counter = counter + 1;
    }

    travdirtime = stop_timer(&t);
    print_timer(travdirtime);       

    for(i=0; i<10; i++)
    {
	cout << h_vec_1[i] << "," << h_vec_2[i] << endl;
    }

    cout << "sort by key" << endl;

    start_timer(&t);

    thrust::device_vector<long> d_vec_1 = h_vec_1;
    thrust::device_vector<long> d_vec_2 = h_vec_2;
    thrust::sort_by_key(d_vec_1.begin(), d_vec_1.end(), d_vec_2.begin());

    travdirtime = stop_timer(&t);
    print_timer(travdirtime);       

    for(i=0; i<10; i++)
    {
	cout << d_vec_1[i] << "," << d_vec_2[i] << endl;
    }      

    size_t nBytes = d_vec_1.size() * sizeof(float);

    cout << "ip to float / device -> host" << endl;

    start_timer(&t);
    float *h_A, *h_B, *h_C, *h_D, *hostRef, *gpuRef;
    h_A     = (float *)malloc(nBytes);
    h_B     = (float *)malloc(nBytes);
    h_C     = (float *)malloc(nBytes);
    h_D     = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef  = (float *)malloc(nBytes);

    const string targetIP = std::string(argv[1]); 

    std::string stringIP;
    std::string IPstring;

    char del2 = '.';

    for (const auto subStr : split_string_2(targetIP, del2)) {
    	unsigned long ipaddr_src;
	ipaddr_src = atol(subStr.c_str());
	std::bitset<8> trans =  std::bitset<8>(ipaddr_src);
	std::string trans_string = trans.to_string();
	IPstring = IPstring + trans_string;
	}

     unsigned long s = bitset<32>(IPstring).to_ulong();
     unsigned int f = (unsigned int)s;
     std::cout << targetIP << "," << f << endl;

     for(i=0;i<d_vec_1.size();i++)
     {	
    	h_A[i] = d_vec_1[i];
	h_B[i] = d_vec_2[i];
	h_C[i] = f;
    }
    travdirtime = stop_timer(&t);
    print_timer(travdirtime);       

    float *d_A, *d_B, *d_C, *d_D;
    hipMalloc((float**)&d_A, nBytes);
    hipMalloc((float**)&d_B, nBytes);
    hipMalloc((float**)&d_C, nBytes);
    hipMalloc((float**)&d_D, nBytes);

    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, nBytes, hipMemcpyHostToDevice);
    // hipMemcpy(d_D, h_D, nBytes, hipMemcpyHostToDevice);

    cout << "division" << endl;
    
    start_timer(&t);  

    dim3 block=32;
    dim3 grid=32;

    sumArraysOnGPU<<<grid, block>>>(d_B, d_C, d_D, d_vec_1.size());
    printf("Execution configure <<<%d, %d>>>\n", grid.x, block.x);

    travdirtime = stop_timer(&t);
    print_timer(travdirtime);

    hipMemcpy(h_A, d_A, nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_B, d_B, nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_C, d_C, nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_D, d_D, nBytes, hipMemcpyDeviceToHost);

    for(i=0;i<10;i++)
    {
	cout << h_A[i] << "," << h_B[i] << "," << h_C[i] << "," << h_D[i] << endl;
	}

    // outputfile.close();

   // std::cout << "map size() is " << result.m.size() << std::endl;
}
