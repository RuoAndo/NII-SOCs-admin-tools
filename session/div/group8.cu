#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <pthread.h>

#include <string>
#include <iostream>
#include <fstream>
#include <sstream>
#include <bitset>

// #include <eigen3/Eigen/Dense>
// #include <eigen3/Eigen/Core>
// #include <eigen3/Eigen/SVD>

#include <random>
#include <map>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
// #include "util.h"
#include "timer.h" 

#include <cstdio>
#include <cctype>
#include <iostream>
#include "tbb/task_scheduler_init.h"
#include "tbb/concurrent_hash_map.h"
#include "tbb/blocked_range.h"
#include "tbb/parallel_for.h"

using namespace tbb;
using namespace std;

#define THREAD_NUM 10

// using namespace Eigen;
using namespace std;

struct HashCompare {
    static size_t hash( const char& x ) {
        return (size_t)x;
    }
    static bool equal( const char& x, const char& y ) {
        return x==y;
    }
};

// typedef concurrent_hash_map<unsigned long, unsigned long, HashCompare> CharTable;
// typedef concurrent_hash_map<unsigned long, unsigned long> CharTable;
// typedef concurrent_hash_map<unsigned long, std::vector<unsigned long>> CharTable;
typedef concurrent_hash_map<unsigned long, std::vector<unsigned long>> CharTable;
// typedef concurrent_hash_map<MyString,std::vector<string>> StringTable; 
static CharTable table;

typedef struct _thread_arg {
    int id;
} thread_arg_t;

typedef struct _result {
  map<long, long> m;
  pthread_mutex_t mutex;
} result_t;
result_t result;

std::vector<std::string> split_string_2(std::string str, char del) {
  int first = 0;
  int last = str.find_first_of(del);

  std::vector<std::string> result;

  while (first < str.size()) {
    std::string subStr(str, first, last - first);

    result.push_back(subStr);

    first = last + 1;
    last = str.find_first_of(del, first);

    if (last == std::string::npos) {
      last = str.size();
    }
  }

  return result;
}

void *thread_func(void *arg) {
    thread_arg_t* targ = (thread_arg_t *)arg;
    int i, j, k;
    int label = 0;
    unsigned int key = 0;
    unsigned int value = 0;
    int counter = 0;
    string src;
    string dst;
    int progress = 0;

    string fname;

    if(std::to_string(targ->id).length()==1)
        fname = "x0" + std::to_string(targ->id);

    if(std::to_string(targ->id).length()==2)
        fname = "x" + std::to_string(targ->id);
	
    // string fname = std::to_string(targ->id);
    std::cout << "thread ID: " << targ->id << " - start." << std::endl;

    ifstream ifs(fname);
 
    string str;
    while(getline(ifs,str)){
      string token;
      istringstream stream(str);
      counter = 0;
      while(getline(stream,token,',')){
	
	if(counter==0)
	  src = token;

	if(counter==4)
	  dst = token;

        counter = counter + 1;
       }

	string tmp_string_first = src;
	string tmp_string_second = dst;

	for(size_t c = tmp_string_first.find_first_of("\""); c != string::npos; c = c = tmp_string_first.find_first_of("\"")){
	      tmp_string_first.erase(c,1);
	}

	for(size_t c = tmp_string_first.find_first_of(":"); c != string::npos; c = c = tmp_string_first.find_first_of(":")){
	      tmp_string_first.erase(c,1);
	}

	for(size_t c = tmp_string_first.find_first_of(" "); c != string::npos; c = c = tmp_string_first.find_first_of(" ")){
	      tmp_string_first.erase(c,1);
	}

	for(size_t c = tmp_string_first.find_first_of("/"); c != string::npos; c = c = tmp_string_first.find_first_of("/")){
	      tmp_string_first.erase(c,1);
	}

	for(size_t c = tmp_string_first.find_first_of("\""); c != string::npos; c = c = tmp_string_first.find_first_of("\"")){
	      tmp_string_first.erase(c,1);
	}

	for(size_t c = tmp_string_first.find_first_of("."); c != string::npos; c = c = tmp_string_first.find_first_of(".")){
	      tmp_string_first.erase(c,1);
	}

	for(size_t c = tmp_string_second.find_first_of("\""); c != string::npos; c = c = tmp_string_second.find_first_of("\"")){
	      tmp_string_second.erase(c,1);
	}

	char del = '.';

	std::string stringIP;
	std::string IPstring;
	    	    
        stringIP = tmp_string_second;
	for (const auto subStr : split_string_2(stringIP, del)) {
	      unsigned long ipaddr_src;
	      ipaddr_src = atoi(subStr.c_str());
	      std::bitset<8> trans =  std::bitset<8>(ipaddr_src);
	      std::string trans_string = trans.to_string();
	      IPstring = IPstring + trans_string;
	}  

	unsigned long n = bitset<64>(IPstring).to_ullong();
	long m = atol(tmp_string_first.c_str());
	
	// cout << src << "," << n << "," << dst << "," << m << endl;

	CharTable::accessor a;
	table.insert(a, m);
	// table.insert(a, n);
	a->second.push_back(n);
	// a->second.push_back( n );

	/*
	pthread_mutex_lock(&result.mutex);
	result.m.insert(pair<long, long>(m,n));
	pthread_mutex_unlock(&result.mutex);
	*/
    
	if(progress%100000==0)
	{
		cout <<  "progress count:" << targ->id << "," << progress << "," << table.size() << endl;
		// cout <<  "progress count:" << targ->id << "," << progress << "," << endl;
        }
	progress = progress + 1;

	/*
	for( CharTable::iterator i=table.begin(); i!=table.end(); ++i )
             cout << i->first << "," << i->second;
        cout << endl;
	*/
    }

    /*    
    for( CharTable::iterator i=table.begin(); i!=table.end(); ++i )
        cout << i->first << "(" << i->second << ") ";
    cout << endl;
    */

    std::cout << "thread ID: " << targ->id << " - done." << std::endl;
    return;
}

__global__ void sumArraysOnGPU(unsigned long *A, unsigned long *B, int *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    //  int i = blockIdx.x; // * blockDim.x + threadIdx.x;
    // C[i] = A[i] + B[i];
    C[i] = A[i] - B[i];
    // B[i] = B[i] + 1000;
    // C[i] = 3;
}

int main(int argc, char *argv[])
{
    pthread_t handle[THREAD_NUM];
    thread_arg_t targ[THREAD_NUM];

    string tmp_string_first;
    string tmp_string_second;

    unsigned int t, travdirtime;   
    int nData = atoi(argv[2]);

    int i;
    int counter;

    cout << "concurrent hash map:" << endl;
    start_timer(&t);

    /* �����J�n */
    for (i = 0; i < THREAD_NUM; i++) {
        targ[i].id = i;
        pthread_create(&handle[i], NULL, thread_func, (void*)&targ[i]);
	// pthread_create(&handle[i], NULL, thread_func, (void*)&targ[i]);
    }

    /* �I����҂� */
    for (i = 0; i < THREAD_NUM; i++) 
        pthread_join(handle[i], NULL);

    travdirtime = stop_timer(&t);
    print_timer(travdirtime);       

    // std::remove("tmp");
    // ofstream outputfile("tmp");

    thrust::host_vector<unsigned long> h_vec_1(nData);
    thrust::host_vector<unsigned long> h_vec_2(nData);

    cout << "concurrent hash map to host_vector:" << endl; 
    start_timer(&t);

    // ofstream outputfile("tmp"); 
    counter = 0;
    for( CharTable::iterator i=table.begin(); i!=table.end(); ++i )
    {
      for(auto itr = i->second.begin(); itr != i->second.end(); ++itr) {
      	       // outputfile << i->first << "," << *itr << endl;
	       h_vec_1[counter] = (unsigned long)(i->first);
   	       h_vec_2[counter] = (unsigned long)(*itr);
	       counter = counter + 1;
      }
      
      if(counter%1000000==0)
      {
	std::cout << "counter:" << counter << endl;
      }
    }
    // outputfile.close();

    cout << "counter:" << counter << endl;

    travdirtime = stop_timer(&t);
    print_timer(travdirtime);       

    for(i=0; i<10; i++)
    {
	cout << h_vec_1[i] << "," << h_vec_2[i] << endl;
    }

    cout << "host->GPU - sort by key:" << endl;

    start_timer(&t);

    // thrust::device_vector<unsigned long> d_vec_1 = h_vec_1;
    // thrust::device_vector<unsigned long> d_vec_2 = h_vec_2;

    thrust::device_vector<unsigned long> d_vec_1(h_vec_1.size()); // = h_vec_1;
    thrust::device_vector<unsigned long> d_vec_2(h_vec_2.size()); //  = h_vec_2;

    thrust::copy(h_vec_1.begin(), h_vec_1.end(), d_vec_1.begin());
    thrust::copy(h_vec_2.begin(), h_vec_2.end(), d_vec_2.begin());

    cout << "sort before:" << d_vec_2.size() << endl;
    thrust::sort_by_key(d_vec_1.begin(), d_vec_1.end(), d_vec_2.begin());
    cout << "sort after:" << d_vec_2.size() << endl;

    travdirtime = stop_timer(&t);
    print_timer(travdirtime);       

    for(i=0; i<10; i++)
    {
	cout << d_vec_1[i] << "," << d_vec_2[i] << endl;
    }      

    size_t nBytes = nData * sizeof(unsigned long);
    size_t iBytes = nData * sizeof(int);

    cout << "GPU to host - ip to long - host to GPU" << endl;

    start_timer(&t);
    // long *h_A, *h_B, *h_C, *h_D, *hostRef, *gpuRef;
    unsigned long *h_A, *h_B, *h_C, *hostRef, *gpuRef;
    int *h_D;
    h_A     = (unsigned long *)malloc(nBytes);
    h_B     = (unsigned long *)malloc(nBytes);
    h_C     = (unsigned long *)malloc(nBytes);
    h_D     = (int *)malloc(iBytes);
    hostRef = (unsigned long *)malloc(nBytes);
    gpuRef  = (unsigned long *)malloc(nBytes);

    cout << "allocated" << endl;

    const string targetIP = std::string(argv[1]); 

    std::string stringIP;
    std::string IPstring;

    char del2 = '.';

    for (const auto subStr : split_string_2(targetIP, del2)) {
    	unsigned long ipaddr_src;
	ipaddr_src = atol(subStr.c_str());
	std::bitset<8> trans =  std::bitset<8>(ipaddr_src);
	std::string trans_string = trans.to_string();
	IPstring = IPstring + trans_string;
	}

     unsigned long s = bitset<32>(IPstring).to_ulong();
     unsigned long f = s;
     std::cout << targetIP << "," << f << endl;

     counter = 0;
     for(i=0;i<nData;i++)
     {	
    	h_A[i] = d_vec_1[i];
	h_B[i] = d_vec_2[i];
	h_C[i] = f;
	
	if(counter%1000000==0)
	  cout << "transfering " << counter << " vectors" << endl;
	counter = counter + 1;
    }

    cout << "done." << endl;

     for(i=0;i<10;i++)
     {	
    	cout << h_A[i] << "," << h_B[i] << "," << h_C[i] << endl;
    }

    travdirtime = stop_timer(&t);
    print_timer(travdirtime);       

    cout << "transfer: host to GPU" << endl;

    start_timer(&t);  

    // long *d_A, *d_B, *d_C, *d_D;
    unsigned long *d_A, *d_B, *d_C;
    int *d_D;
    hipMalloc((unsigned long**)&d_A, nBytes);
    hipMalloc((unsigned long**)&d_B, nBytes);
    hipMalloc((unsigned long**)&d_C, nBytes);
    hipMalloc((int**)&d_D, iBytes);

    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_D, h_D, iBytes, hipMemcpyHostToDevice);

    travdirtime = stop_timer(&t);
    print_timer(travdirtime);       

    cout << "substraction:" << endl;
    
    start_timer(&t);  

    dim3 block=32;
    dim3 grid=32;

    sumArraysOnGPU<<<grid, block>>>(d_B, d_C, d_D, nData);
    printf("Execution configure <<<%d, %d>>>\n", grid.x, block.x);

    travdirtime = stop_timer(&t);
    print_timer(travdirtime);       

    hipMemcpy(h_A, d_A, nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_B, d_B, nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_C, d_C, nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_D, d_D, iBytes, hipMemcpyDeviceToHost);

    travdirtime = stop_timer(&t);
    print_timer(travdirtime);

    for(i=0;i<5;i++)
    {
	cout << h_A[i] << "," << h_B[i] << "," << h_C[i] << "," << h_D[i] << endl;
    }

    cout << "compare : 1" << endl;
    start_timer(&t);

    std::remove("result");
    ofstream outputfile("result");
    for(i=0;i<nData;i++)
    {
       if(h_C[i] == h_B[i])
       {
       	cout << h_A[i] << "," << h_B[i] << "," << h_C[i] << "," << h_D[i] << endl;
	outputfile << h_A[i] << "," << h_B[i] << "," << h_C[i] << "," << h_D[i] << endl;
       }
    }
    outputfile.close();

    travdirtime = stop_timer(&t);
    print_timer(travdirtime);

    // outputfile.close();

   // std::cout << "map size() is " << result.m.size() << std::endl;
}
