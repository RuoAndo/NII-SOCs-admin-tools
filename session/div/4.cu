#include "hip/hip_runtime.h"
#if __linux__ && defined(__INTEL_COMPILER)
#define __sync_fetch_and_add(ptr,addend) _InterlockedExchangeAdd(const_cast<void*>(reinterpret_cast<volatile void*>(ptr)), addend)
#endif
#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>
#include <map>

#include <sys/socket.h>
#include <netinet/in.h>
#include <arpa/inet.h>

#include "csv.hpp"
#include "timer.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include "util.h"

using namespace std;
// static MyString* Data;

__global__ void sumArraysOnGPU(unsigned long long *A, int *B, int *C, const int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // if (i < N) C[i] = A[i] / B[i];

    for(int i=0; i < N; i++)
    {
	if(A[idx] == A[i])
	{
		B[idx]++;
		C[i]=1;
	}
    }
}

__global__ void sumArraysOnGPU_2(unsigned long long *A, int *B, int *C, const int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i=0; i < N; i++)
    {
	if(A[idx] == A[i] && B[idx] == B[i])
	{
		C[idx] = A[idx];
	}
    }
}

int main( int argc, char* argv[] ) {

  int N = atoi(argv[2]);  
  unsigned int t, travdirtime; 
  int counter;
  
  size_t nBytes = N * sizeof(unsigned long long);
  size_t iBytes = N * sizeof(int);
  
  unsigned long long *h_timestamp;
  int *h_count;
  int *h_flag;

  h_timestamp = (unsigned long long *)malloc(nBytes);
  h_count = (int *)malloc(iBytes);
  h_flag = (int *)malloc(iBytes);

  const string csv_file = std::string(argv[1]); 
  vector<vector<string>> data; 

  thrust::host_vector<unsigned long long> h_vec_timestamp(N);
  thrust::host_vector<int> h_vec_timestamp_count(N);

  try {
      	  Csv objCsv(csv_file);
	  if (!objCsv.getCsv(data)) {
	    cout << "read ERROR" << endl;
	    return 1;
	    }

	  for (int row = 0; row < data.size(); row++) {
	    vector<string> rec = data[row];

	    std::string timestamp = rec[0];

	    for(size_t c = timestamp.find_first_of("\""); c != string::npos; c = c = timestamp.find_first_of("\"")){
	      timestamp.erase(c,1);
	    }
	    for(size_t c = timestamp.find_first_of("."); c != string::npos; c = c = timestamp.find_first_of(".")){
	      timestamp.erase(c,1);
	    }
	    for(size_t c = timestamp.find_first_of(" "); c != string::npos; c = c = timestamp.find_first_of(" ")){
	      timestamp.erase(c,1);
	    }
	    for(size_t c = timestamp.find_first_of(":"); c != string::npos; c = c = timestamp.find_first_of(":")){
	      timestamp.erase(c,1);
	    }
	    for(size_t c = timestamp.find_first_of("/"); c != string::npos; c = c = timestamp.find_first_of("/")){
	      timestamp.erase(c,1);
	    }

	    h_timestamp[row] = stoull(timestamp.c_str());
	    h_count[row] = 1;
	    h_flag[row] = 0;

	    h_vec_timestamp[row] = stoull(timestamp.c_str());
	    h_vec_timestamp_count[row] = 1;
	  }

	  unsigned long long *d_timestamp;
	  int *d_count;
	  int *d_flag;
    	  hipMalloc((unsigned long long**)&d_timestamp, nBytes);
    	  hipMalloc((int**)&d_count, iBytes);
	  hipMalloc((int**)&d_flag, iBytes);
 
	  hipMemcpy(d_timestamp, h_timestamp, nBytes, hipMemcpyHostToDevice);
    	  hipMemcpy(d_count, h_count, iBytes, hipMemcpyHostToDevice);
	  hipMemcpy(d_flag, h_flag, iBytes, hipMemcpyHostToDevice);

	  dim3 block (1);
          dim3 grid  (N);
	  
          start_timer(&t);  
          sumArraysOnGPU<<<grid, block>>>(d_timestamp, d_count, d_flag, N);
	  printf("Execution configure <<<%d, %d>>>\n", grid.x, block.x);
          travdirtime = stop_timer(&t);
          print_timer(travdirtime);


          hipMemcpy(h_count, d_count, iBytes, hipMemcpyDeviceToHost);
          hipMemcpy(h_timestamp, d_timestamp, nBytes, hipMemcpyDeviceToHost);
	  
	  thrust::device_vector<unsigned long long> dvec_tmp_1(N);
	  thrust::device_vector<int> dvec_tmp_2(N);

	  for(int i=0; i < N; i++)
	  {
		dvec_tmp_2[i] = h_count[i];
		dvec_tmp_1[i] = h_timestamp[i];
	  }

	  thrust::sort_by_key(dvec_tmp_1.begin(), dvec_tmp_1.end(), dvec_tmp_2.begin());
	  auto new_end_1 = thrust::unique_by_key(dvec_tmp_1.begin(), dvec_tmp_1.end(), dvec_tmp_2.begin());
	  long new_size_1 = new_end_1.first - dvec_tmp_1.begin();

	  for(int i=0 ; i< 4; i++)
	  	  std::cout << dvec_tmp_1[i] <<"," << dvec_tmp_2[i] << std::endl;

    	  hipFree(d_timestamp);
    	  hipFree(d_count);
	  hipFree(d_flag);

	  std::remove("tmp");
	  ofstream outputfile("tmp");

	  counter = 0;
	  for(int i=0;i<N;i++)
	  {
		// outputfile << h_timestamp[i] << "," << h_count[i] << std::endl;
		if(h_count[i]!=0)
	        {
			outputfile << h_timestamp[i] << "," << h_count[i] << std::endl;
		        // counter++;
		}
	  }
	  outputfile.close();

          free(h_timestamp);
          free(h_count);
	  free(h_flag);

          start_timer(&t);  

	  thrust::device_vector<unsigned long long> key_in = h_vec_timestamp;
	  thrust::device_vector<int> value_in = h_vec_timestamp_count;

	  thrust::sort(key_in.begin(), key_in.end());

	  thrust::device_vector<unsigned long long> dkey_out(N,0);
	  thrust::device_vector<int> dvalue_out(N,0);
	  
	  auto new_end = thrust::reduce_by_key(key_in.begin(),
					       key_in.end(),
					       value_in.begin(),
					       dkey_out.begin(),
	  				       dvalue_out.begin());

	  long new_size = new_end.first - dkey_out.begin();

          travdirtime = stop_timer(&t);
	  print_timer(travdirtime);
	  
	  for(int i = 0; i < 3; i++)
		cout << dkey_out[i] << "," << dvalue_out[i] << endl;

	  std::remove("tmp2");
	  ofstream outputfile2("tmp2");
	  
	  for(long i=0; i < new_size; i++)
	    {
	        /*
		if(std::to_string(dkey_out[i]).length()  == 17)
		{
			std::cout << dkey_out[i] << "," << dvalue_out[i] << endl;
	  	}
		*/
		outputfile2 << dkey_out[i] << "," << dvalue_out[i] << endl;
	    }

	  outputfile2.close();

	  // hipDeviceReset();
   }
	  
   catch (...) {
	  cout << "EXCEPTION!" << endl;
	  return 1;
   }
	
   // delete[] Data;       
   return 0;
}
