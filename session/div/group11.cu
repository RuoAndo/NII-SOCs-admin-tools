#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <pthread.h>

#include <string>
#include <iostream>
#include <fstream>
#include <sstream>
#include <bitset>

#include <random>
#include <map>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include "timer.h" 

#include <cstdio>
#include <cctype>
#include <iostream>
#include "tbb/task_scheduler_init.h"
#include "tbb/concurrent_hash_map.h"
#include "tbb/blocked_range.h"
#include "tbb/parallel_for.h"

using namespace tbb;
using namespace std;

#define THREAD_NUM 1

using namespace std;

struct HashCompare {
    static size_t hash( const char& x ) {
        return (size_t)x;
    }
    static bool equal( const char& x, const char& y ) {
        return x==y;
    }
};

typedef concurrent_hash_map<unsigned long, std::vector<unsigned long>> CharTable;
static CharTable table;

struct is_smaller_than_10 {
    __host__ __device__ bool operator()(const int a) const {
                return (a<1);
    }
};

typedef struct _thread_arg {
    int id;
} thread_arg_t;

typedef struct _result {
  map<long, long> m;
  pthread_mutex_t mutex;
} result_t;
result_t result;

std::vector<std::string> split_string_2(std::string str, char del) {
  int first = 0;
  int last = str.find_first_of(del);

  std::vector<std::string> result;

  while (first < str.size()) {
    std::string subStr(str, first, last - first);

    result.push_back(subStr);

    first = last + 1;
    last = str.find_first_of(del, first);

    if (last == std::string::npos) {
      last = str.size();
    }
  }

  return result;
}

void *thread_func(void *arg) {
    thread_arg_t* targ = (thread_arg_t *)arg;
    int i, j, k;
    int label = 0;
    unsigned int key = 0;
    unsigned int value = 0;
    int counter = 0;
    string src;
    string dst;
    int progress = 0;

    string fname;

    if(std::to_string(targ->id).length()==1)
        fname = "x0" + std::to_string(targ->id);

    if(std::to_string(targ->id).length()==2)
        fname = "x" + std::to_string(targ->id);
	
    std::cout << "thread ID: " << targ->id << " - start." << std::endl;

    ifstream ifs(fname);
 
    string str;
    while(getline(ifs,str)){
      string token;
      istringstream stream(str);
      counter = 0;
      while(getline(stream,token,',')){
	
	if(counter==0)
	  src = token;

	if(counter==4)
	  dst = token;

        counter = counter + 1;
       }

	string tmp_string_first = src;
	string tmp_string_second = dst;

	for(size_t c = tmp_string_first.find_first_of("\""); c != string::npos; c = c = tmp_string_first.find_first_of("\"")){
	      tmp_string_first.erase(c,1);
	}

	for(size_t c = tmp_string_first.find_first_of(":"); c != string::npos; c = c = tmp_string_first.find_first_of(":")){
	      tmp_string_first.erase(c,1);
	}

	for(size_t c = tmp_string_first.find_first_of(" "); c != string::npos; c = c = tmp_string_first.find_first_of(" ")){
	      tmp_string_first.erase(c,1);
	}

	for(size_t c = tmp_string_first.find_first_of("/"); c != string::npos; c = c = tmp_string_first.find_first_of("/")){
	      tmp_string_first.erase(c,1);
	}

	for(size_t c = tmp_string_first.find_first_of("\""); c != string::npos; c = c = tmp_string_first.find_first_of("\"")){
	      tmp_string_first.erase(c,1);
	}

	for(size_t c = tmp_string_first.find_first_of("."); c != string::npos; c = c = tmp_string_first.find_first_of(".")){
	      tmp_string_first.erase(c,1);
	}

	for(size_t c = tmp_string_second.find_first_of("\""); c != string::npos; c = c = tmp_string_second.find_first_of("\"")){
	      tmp_string_second.erase(c,1);
	}

	char del = '.';

	std::string stringIP;
	std::string IPstring;
	    	    
        stringIP = tmp_string_second;
	for (const auto subStr : split_string_2(stringIP, del)) {
	      unsigned long ipaddr_src;
	      ipaddr_src = atoi(subStr.c_str());
	      std::bitset<8> trans =  std::bitset<8>(ipaddr_src);
	      std::string trans_string = trans.to_string();
	      IPstring = IPstring + trans_string;
	}  

	unsigned long n = bitset<64>(IPstring).to_ullong();
	long m = atol(tmp_string_first.c_str());
	
	// cout << src << "," << n << "," << dst << "," << m << endl;

	CharTable::accessor a;
	table.insert(a, m);
	a->second.push_back(n);

	/*
	if(progress%100000==0)
	{
		cout <<  "progress count:" << targ->id << "," << progress << "," << table.size() << endl;
        }
	progress = progress + 1;
�@�@�@�@*/
    }

    std::cout << "thread ID: " << targ->id << " - done." << std::endl;
    return;
}

__global__ void sumArraysOnGPU(unsigned long *A, unsigned long *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    C[i] = A[i] % B[i];
}

int main(int argc, char *argv[])
{
    pthread_t handle[THREAD_NUM];
    thread_arg_t targ[THREAD_NUM];

    string tmp_string_first;
    string tmp_string_second;

    unsigned int t, travdirtime;   
    int nData = atoi(argv[2]);
    int nBLK = atoi(argv[3]);
    int nGRD = atoi(argv[4]);

    int i;
    int counter;

    cout << "#1 concurrent hash map:" << endl;
    start_timer(&t);

    /* �����J�n */
    for (i = 0; i < THREAD_NUM; i++) {
        targ[i].id = i;
        pthread_create(&handle[i], NULL, thread_func, (void*)&targ[i]);
	// pthread_create(&handle[i], NULL, thread_func, (void*)&targ[i]);
    }

    /* �I����҂� */
    for (i = 0; i < THREAD_NUM; i++) 
        pthread_join(handle[i], NULL);

    travdirtime = stop_timer(&t);
    print_timer(travdirtime);       

    thrust::host_vector<unsigned long> h_vec_1(nData);
    thrust::host_vector<unsigned long> h_vec_2(nData);

    cout << "#2 concurrent hash map to host_vector:" << endl; 
    start_timer(&t);

    counter = 0;
    for( CharTable::iterator i=table.begin(); i!=table.end(); ++i )
    {
      for(auto itr = i->second.begin(); itr != i->second.end(); ++itr) {
	       h_vec_1[counter] = (unsigned long)(i->first);
   	       h_vec_2[counter] = (unsigned long)(*itr);
	       counter = counter + 1;
      }

      /*
      if(counter%1000000==0)
      {
	std::cout << "counter:" << counter << endl;
      }
      */
    }

    cout << "counter:" << counter << endl;

    travdirtime = stop_timer(&t);
    print_timer(travdirtime);       

    for(i=0; i<10; i++)
    {
	cout << h_vec_1[i] << "," << h_vec_2[i] << endl;
    }

    cout << "#3 host->GPU - sort by key:" << endl;

    start_timer(&t);

    thrust::device_vector<unsigned long> d_vec_1(h_vec_1.size()); // = h_vec_1;
    thrust::device_vector<unsigned long> d_vec_2(h_vec_2.size()); //  = h_vec_2;

    thrust::copy(h_vec_1.begin(), h_vec_1.end(), d_vec_1.begin());
    thrust::copy(h_vec_2.begin(), h_vec_2.end(), d_vec_2.begin());

    cout << "sort before:" << d_vec_2.size() << endl;
    thrust::sort_by_key(d_vec_1.begin(), d_vec_1.end(), d_vec_2.begin());
    cout << "sort after:" << d_vec_2.size() << endl;

    travdirtime = stop_timer(&t);
    print_timer(travdirtime);       

    for(i=0; i<10; i++)
    {
	cout << d_vec_1[i] << "," << d_vec_2[i] << endl;
    }      

    size_t nBytes = nData * sizeof(unsigned long);
    size_t iBytes = nData * sizeof(float);

    cout << "#4 GPU to host - ip to long - host to GPU" << endl;

    start_timer(&t);
    unsigned long *h_A, *h_B, *h_C, *hostRef, *gpuRef;
    float *h_D;
    h_A     = (unsigned long *)malloc(nBytes);
    h_B     = (unsigned long *)malloc(nBytes);
    h_C     = (unsigned long *)malloc(nBytes);
    h_D     = (float *)malloc(iBytes);
    hostRef = (unsigned long *)malloc(nBytes);
    gpuRef  = (unsigned long *)malloc(nBytes);

    cout << "allocated" << endl;

    const string targetIP = std::string(argv[1]); 

    std::string stringIP;
    std::string IPstring;

    char del2 = '.';

    for (const auto subStr : split_string_2(targetIP, del2)) {
    	unsigned long ipaddr_src;
	ipaddr_src = atol(subStr.c_str());
	std::bitset<8> trans =  std::bitset<8>(ipaddr_src);
	std::string trans_string = trans.to_string();
	IPstring = IPstring + trans_string;
	}

     unsigned long s = bitset<32>(IPstring).to_ulong();
     unsigned long f = s;
     std::cout << targetIP << "," << f << endl;

    thrust::copy(d_vec_1.begin(), d_vec_1.end(), h_vec_1.begin());
    thrust::copy(d_vec_2.begin(), d_vec_2.end(), h_vec_2.begin());

     counter = 0;
     for(i=0;i<nData;i++)
     {	
    	h_A[i] = h_vec_1[i];
	h_B[i] = h_vec_2[i];
	h_C[i] = f;

        /*
	if(counter%10000000==0)
	  cout << "transfering " << counter << " vectors" << endl;
	*/
	counter = counter + 1;
    }

    cout << "done." << endl;

     for(i=0;i<10;i++)
     {	
    	cout << h_A[i] << "," << h_B[i] << "," << h_C[i] << endl;
    }

    travdirtime = stop_timer(&t);
    print_timer(travdirtime);       

    cout << "#5 transfer: host to GPU" << endl;

    start_timer(&t);  

    unsigned long *d_A, *d_B, *d_C;
    float *d_D;
    hipMalloc((unsigned long**)&d_A, nBytes);
    hipMalloc((unsigned long**)&d_B, nBytes);
    hipMalloc((unsigned long**)&d_C, nBytes);
    hipMalloc((float**)&d_D, iBytes);

    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_D, h_D, iBytes, hipMemcpyHostToDevice);

    travdirtime = stop_timer(&t);
    print_timer(travdirtime);       

    cout << "#6 substraction:" << endl;
    
    start_timer(&t);  

    dim3 block=nBLK;
    dim3 grid=nGRD;

    sumArraysOnGPU<<<grid, block>>>(d_B, d_C, d_D, nData);
    printf("Execution configure <<<%d, %d>>>\n", grid.x, block.x);

    travdirtime = stop_timer(&t);
    print_timer(travdirtime);       

    hipMemcpy(h_A, d_A, nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_B, d_B, nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_C, d_C, nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_D, d_D, iBytes, hipMemcpyDeviceToHost);

    thrust::host_vector<float> h_vec_D(nData);
    thrust::host_vector<unsigned long> h_vec_A(nData);
    thrust::host_vector<unsigned long> h_vec_B(nData);
    thrust::host_vector<unsigned long> h_vec_C(nData);
    
    thrust::device_vector<float> d_vec_D(nData);
    thrust::device_vector<unsigned long> d_vec_A(nData);
    thrust::device_vector<unsigned long> d_vec_B(nData);
    thrust::device_vector<unsigned long> d_vec_C(nData);

    counter = 0;
     for(i=0;i<nData;i++)
     {
        h_vec_A[i] = h_A[i];
    	h_vec_D[i] = h_D[i];
	h_vec_B[i] = h_B[i];
	h_vec_C[i] = h_C[i];

	/*
	if(counter%10000==0)
	  cout << "transfering " << counter << " vectors" << endl;
	*/
	counter = counter + 1;
    }

    /*
    for(i=0;i<20;i++)
    	cout << h_vec_A[i] << "," << h_vec_B[i] << "," << h_vec_B[i] << "," << h_vec_D[i] << endl;
    */
    
    counter = 0;
    for(i=0;i<h_vec_D.size();i++)
    {
	if(h_vec_D[i]==0)
		counter = counter + 1;
    }

    cout << counter << endl;

    cout << "#3 host->GPU - sort by key:" << endl;

    start_timer(&t);

    thrust::copy(h_vec_A.begin(), h_vec_A.end(), d_vec_A.begin());
    thrust::copy(h_vec_D.begin(), h_vec_D.end(), d_vec_D.begin());

    cout << "sort before:" << d_vec_D.size() << endl;
    thrust::sort_by_key(d_vec_D.begin(), d_vec_D.end(), d_vec_A.begin());
    cout << "sort after:" << d_vec_D.size() << endl;

    /*
    for(i=0;i<20;i++)
    	cout << d_vec_D[i] << "," << d_vec_A[i] << endl;
   */
   
    std::cout << d_vec_D.size() << std::endl;
    int N_prime = thrust::count_if(d_vec_D.begin(), d_vec_D.end(), is_smaller_than_10());
    std::cout << N_prime << std::endl;
    thrust::device_vector<int> d_b(N_prime);
    thrust::copy_if(d_vec_D.begin(), d_vec_D.end(), d_b.begin(), is_smaller_than_10());
    std::cout << d_b.size() << std::endl;

    travdirtime = stop_timer(&t);
    print_timer(travdirtime);       

    /*
    for(i=0;i<d_b.size();i++)
    	cout << d_vec_A[i] << "," << d_vec_D[i] << endl;
    */
}
