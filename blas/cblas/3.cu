
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>
#include <time.h>
#include <sys/time.h>
#define uS_PER_SEC 1000000
#define uS_PER_mS 1000
#define N  1000
#define M 1000

int main(){

    timeval t1, t2;
    float *matrix = (float *) malloc (N * M * sizeof(float));
// Starting the timer
    gettimeofday(&t1, NULL);
    float *matrixT = (float *) malloc (N * M * sizeof(float));
    for (int i = 0; i < N; i++)
        for (int j = 0; j < M; j++)
            matrixT[(j*N)+i] = matrix[(i*M)+j]; // matrix is obviously filled

//Ending the timer
    gettimeofday(&t2, NULL);
    float et1 = (((t2.tv_sec*uS_PER_SEC)+t2.tv_usec) - ((t1.tv_sec*uS_PER_SEC)+t1.tv_usec))/(float)uS_PER_mS;
    printf("CPU time = %fms\n", et1);

    float *h_matrixT , *d_matrixT , *d_matrix;
    h_matrixT = (float *) (malloc (N * M * sizeof(float)));
    hipMalloc((void **)&d_matrixT , N * M * sizeof(float));
    hipMalloc((void**)&d_matrix , N * M * sizeof(float));
    hipMemcpy(d_matrix , matrix , N * M * sizeof(float) , hipMemcpyHostToDevice);

//Starting the timer
    gettimeofday(&t1, NULL);

    const float alpha = 1.0;
    const float beta  = 0.0;
    // gettimeofday(&t1, NULL);
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    gettimeofday(&t1, NULL);
    hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, M, &alpha, d_matrix, M, &beta, d_matrix, N, d_matrixT, N);
    hipDeviceSynchronize();
    gettimeofday(&t2, NULL);
    hipblasDestroy(handle);
//Ending the timer
    float et2 = (((t2.tv_sec*uS_PER_SEC)+t2.tv_usec) - ((t1.tv_sec*uS_PER_SEC)+t1.tv_usec))/(float)uS_PER_mS;
    printf("GPU time = %fms\n", et2);

    hipMemcpy(h_matrixT , d_matrixT , N * M * sizeof(float) , hipMemcpyDeviceToHost);


    hipFree(d_matrix);
    hipFree(d_matrixT);
    return 0;
}


