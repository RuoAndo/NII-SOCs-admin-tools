#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#include <iostream>
#include <fstream>

using namespace std;

/*
 * Generate a vector of length N with random single-precision floating-point
 * values between 0 and 100.
 */
void generate_random_vector(int N, float **outX)
{
    int i;
    double rMax = (double)RAND_MAX;
    float *X = (float *)malloc(sizeof(float) * N);

    for (i = 0; i < N; i++)
    {
        int r = rand();
        double dr = (double)r;
        X[i] = (dr / rMax) * 100.0;
    }

    *outX = X;
}

__global__ void incrementTest(float *X, const int N)
{
    int i = threadIdx.x;
    X[i] = X[i] + 1;
}

// step1: X = (A * X0);
__global__ void F_1_X_A_X0(float *A, float *X0, float *X, const int N)
{
    // int i = blockIdx.x * blockDim.x + threadIdx.x;
    int i = threadIdx.x;
    X[i] = A[i] * X0[i];
}

// step2: P = (A * P0 * A.transpose()) + Q;
// F_2_P_A_P0_Q<<<grid, block>>>(dA, dP, dQ, dP, 1);
__global__ void F_2_P_A_P0_Q(float *P, float *A, float *P0, float *Q, const int N)
{
    // int i = blockIdx.x * blockDim.x + threadIdx.x;
    int i = threadIdx.x;
    P[i] = A[i] * P0[i] + Q[i];
    // P[i] = 5; // P[i] + 1;
}

// step3: K = ( P * H.transpose() ) * ( H * P * H.transpose() + R).inverse();
__global__ void F_3_K_P_H_R(float *P, float *H, float *R, float *K, const int N)
{
    // int i = blockIdx.x * blockDim.x + threadIdx.x;
    int i = threadIdx.x;
    K[i] = 1/P[i] * H[i] * (1/P[i] + R[i]);
}

// step4: X = X + K*(Z - H * X);
__global__ void F_4_X_K_Z_H(float *K, float *Z, float *H, float *X, const int N)
{
    // int i = blockIdx.x * blockDim.x + threadIdx.x;
    int i = threadIdx.x;
    X[i] = X[i] + K[i] * (Z[i] - H[i] * X[i]);
}

// step5: P = (I - K * H) * P;
__global__ void F_5_I_K_H_P(float *I, float *K, float *H, float *P, float *out_5_P, const int N)
{
    int i = threadIdx.x;
    out_5_P[i] = (I[i] - K[i]) * P[i];
    //P2[i] = 4.8; // K[i]; // * H[i]);
}

int main(int argc, char **argv)
{

    /* Fixed Matrix */
    /*
    MatrixXf A; //System dynamics matrix
    MatrixXf B; //Control matrix 
    MatrixXf H; //Mesaurement Adaptation matrix
    MatrixXf Q; //Process Noise Covariance matrix
    MatrixXf R; //Measurement Noise Covariance matrix
    MatrixXf I; //Identity matrix
    */

    /* Variable Matrix */
    /*
    VectorXf X; //(Current) State vector
    MatrixXf P; //State Covariance
    MatrixXf K; //Kalman Gain matrix
    */

    /* Inizial Value */
    /*
    VectorXf X0; //Initial State vector
    MatrixXf P0; //Initial State Covariance matrix
    */
    
   /*
    void KalmanFilter::predict(void){
    	 X = (A * X0);
  	 P = (A * P0 * A.transpose()) + Q;
    }
    */

    float alpha = 1.0f;
    float beta = 1.0f;

    hipblasHandle_t handle = 0;

    // VectorXf X; //Initial State vector
    float *X = 0;
    float *dX = 0;
    
    // MatrixXf A; //System dynamics matrix
    float *A = 0;
    float *dA = 0;

    // VectorXf X0; //Initial State vector
    float *X0 = 0;
    float *dX0 = 0;

    // MatrixXf P0; //Initial State Covariance matrix
    float *P = 0;
    float *dP = 0;

    // MatrixXf Q; //Process Noise Covariance matrix
    float *Q = 0;
    float *dQ = 0;

    // MatrixXf H; //Mesaurement Adaptation matrix
    float *H = 0;
    float *dH = 0;

    // MatrixXf P; //State Covariance
    float *P0 = 0;
    float *dP0 = 0;

    float *out_5_P = 0;
    float *dout_5_P = 0;

    // MatrixXf I; //State Covariance
    float *I = 0;
    float *dI = 0;

    float *K = 0;
    float *dK = 0;

    float *R = 0;
    float *dR = 0;

    float *Z = 0;
    float *dZ = 0;

    size_t nBytes = 1 * sizeof(float);
    X = (float *)malloc(nBytes);
    X0 = (float *)malloc(nBytes);
    A = (float *)malloc(nBytes);
    H = (float *)malloc(nBytes);
    P = (float *)malloc(nBytes);
    out_5_P = (float *)malloc(nBytes);
    P0 = (float *)malloc(nBytes);
    Q = (float *)malloc(nBytes);
    I = (float *)malloc(nBytes);
    R = (float *)malloc(nBytes);
    K = (float *)malloc(nBytes);
    Z = (float *)malloc(nBytes);

    X[0] = 1.0;
    X0[0] = 1.0;
    A[0] = 0.8;
    H[0] = 1.0;
    Q[0] = 1.0;
    I[0] = 1.0;
    P[0] = 1.0;
    P0[0] = 1.0;
    
    out_5_P[0] = 1.0;

    R[0] = 1.0;
    K[0] = 1.0;
    Z[0] = 1.0;
    
    CHECK(hipMalloc((void **)&dX, sizeof(float) * 1));
    CHECK(hipMalloc((void **)&dX0, sizeof(float) * 1));
    CHECK(hipMalloc((void **)&dA, sizeof(float) * 1));
    CHECK(hipMalloc((void **)&dP, sizeof(float) * 1));
    CHECK(hipMalloc((void **)&dP0, sizeof(float) * 1));
    CHECK(hipMalloc((void **)&dQ, sizeof(float) * 1));
    CHECK(hipMalloc((void **)&dH, sizeof(float) * 1));
    CHECK(hipMalloc((void **)&dK, sizeof(float) * 1));
    CHECK(hipMalloc((void **)&dR, sizeof(float) * 1));
    CHECK(hipMalloc((void **)&dZ, sizeof(float) * 1));
    CHECK(hipMalloc((void **)&dI, sizeof(float) * 1));

    CHECK(hipMalloc((void **)&dout_5_P, sizeof(float) * 1));

    hipMemcpy(dX, X, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dX0, X0, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dA, A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dP, P, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dP, P0, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dQ, Q, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dH, H, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dK, K, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dR, R, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dZ, Z, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dI, I, nBytes, hipMemcpyHostToDevice);

    hipMemcpy(dout_5_P, out_5_P, nBytes, hipMemcpyHostToDevice);

    dim3 grid = 1;
    dim3 block = 1;

    /*
    hipMemcpy(dX, X, nBytes, hipMemcpyDeviceToHost);
    for(int i=0; i<4; i++)
    {
    	hipMemcpy(dX, X, nBytes, hipMemcpyHostToDevice);
	incrementTest<<<grid, block>>>(dX, 1);
	hipMemcpy(X, dX, nBytes, hipMemcpyDeviceToHost);
    	printf("(i)X:%f \n", %d, X[0]);
    }
    */

    for(int i=0; i < 4; i++)
    {

    hipMemcpy(dA, A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dX, X, nBytes, hipMemcpyHostToDevice);
    F_1_X_A_X0<<<grid, block>>>(dA, dX, dX, 1);
    hipMemcpy(X, dX, nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(A, dA, nBytes, hipMemcpyDeviceToHost);
    printf("X:%f \n", X[0]);

    printf("STEP2B: P0:%f P:%f \n", P0[0],P[0]);
    hipMemcpy(dP, P, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dA, A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dP0, P0, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dQ, Q, nBytes, hipMemcpyHostToDevice);
    F_2_P_A_P0_Q<<<grid, block>>>(dP, dA, dP0, dQ, 1);
    hipMemcpy(P, dP, nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(A, dA, nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(P0, dP0, nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(Q, dQ, nBytes, hipMemcpyDeviceToHost);
    printf("STEP2A: P0:%f P:%f \n", P0[0],P[0]);

    /*
    void KalmanFilter::correct ( VectorXf Z ) {
    	 K = ( P * H.transpose() ) * ( H * P * H.transpose() + R).inverse();
      	 X = X + K*(Z - H * X);
	 P = (I - K * H) * P;
  	 X0 = X;
  	 P0 = P;	
	 }
    */

    Z[0] = 10;
    
    hipMemcpy(dK, K, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dP, P, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dH, H, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dR, R, nBytes, hipMemcpyHostToDevice);
    // __global__ void F_3_K_P_H_R(float *P, float *H, float *R, float *K, const int N)
    F_3_K_P_H_R<<<grid, block>>>(dP, dH, dR, dK, 1);
    hipMemcpy(K, dK, nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(P, dP, nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(H, dH, nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(R, dR, nBytes, hipMemcpyDeviceToHost);
    printf("K:%f \n", K[0]);

    hipMemcpy(dK, K, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dZ, Z, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dH, H, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dX, X, nBytes, hipMemcpyHostToDevice);
    F_4_X_K_Z_H<<<grid, block>>>(dX, dK, dZ, dH, 1);
    // printf("Execution configure <<<%d, %d>>>\n", grid.x, block.x);
    hipMemcpy(K, dK, nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(Z, dZ, nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(H, dH, nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(X, dX, nBytes, hipMemcpyDeviceToHost);    
    printf("X:%f \n", X[0]);



    printf("STEP5:K:%f \n", K[0]);
    printf("STEP5:P2:%f \n", out_5_P[0]);
    hipMemcpy(dI, I, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dK, K, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dH, H, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dP, P, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dout_5_P, out_5_P, nBytes, hipMemcpyHostToDevice);
    // __global__ void F_5_I_K_H_P(float *I, float *K, float *H, float *P, const int N)
    F_5_I_K_H_P<<<grid, block>>>(dI, dK, dH, dP, dout_5_P, 1);
    hipMemcpy(I, dI, nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(K, dK, nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(H, dH, nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(P, dP, nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(out_5_P, dout_5_P, nBytes, hipMemcpyDeviceToHost);

    // printf("STEP5:P:%f \n", P[0]);
    printf("STEP5:dout_5_P:%f \n", out_5_P[0]);

    X0[0] = X[0];
    P0[0] = out_5_P[0];
    }
    
    return 0;
}
